#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "core/math.hpp"
#include "data/curand_states.h"
#include "framework/config.h"
#include "helpers/ptc_update_helper.hpp"
#include "ptc_updater_gr_ks.h"
#include "systems/physics/geodesic_ks.hpp"
#include "systems/physics/metric_kerr_schild.hpp"
#include "utils/kernel_helper.hpp"
#include "utils/timer.h"
#include "utils/util_functions.h"

namespace Aperture {

namespace {

HOST_DEVICE void
gr_ks_boris_update(Scalar a, const vec_t<Scalar, 3> &x, vec_t<Scalar, 3> &u,
                   const vec_t<Scalar, 3> &B, const vec_t<Scalar, 3> &D,
                   Scalar dt, Scalar e_over_m) {
  Scalar sth = math::sin(x[1]);
  Scalar cth = math::cos(x[1]);

  Scalar g_13 = Metric_KS::g_13(a, x[0], sth, cth);
  Scalar g_11 = Metric_KS::g_11(a, x[0], sth, cth);
  Scalar g_22 = Metric_KS::g_22(a, x[0], sth, cth);
  Scalar g_33 = Metric_KS::g_33(a, x[0], sth, cth);
  Scalar gu11 = Metric_KS::gu11(a, x[0], sth, cth);
  Scalar gu22 = Metric_KS::gu22(a, x[0], sth, cth);
  Scalar gu33 = Metric_KS::gu33(a, x[0], sth, cth);
  Scalar gu13 = Metric_KS::gu13(a, x[0], sth, cth);
  Scalar sqrtg = Metric_KS::sqrt_gamma(a, x[0], sth, cth);

  vec_t<Scalar, 3> D_l = 0.0f;
  D_l[0] = g_11 * D[0] + g_13 * D[2];
  D_l[1] = g_22 * D[1];
  D_l[2] = g_33 * D[2] + g_13 * D[0];
  D_l *= 0.5f * dt * e_over_m * Metric_KS::alpha(a, x[0], sth, cth);

  vec_t<Scalar, 3> u_minus = u + D_l;

  vec_t<Scalar, 3> t =
      B * 0.5f * dt * e_over_m / Metric_KS::u0(a, x[0], sth, cth, u_minus);
  Scalar t2 = g_11 * t[0] * t[0] + g_22 * t[1] * t[1] + g_33 * t[2] * t[2] +
              2.0f * g_13 * t[0] * t[2];
  Scalar s = 2.0f / (1.0f + t2);

  vec_t<Scalar, 3> u_prime = u_minus;
  u_prime[0] += sqrtg * (gu22 * u_minus[1] * t[2] -
                         (gu33 * u_minus[2] + gu13 * u_minus[0]) * t[1]);
  u_prime[1] += sqrtg * ((gu33 * u_minus[2] + gu13 * u_minus[0]) * t[0] -
                         (gu11 * u_minus[0] + gu13 * u_minus[2]) * t[2]);
  u_prime[2] += sqrtg * ((gu11 * u_minus[0] + gu13 * u_minus[2]) * t[1] -
                         gu22 * u_minus[1] * t[0]);

  u = u_minus + D_l;
  u[0] += sqrtg *
          (gu22 * u_prime[1] * t[2] -
           (gu33 * u_prime[2] + gu13 * u_prime[0]) * t[1]) *
          s;
  u[1] += sqrtg *
          ((gu33 * u_prime[2] + gu13 * u_prime[0]) * t[0] -
           (gu11 * u_prime[0] + gu13 * u_prime[2]) * t[2]) *
          s;
  u[2] += sqrtg *
          ((gu11 * u_prime[0] + gu13 * u_prime[2]) * t[1] -
           gu22 * u_prime[1] * t[0]) *
          s;
}

HOST_DEVICE void
gr_ks_geodesic_advance(Scalar a, Scalar dt, vec_t<Scalar, 3> &x,
                       vec_t<Scalar, 3> &u, bool is_photon = false,
                       int n_iter = 3) {
  vec_t<Scalar, 3> x0 = x, x1 = x;
  vec_t<Scalar, 3> u0 = u, u1 = u;

  for (int i = 0; i < n_iter; i++) {
    auto x_tmp = (x0 + x) * 0.5;
    auto u_tmp = (u0 + u) * 0.5;
    x1 = x0 + geodesic_ks_x_rhs(a, x_tmp, u_tmp, is_photon) * dt;
    u1 = u0 + geodesic_ks_u_rhs(a, x_tmp, u_tmp, is_photon) * dt;
    x = x1;
    u = u1;
  }
}

template <typename Conf>
void
process_j_rho(vector_field<Conf>& j,
              typename ptc_updater_cu<Conf>::rho_ptrs_t& rho_ptrs,
              int num_species, const grid_ks_t<Conf>& grid,
              typename Conf::value_t dt) {
  kernel_launch(
      [dt, num_species] __device__(auto j, auto rho, auto grid_ptrs) {
        auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          auto pos = get_pos(idx, ext);
          // if (grid.is_in_bound(pos)) {
          auto w = grid.delta[0] * grid.delta[1] / dt;
          j[0][idx] *= w / grid_ptrs.Ad[0][idx];
          j[1][idx] *= w / grid_ptrs.Ad[1][idx];
          j[2][idx] /= grid_ptrs.Ad[2][idx];
          for (int n = 0; n < num_species; n++) {
            rho[n][idx] /= grid_ptrs.Ad[2][idx];
          }
          // }
          typename Conf::value_t theta = grid.template pos<1>(pos[1], true);
          if (math::abs(theta) < 0.1 * grid.delta[1]) {
            // j[1][idx] = 0.0;
            j[2][idx] = 0.0;
          }
        }
      },
      j.get_ptrs(), rho_ptrs.dev_ptr(), grid.get_grid_ptrs());
  // TODO: Is this necessary?
  // auto a = grid.a;
  // kernel_launch([a, dt, num_species] __device__(auto j, auto rho, auto grid_ptrs) {
  //       auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
  //       auto ext = grid.extent();
  //       for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
  //         auto pos = get_pos(idx, ext);
  //         Scalar r = grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], false));
  //         Scalar th = grid_ks_t<Conf>::theta(grid.template pos<1>(pos[1], true));
  //         auto sth = math::sin(th);
  //         auto cth = math::cos(th);

  //         Scalar rho_total = 0.0f;
  //         for (int n = 0; n < num_species; n++) {
  //           rho_total += 0.5f * (rho[n][idx] + rho[n][idx.inc_x()]);
  //         }
  //         j[0][idx] -= Metric_KS::beta1(a, r, sth, cth) * rho_total;
  //       }
  //   }, j.get_ptrs(), rho_ptrs.dev_ptr(), grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
ptc_outflow(particle_data_t& ptc, const grid_ks_t<Conf>& grid,
            int damping_length) {
  auto ptc_num = ptc.number();
  kernel_launch(
      [ptc_num, damping_length] __device__(auto ptc, auto gp) {
        auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
        for (auto n : grid_stride_range(0, ptc_num)) {
          auto c = ptc.cell[n];
          if (c == empty_cell) continue;

          auto idx = typename Conf::idx_t(c, grid.extent());
          auto pos = idx.get_pos();
          auto flag = ptc.flag[n];
          if (check_flag(flag, PtcFlag::ignore_EM)) continue;
          if (pos[0] > grid.dims[0] - damping_length + 2) {
            flag |= flag_or(PtcFlag::ignore_EM);
            ptc.flag[n] = flag;
          }
        }
      },
      ptc.get_dev_ptrs(), grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

}  // namespace

template <typename Conf>
ptc_updater_gr_ks_cu<Conf>::ptc_updater_gr_ks_cu(sim_environment &env,
                                                 const grid_ks_t<Conf> &grid,
                                                 const domain_comm<Conf> *comm)
    : ptc_updater_cu<Conf>(env, grid, comm), m_ks_grid(grid) {}

template <typename Conf>
void
ptc_updater_gr_ks_cu<Conf>::init() {
  ptc_updater_cu<Conf>::init();

  this->m_env.params().get_value("bh_spin", m_a);
  this->m_env.params().get_value("damping_length", m_damping_length);
}

template <typename Conf>
void
ptc_updater_gr_ks_cu<Conf>::register_data_components() {
  ptc_updater_cu<Conf>::register_data_components();
}

template <typename Conf>
void
ptc_updater_gr_ks_cu<Conf>::update_particles(double dt, uint32_t step) {
  value_t a = m_a;
  auto ptc_num = this->ptc->number();
  using spline_t = typename ptc_updater<Conf>::spline_t;
  using idx_t = typename Conf::idx_t;

  if (ptc_num > 0) {
    auto ptc_kernel = [a, ptc_num, dt, step] __device__(
                          auto ptc, auto B, auto D, auto J, auto Rho,
                          auto rho_interval) {
      auto &grid = dev_grid<Conf::dim, typename Conf::value_t>();
      auto ext = grid.extent();

      for (auto n : grid_stride_range(0, ptc_num)) {
        uint32_t cell = ptc.cell[n];
        if (cell == empty_cell) continue;

        auto idx = idx_t(cell, ext);
        auto pos = idx.get_pos();

        vec_t<value_t, 3> x(ptc.x1[n], ptc.x2[n], ptc.x3[n]);
        vec_t<value_t, 3> u(ptc.p1[n], ptc.p2[n], ptc.p3[n]);
        // value_t u0 = ptc.E[n];

        vec_t<value_t, 3> x_global = grid.pos_global(pos, x);
        x_global[0] = grid_ks_t<Conf>::radius(x_global[0]);
        x_global[1] = grid_ks_t<Conf>::theta(x_global[1]);

        auto flag = ptc.flag[n];
        int sp = get_ptc_type(flag);
        value_t q_over_m = dev_charges[sp] / dev_masses[sp];

        if (!check_flag(flag, PtcFlag::ignore_EM)) {
          auto interp = interpolator<spline_t, Conf::dim>{};
          vec_t<value_t, 3> Dp, Bp;
          Dp[0] = interp(D[0], x, idx, stagger_t(0b110));
          Dp[1] = interp(D[1], x, idx, stagger_t(0b101));
          Dp[2] = interp(D[2], x, idx, stagger_t(0b011));
          Bp[0] = interp(B[0], x, idx, stagger_t(0b001));
          Bp[1] = interp(B[1], x, idx, stagger_t(0b010));
          Bp[2] = interp(B[2], x, idx, stagger_t(0b100));
          gr_ks_boris_update(a, x_global, u, Bp, Dp, dt, q_over_m);
        }

        vec_t<value_t, 3> new_x = x_global;
        gr_ks_geodesic_advance(a, dt, new_x, u, false);
        new_x[0] = x[0] + (grid_ks_t<Conf>::from_radius(new_x[0]) -
                           grid_ks_t<Conf>::from_radius(x_global[0])) *
                              grid.inv_delta[0];
        new_x[1] = x[1] + (grid_ks_t<Conf>::from_theta(new_x[1]) -
                           grid_ks_t<Conf>::from_theta(x_global[1])) *
                              grid.inv_delta[1];
        vec_t<int, 2> dc = 0;
        dc[0] = math::floor(new_x[0]);
        dc[1] = math::floor(new_x[1]);
        pos[0] += dc[0];
        pos[1] += dc[1];
        ptc.x1[n] = new_x[0] - (Scalar)dc[0];
        ptc.x2[n] = new_x[1] - (Scalar)dc[1];
        ptc.x3[n] = new_x[2];
        ptc.cell[n] = idx_t(pos, ext).linear;

        ptc.p1[n] = u[0];
        ptc.p2[n] = u[1];
        ptc.p3[n] = u[2];
        ptc.E[n] = Metric_KS::u0(a, x_global[0], math::sin(x_global[1]),
                                 math::cos(x_global[1]), u);

        if (!check_flag(flag, PtcFlag::ignore_current)) {
          auto weight = dev_charges[sp] * ptc.weight[n];

          deposit_2d<spline_t>(x, new_x, dc, (new_x - x_global) / dt, J, Rho,
                               idx, weight, sp, true);
        }
      }
    };

    kernel_launch(ptc_kernel, this->ptc->get_dev_ptrs(),
                  this->B->get_const_ptrs(), this->E->get_const_ptrs(),
                  this->J->get_ptrs(), this->m_rho_ptrs.dev_ptr(),
                  this->m_rho_interval);
    process_j_rho(*(this->J), this->m_rho_ptrs, this->m_num_species, m_ks_grid, dt);

    if (this->m_comm == nullptr || this->m_comm->domain_info().is_boundary[1]) {
      ptc_outflow(*(this->ptc), m_ks_grid, m_damping_length);
    }

    CudaSafeCall(hipDeviceSynchronize());
    CudaCheckError();
  }
}

template <typename Conf>
void
ptc_updater_gr_ks_cu<Conf>::move_photons_2d(value_t dt, uint32_t step) {
  value_t a = m_a;
  auto ph_num = this->ph->number();

  if (ph_num > 0) {
    auto photon_kernel = [a, ph_num, dt, step] __device__(auto ph, auto rho_ph,
                                                          auto data_interval) {
      auto &grid = dev_grid<Conf::dim, typename Conf::value_t>();
      auto ext = grid.extent();

      for (size_t n : grid_stride_range(0, ph_num)) {
        uint32_t cell = ph.cell[n];
        if (cell == empty_cell) continue;
      }
    };

    kernel_launch(photon_kernel, this->ph->get_dev_ptrs(),
                  this->rho_ph->dev_ndptr(), this->m_data_interval);
  }
}

template <typename Conf>
void
ptc_updater_gr_ks_cu<Conf>::fill_multiplicity(int mult, value_t weight) {
  ptc_updater_cu<Conf>::fill_multiplicity(mult, weight);
}

template class ptc_updater_gr_ks_cu<Config<2>>;

}  // namespace Aperture
