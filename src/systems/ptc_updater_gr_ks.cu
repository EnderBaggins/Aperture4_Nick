#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "core/math.hpp"
#include "data/curand_states.h"
#include "framework/config.h"
#include "helpers/ptc_update_helper.hpp"
#include "ptc_updater_gr_ks.h"
#include "systems/physics/geodesic_ks.hpp"
#include "systems/physics/metric_kerr_schild.hpp"
#include "utils/kernel_helper.hpp"
#include "utils/timer.h"
#include "utils/util_functions.h"

namespace Aperture {

namespace {

template <typename value_t>
HOST_DEVICE void
gr_ks_boris_update(value_t a, const vec_t<value_t, 3> &x, vec_t<value_t, 3> &u,
                   const vec_t<value_t, 3> &B, const vec_t<value_t, 3> &D,
                   value_t dt, value_t e_over_m) {
  value_t sth = math::sin(x[1]);
  value_t cth = math::cos(x[1]);

  value_t g_13 = Metric_KS::g_13(a, x[0], sth, cth);
  value_t g_11 = Metric_KS::g_11(a, x[0], sth, cth);
  value_t g_22 = Metric_KS::g_22(a, x[0], sth, cth);
  value_t g_33 = Metric_KS::g_33(a, x[0], sth, cth);
  value_t gu11 = Metric_KS::gu11(a, x[0], sth, cth);
  value_t gu22 = Metric_KS::gu22(a, x[0], sth, cth);
  value_t gu33 = Metric_KS::gu33(a, x[0], sth, cth);
  value_t gu13 = Metric_KS::gu13(a, x[0], sth, cth);
  value_t sqrtg = Metric_KS::sqrt_gamma(a, x[0], sth, cth);

  vec_t<value_t, 3> D_l = 0.0f;
  D_l[0] = g_11 * D[0] + g_13 * D[2];
  D_l[1] = g_22 * D[1];
  D_l[2] = g_33 * D[2] + g_13 * D[0];
  D_l *= 0.5f * dt * e_over_m * Metric_KS::alpha(a, x[0], sth, cth);

  vec_t<value_t, 3> u_minus = u + D_l;

  vec_t<value_t, 3> t =
      B * 0.5f * dt * e_over_m / Metric_KS::u0(a, x[0], sth, cth, u_minus);
  value_t t2 = g_11 * t[0] * t[0] + g_22 * t[1] * t[1] + g_33 * t[2] * t[2] +
              2.0f * g_13 * t[0] * t[2];
  value_t s = 2.0f / (1.0f + t2);

  vec_t<value_t, 3> u_prime = u_minus;
  u_prime[0] += sqrtg * (gu22 * u_minus[1] * t[2] -
                         (gu33 * u_minus[2] + gu13 * u_minus[0]) * t[1]);
  u_prime[1] += sqrtg * ((gu33 * u_minus[2] + gu13 * u_minus[0]) * t[0] -
                         (gu11 * u_minus[0] + gu13 * u_minus[2]) * t[2]);
  u_prime[2] += sqrtg * ((gu11 * u_minus[0] + gu13 * u_minus[2]) * t[1] -
                         gu22 * u_minus[1] * t[0]);

  u = u_minus + D_l;
  u[0] += sqrtg *
          (gu22 * u_prime[1] * t[2] -
           (gu33 * u_prime[2] + gu13 * u_prime[0]) * t[1]) *
          s;
  u[1] += sqrtg *
          ((gu33 * u_prime[2] + gu13 * u_prime[0]) * t[0] -
           (gu11 * u_prime[0] + gu13 * u_prime[2]) * t[2]) *
          s;
  u[2] += sqrtg *
          ((gu11 * u_prime[0] + gu13 * u_prime[2]) * t[1] -
           gu22 * u_prime[1] * t[0]) *
          s;
}

template <typename value_t>
HOST_DEVICE void
gr_ks_geodesic_advance(value_t a, value_t dt, vec_t<value_t, 3> &x,
                       vec_t<value_t, 3> &u, bool is_photon = false,
                       int n_iter = 3) {
  vec_t<value_t, 3> x0 = x, x1 = x;
  vec_t<value_t, 3> u0 = u, u1 = u;

  for (int i = 0; i < n_iter; i++) {
    auto x_tmp = (x0 + x) * 0.5;
    auto u_tmp = (u0 + u) * 0.5;
    x1 = x0 + geodesic_ks_x_rhs(a, x_tmp, u_tmp, is_photon) * dt;
    u1 = u0 + geodesic_ks_u_rhs(a, x_tmp, u_tmp, is_photon) * dt;
    x = x1;
    u = u1;
  }
}

template <typename Conf>
void
process_j_rho(vector_field<Conf>& j,
              typename ptc_updater_cu<Conf>::rho_ptrs_t& rho_ptrs,
              int num_species, const grid_ks_t<Conf>& grid,
              typename Conf::value_t dt) {
  kernel_launch(
      [dt, num_species] __device__(auto j, auto rho, auto grid_ptrs) {
        auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          auto pos = get_pos(idx, ext);
          // if (grid.is_in_bound(pos)) {
          j[0][idx] /= grid_ptrs.Ad[0][idx] * dt;
          j[1][idx] /= grid_ptrs.Ad[1][idx] * dt;
          j[2][idx] /= grid_ptrs.Ad[2][idx];
          for (int n = 0; n < num_species; n++) {
            rho[n][idx] /= grid_ptrs.Ad[2][idx]; // A_phi is effectively dV
          }
          // }
          typename Conf::value_t theta = grid.template pos<1>(pos[1], true);
          if (theta < 0.1 * grid.delta[1] ||
              theta - M_PI < 0.1 * grid.delta[1]) {
            j[2][idx] = 0.0f;
          }
        }
      },
      j.get_ptrs(), rho_ptrs.dev_ptr(), grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
ptc_outflow(particle_data_t& ptc, const grid_ks_t<Conf>& grid,
            int damping_length) {
  auto ptc_num = ptc.number();
  kernel_launch(
      [ptc_num, damping_length] __device__(auto ptc, auto gp) {
        auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
        for (auto n : grid_stride_range(0, ptc_num)) {
          auto c = ptc.cell[n];
          if (c == empty_cell) continue;

          auto idx = typename Conf::idx_t(c, grid.extent());
          auto pos = idx.get_pos();
          auto flag = ptc.flag[n];
          if (check_flag(flag, PtcFlag::ignore_EM)) continue;
          if (pos[0] > grid.dims[0] - damping_length + 2) {
            flag |= flag_or(PtcFlag::ignore_EM);
            ptc.flag[n] = flag;
          }
        }
      },
      ptc.get_dev_ptrs(), grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

}  // namespace

template <typename Conf>
ptc_updater_gr_ks_cu<Conf>::ptc_updater_gr_ks_cu(sim_environment &env,
                                                 const grid_ks_t<Conf> &grid,
                                                 const domain_comm<Conf> *comm)
    : ptc_updater_cu<Conf>(env, grid, comm), m_ks_grid(grid) {}

template <typename Conf>
void
ptc_updater_gr_ks_cu<Conf>::init() {
  ptc_updater_cu<Conf>::init();

  this->m_env.params().get_value("bh_spin", m_a);
  this->m_env.params().get_value("damping_length", m_damping_length);
}

template <typename Conf>
void
ptc_updater_gr_ks_cu<Conf>::register_data_components() {
  ptc_updater_cu<Conf>::register_data_components();
}

template <typename Conf>
void
ptc_updater_gr_ks_cu<Conf>::update_particles(value_t dt, uint32_t step) {
  value_t a = m_a;
  auto ptc_num = this->ptc->number();
  Logger::print_info("Pushing {} particles in GR Kerr-Schild Coordinates!", ptc_num);
  using spline_t = typename ptc_updater<Conf>::spline_t;
  using idx_t = typename Conf::idx_t;

  this->J->init();
  for (auto& rho : this->Rho) rho->init();

  if (ptc_num > 0) {
    auto ptc_kernel = [a, ptc_num, dt, step] __device__(
                          auto ptc, auto B, auto D, auto J, auto Rho,
                          auto rho_interval) {
      auto &grid = dev_grid<Conf::dim, typename Conf::value_t>();
      auto ext = grid.extent();

      for (auto n : grid_stride_range(0, ptc_num)) {
        uint32_t cell = ptc.cell[n];
        if (cell == empty_cell) continue;

        auto idx = idx_t(cell, ext);
        auto pos = idx.get_pos();

        vec_t<value_t, 3> x(ptc.x1[n], ptc.x2[n], ptc.x3[n]);
        vec_t<value_t, 3> u(ptc.p1[n], ptc.p2[n], ptc.p3[n]);
        // value_t u0 = ptc.E[n];

        vec_t<value_t, 3> x_global = grid.pos_global(pos, x);
        x_global[0] = grid_ks_t<Conf>::radius(x_global[0]);
        x_global[1] = grid_ks_t<Conf>::theta(x_global[1]);

        auto flag = ptc.flag[n];
        int sp = get_ptc_type(flag);
        value_t q_over_m = dev_charges[sp] / dev_masses[sp];

        if (!check_flag(flag, PtcFlag::ignore_EM)) {
          auto interp = interpolator<spline_t, Conf::dim>{};
          vec_t<value_t, 3> Dp, Bp;
          Dp[0] = interp(D[0], x, idx, stagger_t(0b110));
          Dp[1] = interp(D[1], x, idx, stagger_t(0b101));
          Dp[2] = interp(D[2], x, idx, stagger_t(0b011));
          Bp[0] = interp(B[0], x, idx, stagger_t(0b001));
          Bp[1] = interp(B[1], x, idx, stagger_t(0b010));
          Bp[2] = interp(B[2], x, idx, stagger_t(0b100));
          // This step only updates u
          gr_ks_boris_update(a, x_global, u, Bp, Dp, dt, q_over_m);
        }

        vec_t<value_t, 3> new_x = x_global;
        // Both new_x and u are updated
        gr_ks_geodesic_advance(a, dt, new_x, u, false);

        // printf("---- cylindrical radius is %f, z is %f\n", new_x[0] * math::sin(new_x[1]),
        //        new_x[0] * math::cos(new_x[1]));
        new_x[0] = x[0] + (grid_ks_t<Conf>::from_radius(new_x[0]) -
                           grid_ks_t<Conf>::from_radius(x_global[0])) *
                              grid.inv_delta[0];
        new_x[1] = x[1] + (grid_ks_t<Conf>::from_theta(new_x[1]) -
                           grid_ks_t<Conf>::from_theta(x_global[1])) *
                              grid.inv_delta[1];
        vec_t<int, 2> dc = 0;
        dc[0] = math::floor(new_x[0]);
        dc[1] = math::floor(new_x[1]);
        // if (dc[0] > 1 || dc[0] < -1 || dc[1] > 1 || dc[1] < -1)
        //   printf("----------------- Error: moved more than 1 cell!");
        pos[0] += dc[0];
        pos[1] += dc[1];
        ptc.x1[n] = new_x[0] - (value_t)dc[0];
        ptc.x2[n] = new_x[1] - (value_t)dc[1];
        ptc.x3[n] = new_x[2];
        ptc.cell[n] = idx_t(pos, ext).linear;

        ptc.p1[n] = u[0];
        ptc.p2[n] = u[1];
        ptc.p3[n] = u[2];
        ptc.E[n] = Metric_KS::u0(a, x_global[0], math::sin(x_global[1]),
                                 math::cos(x_global[1]), u);

        if (!check_flag(flag, PtcFlag::ignore_current)) {
          auto weight = dev_charges[sp] * ptc.weight[n] * grid.delta[0] * grid.delta[1];

          deposit_2d<spline_t>(x, new_x, dc, (new_x - x_global) / dt, J, Rho,
                               idx, weight, sp, true);
        }
      }
    };

    kernel_launch(ptc_kernel, this->ptc->get_dev_ptrs(),
                  this->B->get_const_ptrs(), this->E->get_const_ptrs(),
                  this->J->get_ptrs(), this->m_rho_ptrs.dev_ptr(),
                  this->m_rho_interval);
    process_j_rho(*(this->J), this->m_rho_ptrs, this->m_num_species, m_ks_grid, dt);

    if (this->m_comm == nullptr || this->m_comm->domain_info().is_boundary[1]) {
      ptc_outflow(*(this->ptc), m_ks_grid, m_damping_length);
    }

    CudaSafeCall(hipDeviceSynchronize());
    CudaCheckError();
  }
}

template <typename Conf>
void
ptc_updater_gr_ks_cu<Conf>::move_photons_2d(value_t dt, uint32_t step) {
  value_t a = m_a;
  auto ph_num = this->ph->number();

  if (ph_num > 0) {
    auto photon_kernel = [a, ph_num, dt, step] __device__(auto ph, auto rho_ph,
                                                          auto data_interval) {
      auto &grid = dev_grid<Conf::dim, typename Conf::value_t>();
      auto ext = grid.extent();

      for (size_t n : grid_stride_range(0, ph_num)) {
        uint32_t cell = ph.cell[n];
        if (cell == empty_cell) continue;
      }
    };

    kernel_launch(photon_kernel, this->ph->get_dev_ptrs(),
                  this->rho_ph->dev_ndptr(), this->m_data_interval);
  }
}

template <typename Conf>
void
ptc_updater_gr_ks_cu<Conf>::fill_multiplicity(int mult, value_t weight) {
  ptc_updater_cu<Conf>::fill_multiplicity(mult, weight);
}

template class ptc_updater_gr_ks_cu<Config<2>>;

}  // namespace Aperture
