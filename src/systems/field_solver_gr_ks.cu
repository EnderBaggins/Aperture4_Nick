#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "core/cuda_control.h"
#include "core/multi_array_exp.hpp"
#include "core/ndsubset_dev.hpp"
#include "field_solver_gr_ks.h"
#include "framework/config.h"
#include "framework/environment.h"
#include "utils/kernel_helper.hpp"
#include "utils/timer.h"
#include <hipsparse.h>

namespace Aperture {

namespace {

hipsparseHandle_t sp_handle;
// buffer<char> sp_buffer;

// template <typename Conf, typename ArrayType>
// HD_INLINE typename Conf::value_t
// H_r(const ArrayType& Br, const ArrayType& Bph, const typename Conf::idx_t&
// idx,
//     const index_t<Conf::dim>& pos, const Grid<Conf::dim>& grid,
//     typename Conf::value_t a) {
//   using namespace Aperture::Metric_KS;
//   auto r = grid_ks_t<Conf>::radius(grid.pos<0>(pos[0], true));
//   auto th = grid_ks_t<Conf>::theta(grid.pos<1>(pos[1], false));

//   return ag_11(a, r, th) * Br[idx] +
//          ag_13(a, r, th) * 0.5f * (Bph[idx] + Bph[idx.dec_x(1)]);
// }

// template <typename Conf, typename ArrayType>
// HD_INLINE typename Conf::value_t
// H_th(const ArrayType& Bth, const ArrayType& Dph,
//      const typename Conf::idx_t& idx, const index_t<Conf::dim>& pos,
//      const Grid<Conf::dim>& grid, typename Conf::value_t a) {
//   using namespace Aperture::Metric_KS;
//   auto r = grid_ks_t<Conf>::radius(grid.pos<0>(pos[0], false));
//   auto th = grid_ks_t<Conf>::theta(grid.pos<1>(pos[1], true));

//   return ag_22(a, r, th) * Bth[idx] +
//          sq_gamma_beta(a, r, th) * 0.5f * (Dph[idx] + Dph[idx.inc_x(1)]);
// }

// template <typename Conf, typename ArrayType>
// HD_INLINE typename Conf::value_t
// H_ph(const ArrayType& Bph, const ArrayType& Br, const ArrayType& Dth,
//      const typename Conf::idx_t& idx, const index_t<Conf::dim>& pos,
//      const Grid<Conf::dim>& grid, typename Conf::value_t a) {
//   using namespace Aperture::Metric_KS;
//   auto r = grid_ks_t<Conf>::radius(grid.pos<0>(pos[0], false));
//   auto th = grid_ks_t<Conf>::theta(grid.pos<1>(pos[1], false));

//   return ag_33(a, r, th) * Bph[idx] +
//          ag_13(a, r, th) * 0.5f * (Br[idx] + Br[idx.inc_x(1)]) -
//          sq_gamma_beta(a, r, th) * (Dth[idx] + Dth[idx.inc_x(1)]);
// }

// template <typename Conf, typename ArrayType>
// HD_INLINE typename Conf::value_t
// E_r(const ArrayType& Dr, const ArrayType& Dph, const typename Conf::idx_t&
// idx,
//     const index_t<Conf::dim>& pos, const Grid<Conf::dim>& grid,
//     typename Conf::value_t a) {
//   using namespace Aperture::Metric_KS;
//   auto r = grid_ks_t<Conf>::radius(grid.pos<0>(pos[0], false));
//   auto th = grid_ks_t<Conf>::theta(grid.pos<1>(pos[1], true));

//   return ag_11(a, r, th) * Dr[idx] +
//          ag_13(a, r, th) * 0.5f * (Dph[idx] + Dph[idx.inc_x(1)]);
// }

// template <typename Conf, typename ArrayType>
// HD_INLINE typename Conf::value_t
// E_th(const ArrayType& Dth, const ArrayType& Bph,
//      const typename Conf::idx_t& idx, const index_t<Conf::dim>& pos,
//      const Grid<Conf::dim>& grid, typename Conf::value_t a) {
//   using namespace Aperture::Metric_KS;
//   auto r = grid_ks_t<Conf>::radius(grid.pos<0>(pos[0], true));
//   auto th = grid_ks_t<Conf>::theta(grid.pos<1>(pos[1], false));

//   return ag_22(a, r, th) * Dth[idx] -
//          sq_gamma_beta(a, r, th) * 0.5f * (Bph[idx] + Bph[idx.dec_x(1)]);
// }

// template <typename Conf, typename ArrayType>
// HD_INLINE typename Conf::value_t
// E_ph(const ArrayType& Dph, const ArrayType& Dr, const ArrayType& Bth,
//      const typename Conf::idx_t& idx, const index_t<Conf::dim>& pos,
//      const Grid<Conf::dim>& grid, typename Conf::value_t a) {
//   using namespace Aperture::Metric_KS;
//   auto r = grid_ks_t<Conf>::radius(grid.pos<0>(pos[0], true));
//   auto th = grid_ks_t<Conf>::theta(grid.pos<1>(pos[1], true));
//   auto sth = math::sin(th);
//   auto cth = math::cos(th);

//   return ag_33(a, r, sth, cth) * Dph[idx] +
//          ag_13(a, r, sth, cth) * 0.5f * (Dr[idx] + Dr[idx.dec_x(1)]) -
//          sq_gamma_beta(a, r, sth, cth) * 0.5f * (Bth[idx] +
//          Bth[idx.dec_x(1)]);
// }

template <typename Conf>
void
axis_boundary_e(vector_field<Conf>& D, const grid_ks_t<Conf>& grid) {
  typedef typename Conf::idx_t idx_t;
  kernel_launch(
      [] __device__(auto D) {
        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto n0 : grid_stride_range(0, grid.dims[0])) {
          auto n1_0 = grid.guard[1];
          auto n1_pi = grid.dims[1] - grid.guard[1];
          if (abs(grid_ks_t<Conf>::theta(grid.template pos<1>(n1_0, true))) <
              0.1f * grid.delta[1]) {
            // At the theta = 0 axis

            // Set E_phi and B_theta to zero
            auto idx = idx_t(index_t<2>(n0, n1_0), ext);
            D[2][idx] = 0.0f;
            // e[1][idx] = 0.0;
            D[1][idx.dec_y()] = D[1][idx];
            D[0][idx.dec_y()] = D[0][idx];
            // e[0][idx] = 0.0f;
          }
          // printf("boundary pi at %f\n", grid.template pos<1>(n1_pi, true));
          if (abs(grid_ks_t<Conf>::theta(grid.template pos<1>(n1_pi, true)) -
                  M_PI) < 0.1f * grid.delta[1]) {
            // At the theta = pi axis
            auto idx = idx_t(index_t<2>(n0, n1_pi), ext);
            D[2][idx] = 0.0f;
            // e[1][idx] = 0.0;
            D[1][idx] = D[1][idx.dec_y()];
            D[0][idx] = D[0][idx.dec_y()];
            // e[0][idx] = 0.0f;
          }
        }
      },
      D.get_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
axis_boundary_b(vector_field<Conf>& B, const grid_ks_t<Conf>& grid) {
  typedef typename Conf::idx_t idx_t;
  kernel_launch(
      [] __device__(auto B) {
        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto n0 : grid_stride_range(0, grid.dims[0])) {
          for (int n1_0 = grid.guard[1]; n1_0 >= 0; n1_0--) {
            if (grid_ks_t<Conf>::theta(grid.template pos<1>(n1_0, true)) <
                0.1f * grid.delta[1]) {
              // At the theta = 0 axis

              // Set E_phi and B_theta to zero
              auto idx = idx_t(index_t<2>(n0, n1_0), ext);
              B[1][idx] = 0.0f;
              B[2][idx] = 0.0f;
              // B[2][idx.dec_y()] = B[2][idx];
              B[0][idx.dec_y()] = B[0][idx];
            }
          }
          for (int n1_pi = grid.dims[1] - grid.guard[1]; n1_pi <= grid.dims[1] - 1; n1_pi++) {
            // printf("boundary pi at %f\n", grid.template pos<1>(n1_pi, true));
            if (abs(grid_ks_t<Conf>::theta(grid.template pos<1>(n1_pi, true)) -
                    M_PI) < 0.1f * grid.delta[1]) {
              // At the theta = pi axis
              auto idx = idx_t(index_t<2>(n0, n1_pi), ext);
              B[1][idx] = 0.0f;
              // B[1][idx.dec_y()] = 0.0f;
              // B[2][idx] = B[2][idx.dec_y()];
              B[2][idx] = 0.0f;
              B[2][idx.dec_y()] = 0.0f;
              B[0][idx] = B[0][idx.dec_y()];
            }
          }
        }
      },
      B.get_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
horizon_boundary(vector_field<Conf>& D, vector_field<Conf>& B,
                 const vector_field<Conf>& D0, const vector_field<Conf>& B0,
                 const grid_ks_t<Conf>& grid) {
  kernel_launch(
      [] __device__(auto D, auto D0, auto B, auto B0) {
        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto n1 : grid_stride_range(0, grid.dims[1])) {
          for (int n0 = 0; n0 < grid.guard[0] + 2; n0++) {
            auto pos = index_t<2>(n0, n1);
            auto idx = Conf::idx(pos, ext);

            B[0][idx] = B0[0][idx];
            B[1][idx] = B0[1][idx];
            B[2][idx] = B0[2][idx];
            D[0][idx] = D0[0][idx];
            D[1][idx] = D0[1][idx];
            D[2][idx] = D0[2][idx];
          }
        }
      },
      D.get_ptrs(), D0.get_ptrs(), B.get_ptrs(), B0.get_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
compute_flux(scalar_field<Conf>& flux, const vector_field<Conf>& b,
             const grid_ks_t<Conf>& grid) {
  flux.init();
  auto ext = grid.extent();
  kernel_launch(
      [ext] __device__(auto flux, auto b, auto a) {
        auto& grid = dev_grid<Conf::dim>();
        for (auto n0 : grid_stride_range(0, grid.dims[0])) {
          auto r = grid_ks_t<Conf>::radius(grid.template pos<0>(n0, true));

          for (int n1 = grid.guard[1]; n1 < grid.dims[1] - grid.guard[1];
               n1++) {
            Scalar th = grid_ks_t<Conf>::theta(grid.template pos<1>(n1, false));
            Scalar th_p =
                grid_ks_t<Conf>::theta(grid.template pos<1>(n1 + 1, true));
            Scalar th_m =
                grid_ks_t<Conf>::theta(grid.template pos<1>(n1, true));
            auto dth = th_p - th_m;

            auto pos = index_t<Conf::dim>(n0, n1);
            auto idx = typename Conf::idx_t(pos, ext);

            flux[idx] = flux[idx.dec_y()] +
                        b[0][idx] * Metric_KS::sqrt_gamma(a, r, th) * dth;
          }
        }
      },
      flux.dev_ndptr(), b.get_ptrs(), grid.a);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

}  // namespace

template <typename Conf>
field_solver_gr_ks_cu<Conf>::~field_solver_gr_ks_cu() {
  hipsparseDestroy(sp_handle);

  // sp_buffer.resize(0);
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::init() {
  field_solver<Conf>::init();

  this->m_env.params().get_value("bh_spin", m_a);

  hipsparseCreate(&sp_handle);
  auto ext = this->m_grid.extent();

  m_tmp_rhs.set_memtype(MemType::device_only);
  m_tmp_rhs.resize(ext);

  m_tmp_prev_field.set_memtype(MemType::device_only);
  m_tmp_prev_field.resize(ext);

  m_tri_dl.set_memtype(MemType::device_only);
  m_tri_dl.resize(this->m_grid.dims[0]);
  m_tri_dl.assign_dev(0.0f);

  m_tri_d.set_memtype(MemType::device_only);
  m_tri_d.resize(this->m_grid.dims[0]);
  m_tri_d.assign_dev(0.0f);

  m_tri_du.set_memtype(MemType::device_only);
  m_tri_du.resize(this->m_grid.dims[0]);
  m_tri_du.assign_dev(0.0f);

  size_t buffer_size;
#if USE_DOUBLE
  hipsparseDgtsv2_bufferSizeExt(sp_handle, ext[0], ext[1], m_tri_dl.dev_ptr(),
                               m_tri_d.dev_ptr(), m_tri_du.dev_ptr(),
                               m_tmp_rhs.dev_ptr(), ext[0], &buffer_size);
#else
  hipsparseSgtsv2_bufferSizeExt(sp_handle, ext[0], ext[1], m_tri_dl.dev_ptr(),
                               m_tri_d.dev_ptr(), m_tri_du.dev_ptr(),
                               m_tmp_rhs.dev_ptr(), ext[0], &buffer_size);
#endif
  Logger::print_info("tri-diagonal buffer size is {}", buffer_size);
  sp_buffer.set_memtype(MemType::device_only);
  sp_buffer.resize(buffer_size);
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::register_data_components() {
  field_solver_cu<Conf>::register_data_components();

  flux = this->m_env.template register_data<scalar_field<Conf>>(
      "flux", this->m_grid, field_type::vert_centered);
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::solve_tridiagonal() {
  // Solve the assembled tri-diagonal system using cusparse
  hipsparseStatus_t status;
  auto ext = this->m_grid.extent();
#if USE_DOUBLE
  status = hipsparseDgtsv2(sp_handle, ext[0], ext[1], m_tri_dl.dev_ptr(),
                          m_tri_d.dev_ptr(), m_tri_du.dev_ptr(),
                          m_tmp_rhs.dev_ptr(), ext[0], sp_buffer.dev_ptr());
#else
  status = hipsparseSgtsv2(sp_handle, ext[0], ext[1], m_tri_dl.dev_ptr(),
                          m_tri_d.dev_ptr(), m_tri_du.dev_ptr(),
                          m_tmp_rhs.dev_ptr(), ext[0], sp_buffer.dev_ptr());
#endif
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    Logger::print_err("cusparse failure at Bth update! Error code {}", status);
  }
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update_Bth(vector_field<Conf>& B,
                                        const vector_field<Conf>& B0,
                                        const vector_field<Conf>& D,
                                        const vector_field<Conf>& D0,
                                        value_t dt) {
  m_tmp_rhs.assign_dev(0.0f);
  m_tmp_prev_field.copy_from(B[1]);

  // First assemble the right hand side and the diagonals of the tri-diagonal
  // equation
  kernel_launch(
      [dt] __device__(auto B, auto B0, auto D, auto D0, auto rhs, auto d,
                      auto dl, auto du, auto a) {
        using namespace Metric_KS;

        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          auto pos = get_pos(idx, ext);
          value_t r =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], false));
          value_t r_sp =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0] + 1, true));
          value_t r_sm =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], true));
          value_t dr = r_sp - r_sm;
          value_t th = grid.template pos<1>(pos[1], true);
          // if (math::abs(th) < TINY) th = sgn(th) * 1.0e-4;
          if (math::abs(th) < TINY)
            // th = (th < 0.0f ? -1.0f : 1.0f) * 0.01 * grid.delta[1];
            th = 0.01f * grid.delta[1];

          value_t cth = math::cos(th);
          value_t sth = math::sin(th);
          value_t prefactor = dt / (sqrt_gamma(a, r, sth, cth) * dr);

          // if (grid.is_in_bound(pos)) {
          if (pos[0] < grid.dims[0] - 1) {
            auto Eph1 = ag_33(a, r_sp, sth, cth) * D[2][idx.inc_x()] +
                        ag_13(a, r_sp, sth, cth) * 0.5f *
                            // (D[0][idx.inc_x()] + D[0][idx] +
                            (D[0][idx.inc_x()] + D[0][idx]) +
                        // D0[0][idx.inc_x()] + D0[0][idx]) +
                        sq_gamma_beta(a, r_sp, sth, cth) * 0.5f *
                            // (B[1][idx.inc_x()] + B[1][idx] +
                            (B[1][idx.inc_x()] + B[1][idx]);
            // B0[1][idx.inc_x()] + B0[1][idx]);

            auto Eph0 = ag_33(a, r_sm, sth, cth) * D[2][idx] +
                        ag_13(a, r_sm, sth, cth) * 0.5f *
                            // (D[0][idx] + D[0][idx.dec_x()] + D0[0][idx] +
                            //  D0[0][idx.dec_x()]) +
                            (D[0][idx] + D[0][idx.dec_x()]) +
                        sq_gamma_beta(a, r_sm, sth, cth) * 0.5f *
                            // (B[1][idx] + B[1][idx.dec_x()] + B0[1][idx] +
                            //  B0[1][idx.dec_x()]);
                            (B[1][idx] + B[1][idx.dec_x()]);

            rhs[idx] = B[1][idx] - prefactor * (Eph0 - Eph1);
          }
          // if (pos[1] == 2 && pos[0] == 200)
          //   printf("rhs is %f, D0 is %f, B1 is %f\n", rhs[idx], D[0][idx], B[1][idx]);

          value_t du_coef = prefactor * 0.5f * sq_gamma_beta(a, r_sp, sth, cth);
          value_t dl_coef =
              -prefactor * 0.5f * sq_gamma_beta(a, r_sm, sth, cth);
          // if (pos[0] == 6 && pos[1] == 3)
          //   printf("du is %f, d is %f\n", du_coef, 1.0f - (du_coef +
          //   dl_coef));
          d[pos[0]] = 1.0f - (du_coef + dl_coef);

          du[pos[0]] = du_coef;
          dl[pos[0]] = dl_coef;

          // if (pos[0] == 6 && pos[1] == 300) {
          //   printf("d is %f, du is %f, dl is %f\n", d[pos[0]], du[pos[0]], dl[pos[0]]);
          // }
        }
      },
      B.get_const_ptrs(), B0.get_const_ptrs(), D.get_const_ptrs(),
      D0.get_const_ptrs(), m_tmp_rhs.dev_ndptr(), m_tri_d.dev_ptr(),
      m_tri_dl.dev_ptr(), m_tri_du.dev_ptr(), m_a);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();

  solve_tridiagonal();

  if (this->m_comm == nullptr || this->m_comm->domain_info().is_boundary[2]) {
    kernel_launch(
        [] __device__(auto f) {
          auto& grid = dev_grid<Conf::dim>();
          auto ext = grid.extent();
          for (auto n0 : grid_stride_range(0, grid.dims[0])) {
            int n1 = grid.guard[1];
            auto idx = Conf::idx({n0, n1}, ext);

            f[idx] = 0.0f;
          }
        },
        m_tmp_rhs.dev_ndptr());
  }

  B[1].copy_from(m_tmp_rhs);
  select_dev(m_tmp_prev_field) = m_tmp_rhs * 0.5f + m_tmp_prev_field * 0.5f;
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update_Bph(vector_field<Conf>& B,
                                        const vector_field<Conf>& B0,
                                        const vector_field<Conf>& D,
                                        const vector_field<Conf>& D0,
                                        value_t dt) {
  m_tmp_rhs.assign_dev(0.0f);

  // First assemble the right hand side and the diagonals of the tri-diagonal
  // equation
  kernel_launch(
      [dt] __device__(auto B, auto B0, auto D, auto D0, auto rhs, auto d,
                      auto dl, auto du, auto a) {
        using namespace Metric_KS;
        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          auto pos = get_pos(idx, ext);
          value_t r =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], false));
          value_t r_sp =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0] + 1, true));
          value_t r_sm =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], true));
          value_t dr = r_sp - r_sm;

          value_t th = grid.template pos<1>(pos[1], false);
          value_t th_sp = grid.template pos<1>(pos[1] + 1, true);
          value_t th_sm = grid.template pos<1>(pos[1], true);
          value_t dth = th_sp - th_sm;
          if (th_sm < TINY) th_sm = 0.01f * grid.delta[1];

          value_t cth = math::cos(th);
          value_t sth = math::sin(th);
          value_t prefactor = dt / (sqrt_gamma(a, r, th) * dr * dth);

          // if (grid.is_in_bound(pos)) {
          if (pos[0] > 0 && pos[0] < grid.dims[0] - 1 &&
              pos[1] < grid.dims[0] - 1) {
            auto Er1 = ag_11(a, r, th_sp) * D[0][idx.inc_y()] +
                       ag_13(a, r, th_sp) * 0.5f *
                           // (D[2][idx.inc_y()] + D[2][idx.inc_y().inc_x()] +
                           //  D0[2][idx.inc_y()] + D0[2][idx.inc_y().inc_x()]);
                           (D[2][idx.inc_y()] + D[2][idx.inc_y().inc_x()]);

            auto Er0 = ag_11(a, r, th_sm) * D[0][idx] +
                       ag_13(a, r, th_sm) * 0.5f *
                           // (D[2][idx] + D[2][idx.inc_x()] + D0[2][idx] +
                           //  D0[2][idx.inc_x()]);
                           (D[2][idx] + D[2][idx.inc_x()]);

            auto Eth1 = ag_22(a, r, sth, cth) * D[1][idx.inc_x()] -
                        sq_gamma_beta(a, r, sth, cth) * 0.5f *
                            // (B[2][idx.inc_x()] + B[2][idx] +
                            //  B0[2][idx.inc_x()] + B0[2][idx]);
                            (B[2][idx.inc_x()] + B[2][idx]);

            auto Eth0 = ag_22(a, r, sth, cth) * D[1][idx] -
                        sq_gamma_beta(a, r, sth, cth) * 0.5f *
                            // (B[2][idx] + B[2][idx.dec_x()] + B0[2][idx] +
                            //  B0[2][idx.dec_x()]);
                            (B[2][idx] + B[2][idx.dec_x()]);

            rhs[idx] = B[2][idx] -
                       prefactor * (dr * (Er0 - Er1) + dth * (Eth1 - Eth0));
          }

          value_t du_coef = prefactor * dth * 0.5f * sq_gamma_beta(a, r_sp, th);
          value_t dl_coef =
              -prefactor * dth * 0.5f * sq_gamma_beta(a, r_sm, th);
          d[pos[0]] = 1.0f - (du_coef + dl_coef);

          du[pos[0]] = du_coef;
          dl[pos[0]] = dl_coef;

          // if (pos[0] == 6 && pos[1] == 300) {
          //   printf("d is %f, du is %f, dl is %f\n", d[pos[0]], du[pos[0]], dl[pos[0]]);
          // }
        }
      },
      B.get_const_ptrs(), B0.get_const_ptrs(), D.get_const_ptrs(),
      D0.get_const_ptrs(), m_tmp_rhs.dev_ndptr(), m_tri_d.dev_ptr(),
      m_tri_dl.dev_ptr(), m_tri_du.dev_ptr(), m_a);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();

  solve_tridiagonal();

  B[2].copy_from(m_tmp_rhs);
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update_Br(vector_field<Conf>& B,
                                       const vector_field<Conf>& B0,
                                       const vector_field<Conf>& D,
                                       const vector_field<Conf>& D0,
                                       value_t dt) {
  kernel_launch(
      [dt] __device__(auto B, auto B0, auto D, auto D0, auto tmp_field,
                      auto a) {
        using namespace Metric_KS;
        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          auto pos = get_pos(idx, ext);
          if (grid.is_in_bound(pos)) {
            value_t r =
                grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], true));

            value_t th = grid.template pos<1>(pos[1], false);
            value_t th_sp = grid.template pos<1>(pos[1] + 1, true);
            value_t th_sm = grid.template pos<1>(pos[1], true);
            value_t dth = th_sp - th_sm;
            if (th_sm < TINY) th_sm = 0.01f * grid.delta[1];

            value_t prefactor = dt / (sqrt_gamma(a, r, th) * dth);

            value_t sth = math::sin(th_sp);
            value_t cth = math::cos(th_sp);
            value_t Eph1 =
                ag_33(a, r, sth, cth) * D[2][idx.inc_y()] +
                ag_13(a, r, sth, cth) * 0.5f *
                    // (D[0][idx.inc_y()] + D[0][idx.inc_y().dec_x()] +
                    //  D0[0][idx.inc_y()] + D0[0][idx.inc_y().dec_x()]) +
                    (D[0][idx.inc_y()] + D[0][idx.inc_y().dec_x()]) +
                sq_gamma_beta(a, r, sth, cth) * 0.5f *
                    // (B[1][idx.inc_y()] + B[1][idx.inc_y().dec_x()] +
                    //  B0[1][idx.inc_y()] + B0[1][idx.inc_y().dec_x()]);
                    (tmp_field[idx.inc_y()] + tmp_field[idx.inc_y().dec_x()]);

            sth = math::sin(th_sm);
            cth = math::cos(th_sm);
            value_t Eph0 = ag_33(a, r, sth, cth) * D[2][idx] +
                           ag_13(a, r, sth, cth) * 0.5f *
                               // (D[0][idx] + D[0][idx.dec_x()] +
                               //  D0[0][idx] + D0[0][idx.dec_x()]) +
                               (D[0][idx] + D[0][idx.dec_x()]) +
                           sq_gamma_beta(a, r, sth, cth) * 0.5f *
                               // (B[1][idx] + B[1][idx.dec_x()] +
                               //  B0[1][idx] + B0[1][idx.dec_x()]);
                               (tmp_field[idx] + tmp_field[idx.dec_x()]);
            // if (pos[0] == 200 && pos[1] == 2) {
            //   printf("Eph1 is %f, Eph0 is %f, D0 is %f, B1 is %f\n", Eph1, Eph0,
            //          D[0][idx], tmp_field[idx]);
            // }

            B[0][idx] = B[0][idx] - prefactor * (Eph1 - Eph0);
          }
        }
      },
      B.get_ptrs(), B0.get_const_ptrs(), D.get_const_ptrs(),
      D0.get_const_ptrs(), m_tmp_prev_field.dev_ndptr_const(), m_a);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update_Dth(vector_field<Conf>& D,
                                        const vector_field<Conf>& D0,
                                        const vector_field<Conf>& B,
                                        const vector_field<Conf>& B0,
                                        const vector_field<Conf>& J,
                                        value_t dt) {
  m_tmp_rhs.assign_dev(0.0f);
  m_tmp_prev_field.copy_from(D[1]);

  // First assemble the right hand side and the diagonals of the tri-diagonal
  // equation
  kernel_launch(
      [dt] __device__(auto D, auto D0, auto B, auto B0, auto J, auto rhs,
                      auto d, auto dl, auto du, auto a) {
        using namespace Metric_KS;

        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          auto pos = get_pos(idx, ext);
          value_t r =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], true));
          value_t r_sp =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], false));
          value_t r_sm =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0] - 1, false));
          value_t dr = r_sp - r_sm;
          value_t th = grid.template pos<1>(pos[1], false);

          value_t sth = math::sin(th);
          value_t cth = math::cos(th);
          value_t prefactor = dt / (Metric_KS::sqrt_gamma(a, r, th) * dr);

          // if (grid.is_in_bound(pos)) {
          if (pos[0] > 0 && pos[0] < grid.dims[0] - 1) {
            auto Hph1 = ag_33(a, r_sp, sth, cth) * B[2][idx] +
                        ag_13(a, r_sp, sth, cth) * 0.5f *
                            (B[0][idx.inc_x()] + B[0][idx]) -
                        sq_gamma_beta(a, r_sp, sth, cth) * 0.5f *
                            (D[1][idx.inc_x()] + D[1][idx]);

            auto Hph0 = ag_33(a, r_sm, sth, cth) * B[2][idx.dec_x()] +
                        ag_13(a, r_sm, sth, cth) * 0.5f *
                            (B[0][idx] + B[0][idx.dec_x()]) -
                        sq_gamma_beta(a, r_sm, sth, cth) * 0.5f *
                            (D[1][idx] + D[1][idx.dec_x()]);

            rhs[idx] = D[1][idx] - dt * J[1][idx] + prefactor * (Hph0 - Hph1);
          }
          value_t du_coef =
              prefactor * 0.5f * Metric_KS::sq_gamma_beta(a, r_sp, th);
          value_t dl_coef =
              -prefactor * 0.5f * Metric_KS::sq_gamma_beta(a, r_sm, th);
          d[pos[0]] = 1.0f - (du_coef + dl_coef);

          du[pos[0]] = du_coef;
          dl[pos[0]] = dl_coef;

          // if (pos[0] == 6 && pos[1] == 300) {
          //   printf("d is %f, du is %f, dl is %f\n", d[pos[0]], du[pos[0]], dl[pos[0]]);
          // }
        }
      },
      D.get_const_ptrs(), D0.get_const_ptrs(), B.get_const_ptrs(),
      B0.get_const_ptrs(), J.get_const_ptrs(), m_tmp_rhs.dev_ndptr(),
      m_tri_d.dev_ptr(), m_tri_dl.dev_ptr(), m_tri_du.dev_ptr(), m_a);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();

  solve_tridiagonal();

  if (this->m_comm == nullptr || this->m_comm->domain_info().is_boundary[2]) {
    kernel_launch(
        [] __device__(auto f) {
          auto& grid = dev_grid<Conf::dim>();
          auto ext = grid.extent();
          for (auto n0 : grid_stride_range(0, grid.dims[0])) {
            int n1 = grid.guard[1];
            auto idx = Conf::idx({n0, n1}, ext);

            f[idx] = 0.0f;
          }
        },
        m_tmp_rhs.dev_ndptr());
  }

  D[1].copy_from(m_tmp_rhs);
  select_dev(m_tmp_prev_field) = m_tmp_rhs * 0.5f + m_tmp_prev_field * 0.5f;
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update_Dph(vector_field<Conf>& D,
                                        const vector_field<Conf>& D0,
                                        const vector_field<Conf>& B,
                                        const vector_field<Conf>& B0,
                                        const vector_field<Conf>& J,
                                        value_t dt) {
  m_tmp_rhs.assign_dev(0.0f);

  // First assemble the right hand side and the diagonals of the tri-diagonal
  // equation
  kernel_launch(
      [dt] __device__(auto D, auto B, auto J, auto rhs, auto d, auto dl,
                      auto du, auto a) {
        using namespace Metric_KS;

        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          auto pos = get_pos(idx, ext);
          value_t r =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], true));
          value_t r_sp =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], false));
          value_t r_sm =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0] - 1, false));
          value_t dr = r_sp - r_sm;

          value_t th = grid.template pos<1>(pos[1], true);
          value_t th_sp = grid.template pos<1>(pos[1], false);
          value_t th_sm = grid.template pos<1>(pos[1] - 1, false);
          value_t dth = th_sp - th_sm;
          if (th < TINY) th = 0.01f * grid.delta[1];

          value_t sth = math::sin(th);
          value_t cth = math::cos(th);
          value_t prefactor = dt / (Metric_KS::sqrt_gamma(a, r, th) * dr * dth);

          // if (grid.is_in_bound(pos)) {
          if (pos[1] > 0 && pos[0] > 0 && pos[0] < grid.dims[0] - 1) {
            auto Hr0 = ag_11(a, r, th_sm) * B[0][idx.dec_y()] +
                       ag_13(a, r, th_sm) * 0.5f *
                           (B[2][idx.dec_y()] + B[2][idx.dec_y().dec_x()]);

            auto Hr1 =
                ag_11(a, r, th_sp) * B[0][idx] +
                ag_13(a, r, th_sp) * 0.5f * (B[2][idx] + B[2][idx.dec_x()]);

            auto Hth0 = ag_22(a, r_sm, sth, cth) * B[1][idx.dec_x()] +
                        sq_gamma_beta(a, r_sm, sth, cth) * 0.5f *
                            (D[2][idx] + D[2][idx.dec_x()]);

            auto Hth1 = ag_22(a, r_sp, sth, cth) * B[1][idx] +
                        sq_gamma_beta(a, r_sp, sth, cth) * 0.5f *
                            (D[2][idx.inc_x()] + D[2][idx]);

            rhs[idx] = D[2][idx] - dt * J[2][idx] +
                       prefactor * (dr * (Hr0 - Hr1) + dth * (Hth1 - Hth0));
            // prefactor * (dr * (H_r<Conf>(B[0], B[2], idx.dec_y(1),
            //                              pos.dec_y(1), grid, a) -
            //                    H_r<Conf>(B[0], B[2], idx, pos, grid, a)) -
            //              dth * (H_th<Conf>(B[1], D[2], idx.dec_x(1),
            //                                pos.dec_x(1), grid, a) -
            //                     H_th<Conf>(B[1], D[2], idx, pos, grid, a)));
          }
          value_t du_coef =
              prefactor * dth * 0.5f * Metric_KS::sq_gamma_beta(a, r_sp, th);
          value_t dl_coef =
              -prefactor * dth * 0.5f * Metric_KS::sq_gamma_beta(a, r_sm, th);

          d[pos[0]] = 1.0f - (du_coef + dl_coef);
          du[pos[0]] = du_coef;
          dl[pos[0]] = dl_coef;

          // if (pos[0] == 6 && pos[1] == 300) {
          //   printf("d is %f, du is %f, dl is %f\n", d[pos[0]], du[pos[0]], dl[pos[0]]);
          // }
        }
      },
      D.get_const_ptrs(), B.get_const_ptrs(), J.get_const_ptrs(),
      m_tmp_rhs.dev_ndptr(), m_tri_d.dev_ptr(), m_tri_dl.dev_ptr(),
      m_tri_du.dev_ptr(), m_a);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();

  solve_tridiagonal();

  D[2].copy_from(m_tmp_rhs);
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update_Dr(vector_field<Conf>& D,
                                       const vector_field<Conf>& D0,
                                       const vector_field<Conf>& B,
                                       const vector_field<Conf>& B0,
                                       const vector_field<Conf>& J,
                                       value_t dt) {
  kernel_launch(
      [dt] __device__(auto D, auto B, auto J, auto tmp_field, auto a) {
        using namespace Metric_KS;

        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          auto pos = get_pos(idx, ext);
          if (grid.is_in_bound(pos)) {
            value_t r =
                grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], false));

            value_t th = grid.template pos<1>(pos[1], true);
            value_t th_sp = grid.template pos<1>(pos[1], false);
            value_t th_sm = grid.template pos<1>(pos[1] - 1, false);
            value_t dth = th_sp - th_sm;
            if (th < TINY) th = 1.0e-5;

            value_t prefactor = dt / (Metric_KS::sqrt_gamma(a, r, th) * dth);

            value_t sth = math::sin(th_sp);
            value_t cth = math::cos(th_sp);
            auto Hph1 =
                ag_33(a, r, sth, cth) * B[2][idx] +
                ag_13(a, r, sth, cth) * 0.5f * (B[0][idx] + B[0][idx.inc_x()]) -
                sq_gamma_beta(a, r, sth, cth) * 0.5f *
                    (tmp_field[idx] + tmp_field[idx.inc_x()]);

            sth = math::sin(th_sm);
            cth = math::cos(th_sm);
            auto Hph0 =
                ag_33(a, r, sth, cth) * B[2][idx.dec_y()] +
                ag_13(a, r, sth, cth) * 0.5f *
                    (B[0][idx.dec_y()] + B[0][idx.dec_y().inc_x()]) -
                sq_gamma_beta(a, r, sth, cth) * 0.5f *
                    (tmp_field[idx.dec_y()] + tmp_field[idx.dec_y().inc_x()]);

            D[0][idx] = D[0][idx] - dt * J[0][idx] + prefactor * (Hph1 - Hph0);

            if (D[0][idx] != D[0][idx]) {
                printf(
                    "NaN detected in Dr update! B2 is %f, B0 is %f, tmp_field is "
                    "%f\n",
                    B[2][idx.dec_y()], B[0][idx.dec_y()], tmp_field[idx.dec_y()]);
                asm("trap;");
            }
            // prefactor *
            //     (H_ph<Conf>(B[2], B[0], tmp_field, idx, pos, grid, a) -
            //      H_ph<Conf>(B[2], B[0], tmp_field, idx.dec_y(1),
            //                 pos.dec_y(1), grid, a));
          }
        }
      },
      D.get_ptrs(), B.get_const_ptrs(), J.get_const_ptrs(),
      m_tmp_prev_field.dev_ndptr_const(), m_a);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update(double dt, uint32_t step) {
  Logger::print_info("In GR KS solver! a is {}", m_a);

  if (this->m_update_b) {
    update_Bth(*(this->B), *(this->B0), *(this->E), *(this->E0), dt);
    update_Bph(*(this->B), *(this->B0), *(this->E), *(this->E0), dt);
    update_Br(*(this->B), *(this->B0), *(this->E), *(this->E0), dt);

    axis_boundary_b(*(this->B), m_ks_grid);
    // Communicate the new B values to guard cells
    if (this->m_comm != nullptr) this->m_comm->send_guard_cells(*(this->B));
  }

  if (this->m_update_e) {
    update_Dth(*(this->E), *(this->E0), *(this->B), *(this->B0), *(this->J),
               dt);
    update_Dph(*(this->E), *(this->E0), *(this->B), *(this->B0), *(this->J),
               dt);
    update_Dr(*(this->E), *(this->E0), *(this->B), *(this->B0), *(this->J), dt);
    axis_boundary_e(*(this->E), m_ks_grid);

    // Communicate the new E values to guard cells
    if (this->m_comm != nullptr) this->m_comm->send_guard_cells(*(this->E));
  }

  if (this->m_comm == nullptr || this->m_comm->domain_info().is_boundary[0]) {
    horizon_boundary(*(this->E), *(this->B), *(this->E0), *(this->B0), m_ks_grid);
  }

  this->Etotal->copy_from(*(this->E));
  // this->Etotal->add_by(*(this->E0));

  this->Btotal->copy_from(*(this->B));
  // this->Btotal->add_by(*(this->B0));

  if (step % this->m_data_interval == 0) {
    compute_flux(*flux, *(this->Btotal), m_ks_grid);
  }

  CudaSafeCall(hipDeviceSynchronize());
}

template class field_solver_gr_ks_cu<Config<2>>;

}  // namespace Aperture
