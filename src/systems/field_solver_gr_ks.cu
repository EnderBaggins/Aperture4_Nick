#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "core/cuda_control.h"
#include "core/multi_array_exp.hpp"
#include "core/ndsubset_dev.hpp"
#include "field_solver_gr_ks.h"
#include "framework/config.h"
#include "framework/environment.h"
#include "utils/kernel_helper.hpp"
#include "utils/timer.h"
// #include <hipsparse.h>

namespace Aperture {

namespace {

template <typename Conf>
void
axis_boundary_e(vector_field<Conf> &D, const grid_ks_t<Conf> &grid) {
  typedef typename Conf::idx_t idx_t;
  kernel_launch(
      [] __device__(auto D) {
        auto &grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto n0 : grid_stride_range(0, grid.dims[0])) {
          auto n1_0 = grid.guard[1];
          auto n1_pi = grid.dims[1] - grid.guard[1];
          if (abs(grid_ks_t<Conf>::theta(grid.template pos<1>(n1_0, true))) <
              0.1f * grid.delta[1]) {
            // At the theta = 0 axis

            // Set E_phi and B_theta to zero
            auto idx = idx_t(index_t<2>(n0, n1_0), ext);
            D[2][idx] = 0.0f;
            // e[1][idx] = 0.0;
            // D[1][idx.dec_y()] = D[1][idx];
            // D[0][idx.dec_y()] = D[0][idx];
            // D[0][idx] = 0.0f;
          }
          // printf("boundary pi at %f\n", grid.template pos<1>(n1_pi, true));
          if (abs(grid_ks_t<Conf>::theta(grid.template pos<1>(n1_pi, true)) -
                  M_PI) < 0.1f * grid.delta[1]) {
            // At the theta = pi axis
            auto idx = idx_t(index_t<2>(n0, n1_pi), ext);
            D[2][idx] = 0.0f;
            // e[1][idx] = 0.0;
            // D[1][idx] = D[1][idx.dec_y()];
            // D[0][idx] = D[0][idx.dec_y()];
            // D[0][idx] = 0.0f;
          }
        }
      },
      D.get_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
axis_boundary_b(vector_field<Conf> &B, const grid_ks_t<Conf> &grid) {
  typedef typename Conf::idx_t idx_t;
  kernel_launch(
      [] __device__(auto B) {
        auto &grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto n0 : grid_stride_range(0, grid.dims[0])) {
          for (int n1_0 = grid.guard[1]; n1_0 >= 0; n1_0--) {
            if (grid_ks_t<Conf>::theta(grid.template pos<1>(n1_0, true)) <
                0.1f * grid.delta[1]) {
              // At the theta = 0 axis

              // Set E_phi and B_theta to zero
              auto idx = idx_t(index_t<2>(n0, n1_0), ext);
              B[1][idx] = 0.0f;
              B[2][idx] = 0.0f;
              B[2][idx.dec_y()] = B[2][idx];
              // B[0][idx.dec_y()] = B[0][idx];
            }
          }
          for (int n1_pi = grid.dims[1] - grid.guard[1];
               n1_pi <= grid.dims[1] - 1; n1_pi++) {
            // printf("boundary pi at %f\n", grid.template pos<1>(n1_pi, true));
            if (abs(grid_ks_t<Conf>::theta(grid.template pos<1>(n1_pi, true)) -
                    M_PI) < 0.1f * grid.delta[1]) {
              // At the theta = pi axis
              auto idx = idx_t(index_t<2>(n0, n1_pi), ext);
              B[1][idx] = 0.0f;
              // B[1][idx.dec_y()] = 0.0f;
              // B[2][idx] = B[2][idx.dec_y()];
              B[2][idx] = 0.0f;
              B[2][idx.dec_y()] = 0.0f;
              // B[0][idx] = B[0][idx.dec_y()];
            }
          }
        }
      },
      B.get_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
horizon_boundary(vector_field<Conf> &D, vector_field<Conf> &B,
                 const vector_field<Conf> &D0, const vector_field<Conf> &B0,
                 const grid_ks_t<Conf> &grid, int damping_length,
                 float damping_coef) {
  using value_t = typename Conf::value_t;
  kernel_launch(
      [damping_length, damping_coef] __device__(auto D, auto D0, auto B,
                                                auto B0, auto grid_ptrs) {
        auto &grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto n1 : grid_stride_range(0, grid.dims[1])) {
          auto pos_ref = index_t<2>(damping_length, n1);
          auto idx_ref = Conf::idx(pos_ref, ext);
          for (int n0 = 0; n0 < damping_length; n0++) {
            auto pos = index_t<2>(n0, n1);
            auto idx = Conf::idx(pos, ext);
            value_t lambda =
                1.0f - damping_coef * cube((value_t)(damping_length - 1 - n0) /
                                           (damping_length - 1));

            // B[0][idx] *= lambda;
            // B[1][idx] *= lambda;
            // B[2][idx] *= lambda;
            // D[0][idx] *= lambda;
            // D[1][idx] *= lambda;
            // D[2][idx] *= lambda;

            // B[1][idx] = B[1][idx_ref];
            // B[2][idx] = B[2][idx_ref];
            D[0][idx] = D[0][idx_ref];
            B[0][idx] = B[0][idx_ref];

            // D[1][idx] = D[1][idx_ref];
            // D[2][idx] = D[2][idx_ref];

            // B[1][idx] = B0[1][idx];
            // B[2][idx] = B0[2][idx];
            // D[0][idx] = D0[0][idx];

            // B[0][idx] = B0[0][idx];
            // D[1][idx] = D0[1][idx];
            // D[2][idx] = D0[2][idx];

            B[1][idx] = 0.0f;
            B[2][idx] = 0.0f;
            // D[0][idx] = 0.0f;

            // B[0][idx] = 0.0f;
            D[1][idx] = 0.0f;
            D[2][idx] = 0.0f;
          }
        }
      },
      D.get_ptrs(), D0.get_ptrs(), B.get_ptrs(), B0.get_ptrs(),
      grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
inner_boundary(vector_field<Conf> &D, vector_field<Conf> &B,
               const grid_ks_t<Conf> &grid, int boundary_cell) {
  using value_t = typename Conf::value_t;
  using namespace Metric_KS;

  kernel_launch(
      [boundary_cell] __device__(auto D, auto B, auto grid_ptrs, auto a) {
        auto &grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto n1 : grid_stride_range(0, grid.dims[1])) {
          int n0 = boundary_cell;
          auto pos = index_t<2>(n0, n1);
          auto idx = Conf::idx(pos, ext);
          // Dr and Br are continuous
          B[0][idx.dec_x()] = B[0][idx];
          D[0][idx.dec_x()] = D[0][idx];

          value_t r =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], true));
          value_t r_p =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0] + 1, true));
          value_t th_s =
              grid_ks_t<Conf>::theta(grid.template pos<1>(pos[1], true));
          value_t th =
              grid_ks_t<Conf>::theta(grid.template pos<1>(pos[1], false));
          if (th_s < TINY) th_s = 0.01f * grid.delta[1];

          auto sth = math::sin(th_s);
          auto cth = math::cos(th_s);

          // First solve for Bth and Dph
          value_t coef_a = grid_ptrs.ag22dth_h[idx.dec_x()];
          value_t coef_b = grid_ptrs.gbetadth_h[idx.dec_x()] -
                           0.5f * grid_ptrs.gbetadth_h[idx];
          value_t coef_c = 0.5f * sq_gamma_beta(a, r, sth, cth);
          value_t coef_d = ag_33(a, r, sth, cth);

          value_t rhs_F = grid_ptrs.ag22dth_h[idx] * B[1][idx] +
                          0.5f * grid_ptrs.gbetadth_h[idx] * D[2][idx.inc_x()];
          value_t rhs_G =
              ag_33(a, r_p, sth, cth) * D[2][idx.inc_x()] +
              ag_13(a, r_p, sth, cth) * 0.5f * (D[0][idx] + D[0][idx.inc_x()]) +
              sq_gamma_beta(a, r_p, sth, cth) * 0.5f *
                  (B[1][idx] + B[1][idx.inc_x()]) -
              ag_13(a, r, sth, cth) * D[0][idx] -
              0.5f * sq_gamma_beta(a, r, sth, cth) * B[1][idx];

          value_t det = coef_b * coef_c - coef_a * coef_d;
          B[1][idx.dec_x()] = (coef_b * rhs_G - coef_d * rhs_F) / det;
          D[2][idx] = (coef_c * rhs_F - coef_a * rhs_G) / det;

          // value_t Hth1 = grid_ptrs.ag22dth_h[idx] * B[1][idx] +
          //       grid_ptrs.gbetadth_h[idx] * 0.5f *
          //           (D[2][idx.inc_x()] + D[2][idx]);
          // value_t Hth0 = grid_ptrs.ag22dth_h[idx.dec_x()] * B[1][idx.dec_x()]
          // +
          //       grid_ptrs.gbetadth_h[idx.dec_x()] * D[2][idx];

          // value_t Eph1 = ag_33(a, r_p, sth, cth) * D[2][idx.inc_x()] +
          //     ag_13(a, r_p, sth, cth) * 0.5f *
          //     (D[0][idx.inc_x()] + D[0][idx]) +
          //     0.5f * sq_gamma_beta(a, r_p, sth, cth) *
          //     (B[1][idx.inc_x()] + B[1][idx]);
          // value_t Eph0 =
          //     ag_33(a, r, sth, cth) * D[2][idx] +
          //     ag_13(a, r, sth, cth) * D[0][idx] +
          //     sq_gamma_beta(a, r, sth, cth) * 0.5f *
          //     (B[1][idx] + B[1][idx.dec_x()]);

          // if (pos[1] == 250) {
          //   printf("Hth1 is %f, Hth0 is %f, Eph1 is %f, Eph0 is %f\n", Hth1,
          //   Hth0, Eph1, Eph0);
          // }
          // if (Eph0 != Eph0 || Eph1 != Eph1) {
          //   printf("NaN in Eph at pos %d \n", pos[1]);
          //   asm("trap;");
          // }

          // Then solve for Bph and Dth
          r = grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], false));
          value_t r_m =
              grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0] - 1, false));
          sth = math::sin(th);
          cth = math::cos(th);

          coef_a = ag_33(a, r_m, sth, cth);
          coef_b = -sq_gamma_beta(a, r_m, sth, cth) +
                   0.5f * sq_gamma_beta(a, r, sth, cth);
          coef_c = -0.5 * grid_ptrs.gbetadth_e[idx];
          coef_d = grid_ptrs.ag22dth_e[idx];

          rhs_F =
              ag_33(a, r, sth, cth) * B[2][idx] +
              ag_13(a, r, sth, cth) * 0.5f * (B[0][idx] + B[0][idx.inc_x()]) -
              sq_gamma_beta(a, r, sth, cth) * 0.5f * D[1][idx.inc_x()] -
              ag_13(a, r_m, sth, cth) * B[0][idx];
          rhs_G = grid_ptrs.ag22dth_e[idx.inc_x()] * D[1][idx.inc_x()] -
                  grid_ptrs.gbetadth_e[idx.inc_x()] * 0.5f *
                      (B[2][idx] + B[2][idx.inc_x()]) +
                  grid_ptrs.gbetadth_e[idx] * 0.5f * B[2][idx];

          det = coef_b * coef_c - coef_a * coef_d;
          B[2][idx.dec_x()] = (coef_b * rhs_G - coef_d * rhs_F) / det;
          D[1][idx] = (coef_c * rhs_F - coef_a * rhs_G) / det;

          // value_t Eth1 =
          //     grid_ptrs.ag22dth_e[idx.inc_x()] * D[1][idx.inc_x()] -
          //     grid_ptrs.gbetadth_e[idx.inc_x()] * 0.5f *
          //     (B[2][idx.inc_x()] + B[2][idx]);
          // value_t Eth0 =
          //     grid_ptrs.ag22dth_e[idx] * D[1][idx] -
          //     grid_ptrs.gbetadth_e[idx] * 0.5f *
          //     (B[2][idx] + B[2][idx.dec_x()]);

          // value_t Hph0 = ag_33(a, r_m, sth, cth) * B[2][idx.dec_x()] +
          //     ag_13(a, r_m, sth, cth) * B[0][idx] -
          //     sq_gamma_beta(a, r_m, sth, cth) * D[1][idx];

          // value_t Hph1 = ag_33(a, r, sth, cth) * B[2][idx] +
          //     ag_13(a, r, sth, cth) * 0.5f *
          //     (B[0][idx.inc_x()] + B[0][idx]) -
          //     sq_gamma_beta(a, r, sth, cth) * 0.5f *
          //     (D[1][idx.inc_x()] + D[1][idx]);

          // if (pos[1] == 250) {
          //   printf("Eth1 is %f, Eth0 is %f, Hph1 is %f, Hph0 is %f\n", Eth1,
          //   Eth0, Hph1, Hph0);
          // }
          // for (int j = 0; j < boundary_cell; j++) {
          //   pos = get_pos(j, n1);
          //   idx = Conf::idx(pos, ext);

          //   if (j < boundary_cell - 1) {

          //   }
          // }
        }
      },
      D.get_ptrs(), B.get_ptrs(), grid.get_grid_ptrs(), grid.a);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
compute_flux(scalar_field<Conf> &flux, const vector_field<Conf> &b,
             const grid_ks_t<Conf> &grid) {
  flux.init();
  auto ext = grid.extent();
  kernel_launch(
      [ext] __device__(auto flux, auto b, auto a, auto grid_ptrs) {
        auto &grid = dev_grid<Conf::dim>();
        for (auto n0 : grid_stride_range(0, grid.dims[0])) {
          auto r = grid_ks_t<Conf>::radius(grid.template pos<0>(n0, true));

          for (int n1 = grid.guard[1]; n1 < grid.dims[1] - grid.guard[1];
               n1++) {
            Scalar th = grid_ks_t<Conf>::theta(grid.template pos<1>(n1, false));
            Scalar th_p =
                grid_ks_t<Conf>::theta(grid.template pos<1>(n1 + 1, true));
            Scalar th_m =
                grid_ks_t<Conf>::theta(grid.template pos<1>(n1, true));
            auto dth = th_p - th_m;

            auto pos = index_t<Conf::dim>(n0, n1);
            auto idx = typename Conf::idx_t(pos, ext);

            flux[idx] = flux[idx.dec_y()] +
                        // b[0][idx] * Metric_KS::sqrt_gamma(a, r, th) * dth;
                        b[0][idx] * grid_ptrs.Ab[0][idx];
          }
        }
      },
      flux.dev_ndptr(), b.get_ptrs(), grid.a, grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
compute_divs(scalar_field<Conf> &divD, scalar_field<Conf> &divB,
             const vector_field<Conf> &D, const vector_field<Conf> &B,
             const grid_ks_t<Conf> &grid) {
  kernel_launch(
      [] __device__(auto div_e, auto e, auto div_b, auto b, auto grid_ptrs) {
        auto &grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          auto pos = get_pos(idx, ext);
          if (grid.is_in_bound(pos)) {
            div_b[idx] = (b[0][idx.inc_x()] * grid_ptrs.Ab[0][idx.inc_x()] -
                          b[0][idx] * grid_ptrs.Ab[0][idx] +
                          b[1][idx.inc_y()] * grid_ptrs.Ab[1][idx.inc_y()] -
                          b[1][idx] * grid_ptrs.Ab[1][idx]) /
                         grid_ptrs.Ab[2][idx];

            div_e[idx] = (e[0][idx] * grid_ptrs.Ad[0][idx] -
                          e[0][idx.dec_x()] * grid_ptrs.Ad[0][idx.dec_x()] +
                          e[1][idx] * grid_ptrs.Ad[1][idx] -
                          e[1][idx.dec_y()] * grid_ptrs.Ad[1][idx.dec_y()]) /
                         grid_ptrs.Ad[2][idx];

            if (pos[0] == 3 && pos[1] == 200) {
              printf(
                  "divD is %f, eA0_p is %f, eA0_m is %f, eA1_p is %f, eA1_m is "
                  "%f\n",
                  div_e[idx] * grid_ptrs.Ad[2][idx],
                  e[0][idx] * grid_ptrs.Ad[0][idx],
                  e[0][idx.dec_x()] * grid_ptrs.Ad[0][idx.dec_x()],
                  e[1][idx] * grid_ptrs.Ad[1][idx],
                  e[1][idx.dec_y()] * grid_ptrs.Ad[1][idx.dec_y()]);
            }
          }
        }
      },
      divD[0].dev_ndptr(), D.get_const_ptrs(), divB[0].dev_ndptr(),
      B.get_const_ptrs(), grid.get_grid_ptrs());
}

}  // namespace

template <typename Conf>
field_solver_gr_ks_cu<Conf>::~field_solver_gr_ks_cu() {
  // sp_buffer.resize(0);
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::init() {
  field_solver<Conf>::init();

  this->m_env.params().get_value("bh_spin", m_a);
  Logger::print_info("bh_spin in field solver is {}", m_a);
  this->m_env.params().get_value("implicit_beta", this->m_beta);
  this->m_env.params().get_value("damping_length", m_damping_length);
  this->m_env.params().get_value("damping_coef", m_damping_coef);

  m_tmp_th_field.set_memtype(MemType::device_only);
  m_tmp_th_field.resize(this->m_grid.extent());
  m_tmp_prev_field.set_memtype(MemType::device_only);
  m_tmp_prev_field.resize(this->m_grid.extent());
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::register_data_components() {
  field_solver_cu<Conf>::register_data_components();

  flux = this->m_env.template register_data<scalar_field<Conf>>(
      "flux", this->m_grid, field_type::vert_centered);
}

// template <typename Conf>
// void
// field_solver_gr_ks_cu<Conf>::solve_tridiagonal() {
//   // Solve the assembled tri-diagonal system using cusparse
//   hipsparseStatus_t status;
//   auto ext = this->m_grid.extent_less();
// #if USE_DOUBLE
//   status = hipsparseDgtsv2(sp_handle, ext[0], ext[1], m_tri_dl.dev_ptr(),
//                           m_tri_d.dev_ptr(), m_tri_du.dev_ptr(),
//                           m_tmp_rhs.dev_ptr(), ext[0], sp_buffer.dev_ptr());
// #else
//   status = hipsparseSgtsv2(sp_handle, ext[0], ext[1], m_tri_dl.dev_ptr(),
//                           m_tri_d.dev_ptr(), m_tri_du.dev_ptr(),
//                           m_tmp_rhs.dev_ptr(), ext[0], sp_buffer.dev_ptr());
// #endif
//   CudaSafeCall(hipDeviceSynchronize());
//   if (status != HIPSPARSE_STATUS_SUCCESS) {
//     Logger::print_err("cusparse failure during field update! Error code {}",
//                       status);
//   }
// }

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update_Bth(vector_field<Conf> &B,
                                        const vector_field<Conf> &B0,
                                        const vector_field<Conf> &D,
                                        const vector_field<Conf> &D0,
                                        value_t dt) {
  m_tmp_prev_field.copy_from(B[1]);

  // Predictor-corrector approach to update Bth
  auto Bth_kernel = [dt] __device__(auto B, auto B0, auto D, auto D0,
                                    auto B1_0, auto B1_1, auto a, auto grid_ptrs) {
    using namespace Metric_KS;

    auto &grid = dev_grid<Conf::dim>();
    auto ext = grid.extent();
    auto extl = grid.extent_less();
    for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
      auto pos = get_pos(idx, ext);
      if (grid.is_in_bound(pos)) {
        value_t r =
            grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], false));
        value_t r_sp =
            grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0] + 1, true));
        value_t r_sm =
            grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], true));

        value_t th = grid.template pos<1>(pos[1], true);
        if (math::abs(th) < TINY) th = 0.01f * grid.delta[1];

        value_t sth = math::sin(th);
        value_t cth = math::cos(th);
        value_t prefactor = dt / grid_ptrs.Ab[1][idx];

        auto Eph1 =
            ag_33(a, r_sp, sth, cth) * D[2][idx.inc_x()] +
            ag_13(a, r_sp, sth, cth) * 0.5f * (D[0][idx.inc_x()] + D[0][idx]) +
            0.25f * sq_gamma_beta(a, r_sp, sth, cth) *
                (B1_0[idx.inc_x()] + B1_0[idx] + B1_1[idx.inc_x()] + B1_1[idx]);

        auto Eph0 =
            ag_33(a, r_sm, sth, cth) * D[2][idx] +
            ag_13(a, r_sm, sth, cth) * 0.5f * (D[0][idx] + D[0][idx.dec_x()]) +
            0.25f * sq_gamma_beta(a, r_sm, sth, cth) *
                (B1_0[idx] + B1_0[idx.dec_x()] + B1_1[idx] + B1_1[idx.dec_x()]);

        B[1][idx] = B1_0[idx] - prefactor * (Eph0 - Eph1);

        // if (pos[0] == 6 && pos[1] == 200) {
        // printf(
        //     "Eph1 is %f, Eph0 is %f, dEphi is %f, B1 is %f, "
        //     "rhs is %f\n",
        //     Eph1, Eph0, (Eph0 - Eph1), B[1][idx],
        //     rhs[idx]);
        // printf("ag33 is %f, ag13 is %f, sqgb is %f\n",
        //        ag_33(a, r, sth, cth), ag_13(a, r, sth, cth),
        //        sq_gamma_beta(a, r, sth, cth));
        // }
        // if (pos[1] == 2 && pos[0] == 200)
        //   printf("rhs is %f, D0 is %f, B1 is %f\n", rhs[idx], D[0][idx],
        //   B[1][idx]);
      }
    }
  };
  kernel_launch(Bth_kernel, B.get_ptrs(), B0.get_const_ptrs(),
                D.get_const_ptrs(), D0.get_const_ptrs(), B[1].dev_ndptr_const(),
                B[1].dev_ndptr_const(), m_a, m_ks_grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  kernel_launch(Bth_kernel, B.get_ptrs(), B0.get_const_ptrs(),
                D.get_const_ptrs(), D0.get_const_ptrs(),
                m_tmp_prev_field.dev_ndptr_const(), B[1].dev_ndptr_const(),
                m_a, m_ks_grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();

  select_dev(m_tmp_th_field) = B[1] * 0.5f + m_tmp_prev_field * 0.5f;
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update_Bph(vector_field<Conf> &B,
                                        const vector_field<Conf> &B0,
                                        const vector_field<Conf> &D,
                                        const vector_field<Conf> &D0,
                                        value_t dt) {
  m_tmp_prev_field.copy_from(B[2]);

  // Use a predictor-corrector step to update Bph too
  auto Bph_kernel = [dt] __device__(auto B, auto B0, auto D, auto D0, auto B2_0,
                                    auto B2_1, auto a, auto grid_ptrs) {
    using namespace Metric_KS;
    auto &grid = dev_grid<Conf::dim>();
    auto ext = grid.extent();
    auto extl = grid.extent_less();
    for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
      auto pos = get_pos(idx, ext);
      if (grid.is_in_bound(pos)) {
        value_t r =
            grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], false));
        value_t r_sp =
            grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0] + 1, true));
        value_t r_sm =
            grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], true));
        // value_t dr = r_sp - r_sm;

        value_t th = grid.template pos<1>(pos[1], false);
        value_t th_sp = grid.template pos<1>(pos[1] + 1, true);
        value_t th_sm = grid.template pos<1>(pos[1], true);
        // value_t dth = th_sp - th_sm;
        if (th_sm < TINY) th_sm = 0.01f * grid.delta[1];

        value_t sth = math::sin(th);
        value_t cth = math::cos(th);
        value_t prefactor = dt / grid_ptrs.Ab[2][idx];

        auto Er1 = grid_ptrs.ag11dr_e[idx.inc_y()] * D[0][idx.inc_y()] +
                   grid_ptrs.ag13dr_e[idx.inc_y()] * 0.5f *
                       (D[2][idx.inc_y()] + D[2][idx.inc_y().inc_x()]);

        auto Er0 =
            grid_ptrs.ag11dr_e[idx] * D[0][idx] +
            grid_ptrs.ag13dr_e[idx] * 0.5f * (D[2][idx] + D[2][idx.inc_x()]);

        auto Eth1 =
            grid_ptrs.ag22dth_e[idx.inc_x()] * D[1][idx.inc_x()] -
            grid_ptrs.gbetadth_e[idx.inc_x()] * 0.25f *
                (B2_0[idx.inc_x()] + B2_0[idx] + B2_1[idx.inc_x()] + B2_1[idx]);

        auto Eth0 =
            grid_ptrs.ag22dth_e[idx] * D[1][idx] -
            grid_ptrs.gbetadth_e[idx] * 0.25f *
                (B2_0[idx] + B2_0[idx.dec_x()] + B2_1[idx] + B2_1[idx.dec_x()]);

        B[2][idx] = B2_0[idx] - prefactor * ((Er0 - Er1) + (Eth1 - Eth0));
      }
    }
  };
  kernel_launch(Bph_kernel, B.get_ptrs(), B0.get_const_ptrs(),
                D.get_const_ptrs(), D0.get_const_ptrs(), B[2].dev_ndptr_const(),
                B[2].dev_ndptr_const(), m_a, m_ks_grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  kernel_launch(Bph_kernel, B.get_ptrs(), B0.get_const_ptrs(),
                D.get_const_ptrs(), D0.get_const_ptrs(),
                m_tmp_prev_field.dev_ndptr_const(), B[2].dev_ndptr_const(), m_a,
                m_ks_grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update_Br(vector_field<Conf> &B,
                                       const vector_field<Conf> &B0,
                                       const vector_field<Conf> &D,
                                       const vector_field<Conf> &D0,
                                       value_t dt) {
  kernel_launch(
      [dt] __device__(auto B, auto B0, auto D, auto D0, auto tmp_field, auto a,
                      auto grid_ptrs) {
        using namespace Metric_KS;
        auto &grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          auto pos = get_pos(idx, ext);
          if (grid.is_in_bound(pos)) {
            value_t r =
                grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], true));
            value_t r_sp =
                grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], false));
            value_t r_sm = grid_ks_t<Conf>::radius(
                grid.template pos<0>(pos[0] - 1, false));

            value_t th = grid.template pos<1>(pos[1], false);
            value_t th_sp = grid.template pos<1>(pos[1] + 1, true);
            value_t th_sm = grid.template pos<1>(pos[1], true);
            if (th_sm < TINY) th_sm = 0.01f * grid.delta[1];

            value_t prefactor = dt / grid_ptrs.Ab[0][idx];

            value_t sth = math::sin(th_sp);
            value_t cth = math::cos(th_sp);
            value_t Eph1 =
                ag_33(a, r, sth, cth) * D[2][idx.inc_y()] +
                ag_13(a, r, sth, cth) * 0.5f *
                    (D[0][idx.inc_y()] + D[0][idx.inc_y().dec_x()]) +
                sq_gamma_beta(a, r, sth, cth) * 0.5f *
                    (tmp_field[idx.inc_y()] + tmp_field[idx.inc_y().dec_x()]);

            sth = math::sin(th_sm);
            cth = math::cos(th_sm);
            value_t Eph0 =
                ag_33(a, r, sth, cth) * D[2][idx] +
                ag_13(a, r, sth, cth) * 0.5f * (D[0][idx] + D[0][idx.dec_x()]) +
                sq_gamma_beta(a, r, sth, cth) * 0.5f *
                    (tmp_field[idx] + tmp_field[idx.dec_x()]);

            // if (pos[0] == 6 && pos[1] == 200) {
            //   printf(
            //       "Eph1 is %f, Eph0 is %f, dEphi is %f, tmpf is %f, "
            //       "tmpf+ is %f\n",
            //       Eph1, Eph0, Eph1 - Eph0, tmp_field[idx],
            //       tmp_field[idx.inc_y()]);
            //   printf("ag33 is %f, ag13 is %f, sqgb is %f\n",
            //          ag_33(a, r, sth, cth), ag_13(a, r, sth, cth),
            //          sq_gamma_beta(a, r, sth, cth));
            // }

            B[0][idx] += -prefactor * (Eph1 - Eph0);
          }
        }
      },
      B.get_ptrs(), B0.get_const_ptrs(), D.get_const_ptrs(),
      D0.get_const_ptrs(), m_tmp_th_field.dev_ndptr_const(), m_a,
      m_ks_grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update_Dth(vector_field<Conf> &D,
                                        const vector_field<Conf> &D0,
                                        const vector_field<Conf> &B,
                                        const vector_field<Conf> &B0,
                                        const vector_field<Conf> &J,
                                        value_t dt) {
  m_tmp_prev_field.copy_from(D[1]);

  // Predictor-corrector approach to update Dth
  auto Dth_kernel = [dt] __device__(auto D, auto D0, auto B, auto B0, auto J,
                                    auto D1_0, auto D1_1, auto a,
                                    auto grid_ptrs) {
    using namespace Metric_KS;

    auto &grid = dev_grid<Conf::dim>();
    auto ext = grid.extent();
    auto extl = grid.extent_less();
    for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
      auto pos = get_pos(idx, ext);
      if (grid.is_in_bound(pos)) {
        value_t r = grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], true));
        value_t r_sp =
            grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], false));
        value_t r_sm =
            grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0] - 1, false));
        value_t dr = r_sp - r_sm;

        value_t th = grid.template pos<1>(pos[1], false);

        value_t sth = math::sin(th);
        value_t cth = math::cos(th);
        value_t prefactor = dt / grid_ptrs.Ad[1][idx];

        auto Hph1 =
            ag_33(a, r_sp, sth, cth) * B[2][idx] +
            ag_13(a, r_sp, sth, cth) * 0.5f * (B[0][idx.inc_x()] + B[0][idx]) -
            sq_gamma_beta(a, r_sp, sth, cth) * 0.25f *
                (D1_0[idx.inc_x()] + D1_0[idx] + D1_1[idx.inc_x()] + D1_1[idx]);

        auto Hph0 =
            ag_33(a, r_sm, sth, cth) * B[2][idx.dec_x()] +
            ag_13(a, r_sm, sth, cth) * 0.5f * (B[0][idx] + B[0][idx.dec_x()]) -
            sq_gamma_beta(a, r_sm, sth, cth) * 0.25f *
                (D1_0[idx] + D1_0[idx.dec_x()] + D1_1[idx] + D1_1[idx.dec_x()]);

        // TODO: Fix boundary node problenm!
        if (pos[0] == grid.guard[0]) {
          Hph0 = Hph1;
        }

        D[1][idx] = D1_0[idx] + prefactor * (Hph0 - Hph1) - dt * J[1][idx];
      }
    }
  };
  kernel_launch(Dth_kernel, D.get_ptrs(), D0.get_const_ptrs(),
                B.get_const_ptrs(), B0.get_const_ptrs(), J.get_const_ptrs(),
                D[1].dev_ndptr_const(), D[1].dev_ndptr_const(), m_a,
                m_ks_grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  kernel_launch(Dth_kernel, D.get_ptrs(), D0.get_const_ptrs(),
                B.get_const_ptrs(), B0.get_const_ptrs(), J.get_const_ptrs(),
                m_tmp_prev_field.dev_ndptr_const(), D[1].dev_ndptr_const(), m_a,
                m_ks_grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();

  select_dev(m_tmp_th_field) = D[1] * 0.5f + m_tmp_prev_field * 0.5f;
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update_Dph(vector_field<Conf> &D,
                                        const vector_field<Conf> &D0,
                                        const vector_field<Conf> &B,
                                        const vector_field<Conf> &B0,
                                        const vector_field<Conf> &J,
                                        value_t dt) {
  m_tmp_prev_field.copy_from(D[2]);

  // First assemble the right hand side and the diagonals of the tri-diagonal
  // equation
  auto Dph_kernel = [dt] __device__(auto D, auto B, auto J, auto D2_0,
                                    auto D2_1, auto a, auto grid_ptrs) {
    using namespace Metric_KS;

    auto &grid = dev_grid<Conf::dim>();
    auto ext = grid.extent();
    auto extl = grid.extent_less();
    for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
      auto pos = get_pos(idx, ext);
      if (grid.is_in_bound(pos)) {
        value_t r = grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], true));
        value_t r_sp =
            grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], false));
        value_t r_sm =
            grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0] - 1, false));
        value_t dr = r_sp - r_sm;

        value_t th = grid.template pos<1>(pos[1], true);
        value_t th_sp = grid.template pos<1>(pos[1], false);
        value_t th_sm = grid.template pos<1>(pos[1] - 1, false);
        if (th < TINY) th = 0.01f * grid.delta[1];

        value_t sth = math::sin(th);
        value_t cth = math::cos(th);
        value_t prefactor = dt / grid_ptrs.Ad[2][idx];

        auto Hr0 = grid_ptrs.ag11dr_h[idx.dec_y()] * B[0][idx.dec_y()] +
                   grid_ptrs.ag13dr_h[idx.dec_y()] * 0.5f *
                       (B[2][idx.dec_y()] + B[2][idx.dec_y().dec_x()]);

        auto Hr1 =
            grid_ptrs.ag11dr_h[idx] * B[0][idx] +
            grid_ptrs.ag13dr_h[idx] * 0.5f * (B[2][idx] + B[2][idx.dec_x()]);

        auto Hth0 =
            grid_ptrs.ag22dth_h[idx.dec_x()] * B[1][idx.dec_x()] +
            grid_ptrs.gbetadth_h[idx.dec_x()] * 0.25f *
                (D2_0[idx] + D2_0[idx.dec_x()] + D2_1[idx] + D2_1[idx.dec_x()]);

        auto Hth1 =
            grid_ptrs.ag22dth_h[idx] * B[1][idx] +
            grid_ptrs.gbetadth_h[idx] * 0.25f *
                (D2_0[idx.inc_x()] + D2_0[idx] + D2_1[idx.inc_x()] + D2_1[idx]);

        // TODO: Fix boundary node problenm!
        if (pos[0] == grid.guard[0]) {
          Hth0 = Hth1;
        }

        D[2][idx] = D2_0[idx] + prefactor * ((Hr0 - Hr1) + (Hth1 - Hth0)) - dt * J[2][idx];

        if (pos[0] == 10 && pos[1] == 250) {
          printf("Hr0 is %f, Hr1 is %f, Hth0 is %f, Hth1 is %f, dDphi is %f\n",
                 Hr0, Hr1, Hth0, Hth1,
                 prefactor * ((Hr0 - Hr1) + (Hth1 - Hth0)));
        }
      }
    }
  };
  kernel_launch(Dph_kernel, D.get_ptrs(), B.get_const_ptrs(),
                J.get_const_ptrs(), D[2].dev_ndptr_const(),
                D[2].dev_ndptr_const(), m_a, m_ks_grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  kernel_launch(Dph_kernel, D.get_ptrs(), B.get_const_ptrs(),
                J.get_const_ptrs(), m_tmp_prev_field.dev_ndptr_const(),
                D[2].dev_ndptr_const(), m_a,
                m_ks_grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update_Dr(vector_field<Conf> &D,
                                       const vector_field<Conf> &D0,
                                       const vector_field<Conf> &B,
                                       const vector_field<Conf> &B0,
                                       const vector_field<Conf> &J,
                                       value_t dt) {
  kernel_launch(
      [dt] __device__(auto D, auto B, auto J, auto tmp_field, auto a,
                      auto grid_ptrs) {
        using namespace Metric_KS;

        auto &grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          auto pos = get_pos(idx, ext);
          if (grid.is_in_bound(pos)) {
            value_t r =
                grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], false));
            value_t r_sp =
                grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0] + 1, true));
            value_t r_sm =
                grid_ks_t<Conf>::radius(grid.template pos<0>(pos[0], true));

            value_t th = grid.template pos<1>(pos[1], true);
            value_t th_sp = grid.template pos<1>(pos[1], false);
            value_t th_sm = grid.template pos<1>(pos[1] - 1, false);
            // bool is_axis = false;
            if (th < TINY) {
              th = 1.0e-5;
              // is_axis = true;
            }

            value_t prefactor = dt / grid_ptrs.Ad[0][idx];

            value_t sth = math::sin(th_sp);
            value_t cth = math::cos(th_sp);
            auto Hph1 =
                ag_33(a, r, sth, cth) * B[2][idx] +
                ag_13(a, r, sth, cth) * 0.5f * (B[0][idx] + B[0][idx.inc_x()]) -
                sq_gamma_beta(a, r, sth, cth) * 0.5f *
                    (tmp_field[idx] + tmp_field[idx.inc_x()]);

            sth = math::sin(th_sm);
            cth = math::cos(th_sm);
            auto Hph0 =
                ag_33(a, r, sth, cth) * B[2][idx.dec_y()] +
                ag_13(a, r, sth, cth) * 0.5f *
                    (B[0][idx.dec_y()] + B[0][idx.dec_y().inc_x()]) -
                sq_gamma_beta(a, r, sth, cth) * 0.5f *
                    (tmp_field[idx.dec_y()] + tmp_field[idx.dec_y().inc_x()]);

            // if (is_axis) {
            //   Hph0 = -Hph1;
            // }

            D[0][idx] += prefactor * (Hph1 - Hph0) - dt * J[0][idx];

            if (D[0][idx] != D[0][idx]) {
              printf(
                  "NaN detected in Dr update! B2 is %f, B0 is %f, tmp_field is "
                  "%f\n",
                  B[2][idx.dec_y()], B[0][idx.dec_y()], tmp_field[idx.dec_y()]);
              asm("trap;");
            }
          }
        }
      },
      D.get_ptrs(), B.get_const_ptrs(), J.get_const_ptrs(),
      m_tmp_th_field.dev_ndptr_const(), m_a, m_ks_grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
field_solver_gr_ks_cu<Conf>::update(double dt, uint32_t step) {
  Logger::print_info("In GR KS solver! a is {}", m_a);

  if (this->m_update_b) {
    update_Bph(*(this->B), *(this->B0), *(this->E), *(this->E0), dt);
    update_Bth(*(this->B), *(this->B0), *(this->E), *(this->E0), dt);
    update_Br(*(this->B), *(this->B0), *(this->E), *(this->E0), dt);

    axis_boundary_b(*(this->B), m_ks_grid);
    // Communicate the new B values to guard cells
    if (this->m_comm != nullptr) this->m_comm->send_guard_cells(*(this->B));
  }

  if (this->m_update_e) {
    update_Dph(*(this->E), *(this->E0), *(this->B), *(this->B0), *(this->J),
               dt);
    update_Dth(*(this->E), *(this->E0), *(this->B), *(this->B0), *(this->J),
               dt);
    update_Dr(*(this->E), *(this->E0), *(this->B), *(this->B0), *(this->J), dt);

    axis_boundary_e(*(this->E), m_ks_grid);
    // Communicate the new E values to guard cells
    if (this->m_comm != nullptr) this->m_comm->send_guard_cells(*(this->E));
  }

  if (this->m_comm == nullptr || this->m_comm->domain_info().is_boundary[0]) {
    // horizon_boundary(*(this->E), *(this->B), *(this->E0), *(this->B0),
    //                  m_ks_grid, m_damping_length, m_damping_coef);
    // inner_boundary(*(this->E), *(this->B), m_ks_grid, m_damping_length);
  }

  compute_divs(*(this->divE), *(this->divB), *(this->E), *(this->B), m_ks_grid);

  this->Etotal->copy_from(*(this->E));
  // this->Etotal->add_by(*(this->E0));

  this->Btotal->copy_from(*(this->B));
  // this->Btotal->add_by(*(this->B0));

  if (step % this->m_data_interval == 0) {
    compute_flux(*flux, *(this->Btotal), m_ks_grid);
  }

  CudaSafeCall(hipDeviceSynchronize());
}

template class field_solver_gr_ks_cu<Config<2>>;

}  // namespace Aperture
