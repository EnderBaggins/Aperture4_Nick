#include "hip/hip_runtime.h"
#include "core/constant_mem_func.h"
#include "core/detail/multi_array_helpers.h"
#include "framework/config.h"
#include "helpers/ptc_update_helper.hpp"
#include "ptc_updater.h"
#include "utils/double_buffer.h"
#include "utils/interpolation.hpp"
#include "utils/kernel_helper.hpp"
#include "utils/range.hpp"
#include "utils/util_functions.h"

namespace Aperture {

template <typename Conf>
void
ptc_updater_cu<Conf>::init() {
  this->init_charge_mass();
  init_dev_charge_mass(this->m_charges, this->m_masses);

  m_rho_ptrs.set_memtype(MemType::host_device);
  m_rho_ptrs.resize(this->m_num_species);
  for (int i = 0; i < this->m_num_species; i++) {
    m_rho_ptrs[i] = this->Rho[i]->get_ptr();
  }
  m_rho_ptrs.copy_to_device();

  // Allocate the tmp array for current filtering
  this->jtmp = std::make_unique<typename Conf::multi_array_t>
      (this->m_grid.extent(), MemType::host_device);
}

template <typename Conf>
void
ptc_updater_cu<Conf>::register_dependencies() {
  size_t max_ptc_num = 1000000;
  this->m_env.params().get_value("max_ptc_num", max_ptc_num);
  // Prefer device_only, but can take other possibilities if data is already
  // there
  this->ptc = this->m_env.template register_data<particle_data_t>(
      "particles", max_ptc_num, MemType::device_only);

  this->E = this->m_env.template register_data<vector_field<Conf>>(
      "E", this->m_grid, field_type::edge_centered, MemType::host_device);
  this->B = this->m_env.template register_data<vector_field<Conf>>(
      "B", this->m_grid, field_type::face_centered, MemType::host_device);
  this->J = this->m_env.template register_data<vector_field<Conf>>(
      "J", this->m_grid, field_type::edge_centered, MemType::host_device);

  this->m_env.params().get_value("num_species", this->m_num_species);
  this->Rho.resize(this->m_num_species);
  for (int i = 0; i < this->m_num_species; i++) {
    this->Rho[i] = this->m_env.template register_data<scalar_field<Conf>>(
        std::string("Rho_") + ptc_type_name(i), this->m_grid,
        field_type::vert_centered, MemType::host_device);
  }

  int rand_seed = 1234;
  this->m_env.params().get_value("rand_seed", rand_seed);
  m_rand_states = this->m_env.template register_data<curand_states_t>(
      "rand_states", size_t(512 * 1024), rand_seed);
}

// template <typename Conf>
// void
// ptc_updater_cu<Conf>::update(double dt, uint32_t step) {
//   if (this->m_pusher == Pusher::boris) {
//     push<boris_pusher>(dt, true);
//   } else if (this->m_pusher == Pusher::vay) {
//     push<vay_pusher>(dt, true);
//   } else if (this->m_pusher == Pusher::higuera) {
//     push<higuera_pusher>(dt, true);
//   }

// }

template <typename Conf>
void
ptc_updater_cu<Conf>::push_default(double dt) {
  // dispatch according to enum
  if (this->m_pusher == Pusher::boris) {
    push<boris_pusher>(dt);
  } else if (this->m_pusher == Pusher::vay) {
    push<vay_pusher>(dt);
  } else if (this->m_pusher == Pusher::higuera) {
    push<higuera_pusher>(dt);
  }
}

template <typename Conf>
template <typename P>
void
ptc_updater_cu<Conf>::push(double dt) {
  auto num = this->ptc->number();
  auto ext = this->m_grid.extent();
  P pusher;

  auto pusher_kernel = [dt, num, ext] __device__(auto ptrs, auto E, auto B,
                                                 auto pusher) {
    for (auto n : grid_stride_range(0, num)) {
      uint32_t cell = ptrs.cell[n];
      if (cell == empty_cell) continue;
      auto idx = E[0].idx_at(cell, ext);
      // auto pos = idx.get_pos();

      auto interp = interpolator<bspline<1>, Conf::dim>{};
      auto flag = ptrs.flag[n];
      int sp = get_ptc_type(flag);

      Scalar qdt_over_2m = dt * 0.5f * dev_charges[sp] / dev_masses[sp];

      auto x = vec_t<Pos_t, 3>(ptrs.x1[n], ptrs.x2[n], ptrs.x3[n]);
      //  Grab E & M fields at the particle position
      Scalar E1 = interp(E[0], x, idx, stagger_t(0b110));
      Scalar E2 = interp(E[1], x, idx, stagger_t(0b101));
      Scalar E3 = interp(E[2], x, idx, stagger_t(0b011));
      Scalar B1 = interp(B[0], x, idx, stagger_t(0b001));
      Scalar B2 = interp(B[1], x, idx, stagger_t(0b010));
      Scalar B3 = interp(B[2], x, idx, stagger_t(0b100));

      //  Push particles
      Scalar p1 = ptrs.p1[n], p2 = ptrs.p2[n], p3 = ptrs.p3[n],
             gamma = ptrs.E[n];
      if (p1 != p1 || p2 != p2 || p3 != p3) {
        printf(
            "NaN detected in push! p1 is %f, p2 is %f, p3 is %f, gamma "
            "is %f\n",
            p1, p2, p3, gamma);
        asm("trap;");
      }

      if (!check_flag(flag, PtcFlag::ignore_EM)) {
        pusher(p1, p2, p3, gamma, E1, E2, E3, B1, B2, B3, qdt_over_2m,
               (Scalar)dt);
      }

      // if (dev_params.rad_cooling_on && sp != (int)ParticleType::ion) {
      //   sync_kill_perp(p1, p2, p3, gamma, B1, B2, B3, E1, E2, E3,
      //                  q_over_m);
      // }
      ptrs.p1[n] = p1;
      ptrs.p2[n] = p2;
      ptrs.p3[n] = p3;
      ptrs.E[n] = gamma;
    }
  };

  if (num > 0) {
    // exec_policy p;
    // configure_grid(p, pusher_kernel, this->ptc->dev_ptrs(), this->E->get_ptrs(),
    //               this->B->get_ptrs(), pusher);
    // Logger::print_info(
    //     "pusher kernel: block_size: {}, grid_size: {}, shared_mem: {}",
    //     p.get_block_size(), p.get_grid_size(), p.get_shared_mem_bytes());

    kernel_launch(pusher_kernel, this->ptc->dev_ptrs(), this->E->get_ptrs(),
                  this->B->get_ptrs(), pusher);
  }
}

template <typename Conf>
void
ptc_updater_cu<Conf>::move_deposit_1d(double dt, uint32_t step) {
  auto num = this->ptc->number();
  if (num > 0) {
    auto ext = this->m_grid.extent();

    kernel_launch(
        [ext, num, dt, step] __device__(auto ptc, auto J, auto Rho,
                                        auto data_interval) {
          using spline_t = typename base_class::spline_t;
          for (auto n : grid_stride_range(0, num)) {
            uint32_t cell = ptc.cell[n];
            if (cell == empty_cell) continue;

            auto idx = J[0].idx_at(cell, ext);
            auto pos = idx.get_pos();

            // step 1: Move particles
            auto x1 = ptc.x1[n], x2 = ptc.x2[n], x3 = ptc.x3[n];
            Scalar v1 = ptc.p1[n], v2 = ptc.p2[n], v3 = ptc.p3[n],
                   gamma = ptc.E[n];

            v1 /= gamma;
            v2 /= gamma;
            v3 /= gamma;

            auto new_x1 = x1 + (v1 * dt) * dev_grid_1d.inv_delta[0];
            int dc1 = std::floor(new_x1);
            pos[0] += dc1;
            ptc.x1[n] = new_x1 - (Pos_t)dc1;
            ptc.x2[n] = x2 + v2 * dt;
            ptc.x3[n] = x3 + v3 * dt;

            ptc.cell[n] = J[0].get_idx(pos, ext).linear;

            // step 2: Deposit current
            auto flag = ptc.flag[n];
            auto sp = get_ptc_type(flag);
            auto interp = spline_t{};
            if (check_flag(flag, PtcFlag::ignore_current)) continue;
            auto weight = dev_charges[sp] * ptc.weight[n];

            int i_0 = (dc1 == -1 ? -spline_t::radius : 1 - spline_t::radius);
            int i_1 = (dc1 == 1 ? spline_t::radius + 1 : spline_t::radius);
            Scalar djx = 0.0f;
            for (int i = i_0; i <= i_1; i++) {
              Scalar sx0 = interp(-x1 + i);
              Scalar sx1 = interp(-new_x1 + i);

              // j1 is movement in x1
              int offset = i + pos[0] - dc1;
              djx += sx1 - sx0;
              atomicAdd(&J[0][offset], -weight * djx);
              // Logger::print_debug("J0 is {}", (*J)[0][offset]);

              // j2 is simply v2 times rho at center
              Scalar val1 = 0.5f * (sx0 + sx1);
              atomicAdd(&J[1][offset], weight * v2 * val1);

              // j3 is simply v3 times rho at center
              atomicAdd(&J[2][offset], weight * v3 * val1);

              // rho is deposited at the final position
              if ((step + 1) % data_interval == 0) {
                atomicAdd(&Rho[sp][offset], weight * sx1);
              }
            }
          }
        },
        this->ptc->dev_ptrs(), this->J->get_ptrs(), m_rho_ptrs.dev_ptr(),
        this->m_data_interval);
  }
}

template <typename Conf>
void
ptc_updater_cu<Conf>::move_deposit_2d(double dt, uint32_t step) {
  auto num = this->ptc->number();
  if (num > 0) {
    auto ext = this->m_grid.extent();

    kernel_launch(
        [ext, num, dt, step] __device__(auto ptc, auto J, auto Rho,
                                        auto data_interval) {
          using spline_t = typename base_class::spline_t;
          for (auto n : grid_stride_range(0, num)) {
            uint32_t cell = ptc.cell[n];
            if (cell == empty_cell) continue;

            auto idx = J[0].idx_at(cell, ext);
            auto pos = idx.get_pos();

            // step 1: Move particles
            auto x1 = ptc.x1[n], x2 = ptc.x2[n], x3 = ptc.x3[n];
            Scalar v1 = ptc.p1[n], v2 = ptc.p2[n], v3 = ptc.p3[n],
                   gamma = ptc.E[n];

            v1 /= gamma;
            v2 /= gamma;
            v3 /= gamma;

            auto new_x1 = x1 + (v1 * dt) * dev_grid_2d.inv_delta[0];
            int dc1 = std::floor(new_x1);
            pos[0] += dc1;
            ptc.x1[n] = new_x1 - (Pos_t)dc1;

            auto new_x2 = x2 + (v2 * dt) * dev_grid_2d.inv_delta[1];
            int dc2 = std::floor(new_x2);
            pos[1] += dc2;
            ptc.x2[n] = new_x2 - (Pos_t)dc2;

            ptc.x3[n] = x3 + v3 * dt;

            ptc.cell[n] = J[0].get_idx(pos, ext).linear;

            // step 2: Deposit current
            auto flag = ptc.flag[n];
            auto sp = get_ptc_type(flag);
            auto interp = spline_t{};
            if (check_flag(flag, PtcFlag::ignore_current)) continue;
            auto weight = dev_charges[sp] * ptc.weight[n];

            int j_0 = (dc2 == -1 ? -spline_t::radius : 1 - spline_t::radius);
            int j_1 = (dc2 == 1 ? spline_t::radius + 1 : spline_t::radius);
            int i_0 = (dc1 == -1 ? -spline_t::radius : 1 - spline_t::radius);
            int i_1 = (dc1 == 1 ? spline_t::radius + 1 : spline_t::radius);

            Scalar djy[2 * spline_t::radius + 1] = {};
            for (int j = j_0; j <= j_1; j++) {
              Scalar sy0 = interp(-x2 + j);
              Scalar sy1 = interp(-new_x2 + j);

              Scalar djx = 0.0f;
              for (int i = i_0; i <= i_1; i++) {
                Scalar sx0 = interp(-x1 + i);
                Scalar sx1 = interp(-new_x1 + i);

                // j1 is movement in x1
                auto offset = idx.inc_x(i).inc_y(j);
                djx += movement2d(sy0, sy1, sx0, sx1);
                atomicAdd(&J[0][offset], -weight * djx);
                // Logger::print_debug("J0 is {}", (*J)[0][offset]);

                // j2 is movement in x2
                djy[i - i_0] += movement2d(sx0, sx1, sy0, sy1);
                atomicAdd(&J[1][offset], -weight * djy[i - i_0]);

                // j3 is simply v3 times rho at center
                atomicAdd(&J[2][offset],
                          weight * v3 * center2d(sx0, sx1, sy0, sy1));

                // rho is deposited at the final position
                if ((step + 1) % data_interval == 0) {
                  atomicAdd(&Rho[sp][offset], weight * sx1 * sy1);
                }
              }
            }
          }
        },
        this->ptc->dev_ptrs(), this->J->get_ptrs(), m_rho_ptrs.dev_ptr(),
        this->m_data_interval);
  }
}

template <typename Conf>
void
ptc_updater_cu<Conf>::move_deposit_3d(double dt, uint32_t step) {
  auto num = this->ptc->number();
  if (num > 0) {
    auto ext = this->m_grid.extent();

    kernel_launch(
        [ext, num, dt, step] __device__(auto ptc, auto J, auto Rho,
                                        auto data_interval) {
          using spline_t = typename base_class::spline_t;
          for (auto n : grid_stride_range(0, num)) {
            uint32_t cell = ptc.cell[n];
            if (cell == empty_cell) continue;

            auto idx = J[0].idx_at(cell, ext);
            auto pos = idx.get_pos();

            // step 1: Move particles
            auto x1 = ptc.x1[n], x2 = ptc.x2[n], x3 = ptc.x3[n];
            Scalar v1 = ptc.p1[n], v2 = ptc.p2[n], v3 = ptc.p3[n],
                   gamma = ptc.E[n];

            v1 /= gamma;
            v2 /= gamma;
            v3 /= gamma;

            auto new_x1 = x1 + (v1 * dt) * dev_grid_3d.inv_delta[0];
            int dc1 = std::floor(new_x1);
            pos[0] += dc1;
            ptc.x1[n] = new_x1 - (Pos_t)dc1;

            auto new_x2 = x2 + (v2 * dt) * dev_grid_3d.inv_delta[1];
            int dc2 = std::floor(new_x2);
            pos[1] += dc2;
            ptc.x2[n] = new_x2 - (Pos_t)dc2;

            auto new_x3 = x3 + (v3 * dt) * dev_grid_3d.inv_delta[2];
            int dc3 = std::floor(new_x3);
            pos[2] += dc3;
            ptc.x3[n] = new_x3 - (Pos_t)dc3;

            ptc.cell[n] = J[0].get_idx(pos, ext).linear;

            // step 2: Deposit current
            auto flag = ptc.flag[n];
            auto sp = get_ptc_type(flag);
            auto interp = spline_t{};
            if (check_flag(flag, PtcFlag::ignore_current)) continue;
            auto weight = dev_charges[sp] * ptc.weight[n];

            int k_0 = (dc3 == -1 ? -spline_t::radius : 1 - spline_t::radius);
            int k_1 = (dc3 == 1 ? spline_t::radius + 1 : spline_t::radius);
            int j_0 = (dc2 == -1 ? -spline_t::radius : 1 - spline_t::radius);
            int j_1 = (dc2 == 1 ? spline_t::radius + 1 : spline_t::radius);
            int i_0 = (dc1 == -1 ? -spline_t::radius : 1 - spline_t::radius);
            int i_1 = (dc1 == 1 ? spline_t::radius + 1 : spline_t::radius);

            Scalar djz[2 * spline_t::radius + 1][2 * spline_t::radius + 1] = {};
            for (int k = k_0; k <= k_1; k++) {
              Scalar sz0 = interp(-x3 + k);
              Scalar sz1 = interp(-new_x3 + k);

              Scalar djy[2 * spline_t::radius + 1] = {};
              for (int j = j_0; j <= j_1; j++) {
                Scalar sy0 = interp(-x2 + j);
                Scalar sy1 = interp(-new_x2 + j);

                Scalar djx = 0.0f;
                for (int i = i_0; i <= i_1; i++) {
                  Scalar sx0 = interp(-x1 + i);
                  Scalar sx1 = interp(-new_x1 + i);

                  // j1 is movement in x1
                  auto offset = idx.inc_x(i).inc_y(j).inc_z(k);
                  djx += movement3d(sy0, sy1, sz0, sz1, sx0, sx1);
                  atomicAdd(&J[0][offset], -weight * djx);
                  // Logger::print_debug("J0 is {}", (*J)[0][offset]);

                  // j2 is movement in x2
                  djy[i - i_0] += movement3d(sz0, sz1, sx0, sx1, sy0, sy1);
                  atomicAdd(&J[1][offset], -weight * djy[i - i_0]);

                  // j3 is movement in x3
                  djz[j - j_0][i - i_0] +=
                      movement3d(sx0, sx1, sy0, sy1, sz0, sz1);
                  atomicAdd(&J[2][offset], -weight * djz[j - j_0][i - i_0]);

                  // rho is deposited at the final position
                  if ((step + 1) % data_interval == 0) {
                    atomicAdd(&Rho[sp][offset], weight * sx1 * sy1 * sz1);
                  }
                }
              }
            }
          }
        },
        this->ptc->dev_ptrs(), this->J->get_ptrs(), m_rho_ptrs.dev_ptr(),
        this->m_data_interval);
  }
}

template <typename Conf>
void
ptc_updater_cu<Conf>::clear_guard_cells() {
  auto ext = this->m_grid.extent();
  auto num = this->ptc->number();
  kernel_launch(
      [ext, num] __device__(auto ptc) {
        auto& grid = dev_grid<Conf::dim>();
        for (auto n : grid_stride_range(0, num)) {
          uint32_t cell = ptc.cell[n];
          if (cell == empty_cell) continue;
          auto idx = typename Conf::idx_t(cell, ext);
          auto pos = idx.get_pos();

          if (!grid.is_in_bound(pos)) ptc.cell[n] = empty_cell;
        }
      },
      this->ptc->dev_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
}

template <typename Conf>
void
ptc_updater_cu<Conf>::sort_particles() {
  this->ptc->sort_by_cell_dev(this->m_grid.extent().size());
}

template <typename Conf>
void
ptc_updater_cu<Conf>::fill_multiplicity(int mult,
                                        typename Conf::value_t weight) {
  auto num = this->ptc->number();
  using idx_t = typename Conf::idx_t;

  kernel_launch(
      [num, mult, weight] __device__(auto ptc, auto states) {
        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        int id = threadIdx.x + blockIdx.x * blockDim.x;
        cuda_rng_t rng(&states[id]);
        for (auto n : grid_stride_range(0, ext.size())) {
          auto idx = idx_t(n, ext);
          auto pos = idx.get_pos();
          if (grid.is_in_bound(pos)) {
            for (int i = 0; i < mult; i++) {
              uint32_t offset = num + idx.linear * mult * 2 + i * 2;

              ptc.x1[offset] = ptc.x1[offset + 1] = rng();
              ptc.x2[offset] = ptc.x2[offset + 1] = rng();
              ptc.x3[offset] = ptc.x3[offset + 1] = rng();
              ptc.p1[offset] = ptc.p1[offset + 1] = 0.0;
              ptc.p2[offset] = ptc.p2[offset + 1] = 0.0;
              ptc.p3[offset] = ptc.p3[offset + 1] = 0.0;
              ptc.E[offset] = ptc.E[offset + 1] = 1.0;
              ptc.cell[offset] = ptc.cell[offset + 1] = idx.linear;
              ptc.weight[offset] = ptc.weight[offset + 1] = weight;
              ptc.flag[offset] = set_ptc_type_flag(bit_or(PtcFlag::primary),
                                                   PtcType::electron);
              ptc.flag[offset + 1] = set_ptc_type_flag(bit_or(PtcFlag::primary),
                                                       PtcType::positron);
            }
          }
        }
      },
      this->ptc->dev_ptrs(), m_rand_states->states());
  CudaSafeCall(hipDeviceSynchronize());
  this->ptc->set_num(num + mult * 2 * this->m_grid.extent().size());
}

template <typename Conf>
void
ptc_updater_cu<Conf>::filter_field(vector_field<Conf>& f, int comp) {}

template <typename Conf>
void
ptc_updater_cu<Conf>::filter_field(scalar_field<Conf>& f) {}

template class ptc_updater_cu<Config<1>>;
template class ptc_updater_cu<Config<2>>;
template class ptc_updater_cu<Config<3>>;

}  // namespace Aperture
