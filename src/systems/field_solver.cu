#include "core/cuda_control.h"
#include "field_solver.h"
#include "framework/config.h"
#include "systems/helpers/finite_diff_helper.hpp"
#include "utils/kernel_helper.hpp"
#include "utils/timer.h"

namespace Aperture {

template <typename Conf>
void
compute_e_update_explicit(vector_field<Conf>& result,
                          const vector_field<Conf>& b,
                          const vector_field<Conf>& j,
                          typename Conf::value_t dt) {
  kernel_launch(
      [dt] __device__(auto result, auto b, auto stagger, auto j) {
        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          auto pos = idx.get_pos();
          if (grid.is_in_bound(pos)) {
            result[0][idx] += dt * (finite_diff<Conf::dim>::curl0(b, idx, stagger)
                                    - j[0][idx]);

            result[1][idx] += dt * (finite_diff<Conf::dim>::curl1(b, idx, stagger)
                                    - j[1][idx]);

            result[2][idx] += dt * (finite_diff<Conf::dim>::curl2(b, idx, stagger)
                                    - j[1][idx]);
          }
        }
      },
      result.get_ptrs(), b.get_ptrs(), b.stagger_vec(), j.get_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
compute_b_update_explicit(vector_field<Conf>& result,
                          const vector_field<Conf>& e,
                          typename Conf::value_t dt) {
  kernel_launch(
      [dt] __device__(auto result, auto e, auto stagger) {
        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          auto pos = idx.get_pos();
          if (grid.is_in_bound(pos)) {
            result[0][idx] += dt * finite_diff<Conf::dim>::curl0(e, idx, stagger);

            result[1][idx] += dt * finite_diff<Conf::dim>::curl1(e, idx, stagger);

            result[2][idx] += dt * finite_diff<Conf::dim>::curl2(e, idx, stagger);
          }
        }
      },
      result.get_ptrs(), e.get_ptrs(), e.stagger_vec());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
field_solver_cu<Conf>::init_impl_tmp_fields() {
  if (this->m_use_implicit) {
    this->m_tmp_b1 = std::make_unique<vector_field<Conf>>(this->m_grid,
                                                          MemType::device_only);
    this->m_tmp_b2 = std::make_unique<vector_field<Conf>>(this->m_grid,
                                                          MemType::device_only);
    this->m_bnew = std::make_unique<vector_field<Conf>>(this->m_grid,
                                                        MemType::device_only);
  }
}

template <typename Conf>
void
field_solver_cu<Conf>::register_data_components() {
  this->register_data_impl(MemType::host_device);
}

template <typename Conf>
void
field_solver_cu<Conf>::update_explicit(double dt, double time) {
  timer::stamp("field_update");
  if (time < TINY) {
    compute_e_update_explicit(*(this->E), *(this->B), *(this->J), 0.5f * dt);
    if (this->m_comm != nullptr) this->m_comm->send_guard_cells(*(this->E));
  }

  compute_b_update_explicit(*(this->B), *(this->E), dt);

  // Communicate the new B values to guard cells
  if (this->m_comm != nullptr) this->m_comm->send_guard_cells(*(this->B));

  compute_e_update_explicit(*(this->E), *(this->B), *(this->J), dt);
  // Communicate the new E values to guard cells
  if (this->m_comm != nullptr) this->m_comm->send_guard_cells(*(this->E));

  // compute_divs();

  CudaSafeCall(hipDeviceSynchronize());
  timer::show_duration_since_stamp("Field update", "ms", "field_update");
}

template <typename Conf>
void
field_solver_cu<Conf>::update_semi_implicit(double dt, double alpha,
                                            double beta, double time) {
  // FIXME: implement semi implicit update!!!
}

INSTANTIATE_CONFIG(field_solver_cu);

}  // namespace Aperture
