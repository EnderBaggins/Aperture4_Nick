#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "compute_lorentz_factor.h"
#include "framework/environment.h"
#include "framework/config.h"
#include "utils/kernel_helper.hpp"
#include "utils/range.hpp"
#include <memory>

namespace Aperture {

template <typename Conf>
void
compute_lorentz_factor_cu<Conf>::register_data_components() {
  this->register_data_impl(MemType::host_device);
}

template <typename Conf>
void
compute_lorentz_factor_cu<Conf>::init() {
  compute_lorentz_factor<Conf>::init();

  m_nums.resize(this->gamma.size());
  for (auto& p : m_nums) {
    p = std::make_unique<scalar_field<Conf>>(
        this->m_grid, field_type::cell_centered, MemType::device_only);
  }

  // Initialize the gamma and particle number pointers
  m_gamma_ptrs.set_memtype(MemType::host_device);
  m_nums_ptrs.set_memtype(MemType::host_device);
  m_gamma_ptrs.resize(this->gamma.size());
  m_nums_ptrs.resize(this->gamma.size());
  for (int i = 0; i < this->gamma.size(); i++) {
    m_gamma_ptrs[i] = this->gamma[i]->get_ptr();
    m_nums_ptrs[i] = m_nums[i]->get_ptr();
  }
  m_gamma_ptrs.copy_to_device();
  m_nums_ptrs.copy_to_device();
}

template <typename Conf>
void
compute_lorentz_factor_cu<Conf>::update(double dt, uint32_t step) {
  if (step % this->m_data_interval != 0) return;

  // Compute average Lorentz factors of all particles in every cell
  for (auto g : this->gamma) g->init();
  for (auto& p : this->m_nums) p->init();

  auto num = this->ptc->number();
  if (num > 0) {
    kernel_launch([num] __device__(auto ptc, auto gammas, auto nums) {
          auto& grid = dev_grid<Conf::dim>();
          auto ext = grid.extent();
          for (auto n : grid_stride_range(0, num)) {
            uint32_t cell = ptc.cell[n];
            if (cell == empty_cell) return;

            auto idx = typename Conf::idx_t(cell, ext);

            auto g = ptc.E[n];
            auto weight = ptc.weight[n];
            auto flag = ptc.flag[n];
            auto sp = get_ptc_type(flag);

            atomicAdd(&gammas[sp][idx], weight * g);
            atomicAdd(&nums[sp][idx], weight);
          }
      }, this->ptc->dev_ptrs(), this->m_gamma_ptrs.dev_ptr(),
      m_nums_ptrs.dev_ptr());
    CudaCheckError();

    int num_species = this->m_num_species;
    kernel_launch([num_species] __device__(auto gammas, auto nums) {
        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto idx : grid_stride_range(Conf::begin(ext), Conf::end(ext))) {
          for (int i = 0; i < num_species; i++) {
            if (nums[i][idx] > TINY) {
              gammas[i][idx] /= nums[i][idx];
            } else {
              gammas[i][idx] = 0.0f;
            }
          }
        }
      }, this->m_gamma_ptrs.dev_ptr(), m_nums_ptrs.dev_ptr());
    CudaSafeCall(hipDeviceSynchronize());
    CudaCheckError();
  }
}

INSTANTIATE_WITH_CONFIG(compute_lorentz_factor_cu);

}  // namespace Aperture
