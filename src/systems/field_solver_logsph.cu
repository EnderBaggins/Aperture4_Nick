#include "hip/hip_runtime.h"
#include "field_solver_logsph.h"
#include "framework/config.h"
#include "framework/environment.hpp"
#include "utils/kernel_helper.hpp"
#include "utils/timer.h"

namespace Aperture {

template <typename PtrType, typename Idx_t>
HD_INLINE typename PtrType::value_t
circ0(const vec_t<PtrType, 3>& f, const vec_t<PtrType, 3>& dl, const Idx_t& idx,
      const Idx_t& idx_py) {
  return f[2][idx_py] * dl[2][idx_py] - f[2][idx] * dl[2][idx];
}

template <typename PtrType, typename Idx_t>
HD_INLINE typename PtrType::value_t
circ1(const vec_t<PtrType, 3>& f, const vec_t<PtrType, 3>& dl, const Idx_t& idx,
      const Idx_t& idx_px) {
  return f[2][idx] * dl[2][idx] - f[2][idx_px] * dl[2][idx_px];
}

template <typename PtrType, typename Idx_t>
HD_INLINE typename PtrType::value_t
circ2(const vec_t<PtrType, 3>& f, const vec_t<PtrType, 3>& dl,
      const Idx_t& idx_mx, const Idx_t& idx_my, const Idx_t& idx_px,
      const Idx_t& idx_py) {
  return f[1][idx_px] * dl[1][idx_px] - f[1][idx_mx] * dl[1][idx_mx] +
         f[0][idx_my] * dl[0][idx_my] - f[0][idx_py] * dl[0][idx_py];
}

template <typename Conf>
void
compute_double_circ(vector_field<Conf>& result, const vector_field<Conf>& b,
                    const grid_logsph_t<Conf>& grid,
                    typename Conf::value_t coef) {
  auto ext = grid.extent();
  kernel_launch(
      [coef, ext] __device__(auto result, auto b, auto grid_ptrs) {
        for (auto n : grid_stride_range(0, ext.size())) {
        }
      },
      result.get_ptrs(), b.get_ptrs(), grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
}

template <typename Conf>
void
compute_implicit_rhs(vector_field<Conf>& result, const vector_field<Conf>& e,
                     const vector_field<Conf>& e0, const vector_field<Conf>& j,
                     const grid_logsph_t<Conf>& grid,
                     typename Conf::value_t alpha, typename Conf::value_t beta,
                     typename Conf::value_t dt) {
  auto ext = grid.extent();
  kernel_launch(
      [alpha, beta, dt, ext] __device__(auto result, auto e, auto e0, auto j,
                                        auto gp) {
        // gp is short for grid_ptrs
        for (auto n : grid_stride_range(0, ext.size())) {
          auto idx = result[0].idx_at(n, ext);
          auto pos = idx.get_pos();
          if (dev_grid<Conf>().is_in_bound(pos)) {
            auto idx_py = idx.inc_y();
            result[0][idx] +=
                -dt *
                ((alpha + beta) * (circ0(e, gp.le, idx, idx_py) -
                                   circ0(e0, gp.le, idx, idx_py)) -
                 dt * beta * circ0(j, gp.le, idx, idx_py)) /
                gp.Ab[0][idx];

            auto idx_px = idx.inc_x();
            result[1][idx] +=
                -dt *
                ((alpha + beta) * (circ1(e, gp.le, idx, idx_px) -
                                   circ1(e0, gp.le, idx, idx_px)) -
                 dt * beta * circ1(j, gp.le, idx, idx_px)) /
                gp.Ab[1][idx];

            result[2][idx] +=
                -dt *
                ((alpha + beta) * (circ2(e, gp.le, idx, idx, idx_px, idx_py) -
                                   circ2(e0, gp.le, idx, idx, idx_px, idx_py)) -
                 dt * beta * circ2(j, gp.le, idx, idx, idx_px, idx_py)) /
                gp.Ab[2][idx];
          }
        }
      },
      result.get_ptrs(), e.get_ptrs(), e0.get_ptrs(), j.get_ptrs(),
      grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
}

template <typename Conf>
void
compute_e_update_explicit(vector_field<Conf>& result,
                          const vector_field<Conf>& b,
                          const vector_field<Conf>& b0,
                          const vector_field<Conf>& j,
                          const grid_logsph_t<Conf>& grid,
                          typename Conf::value_t dt) {
  auto ext = grid.extent();
  kernel_launch(
      [dt, ext] __device__(auto result, auto b, auto b0, auto j, auto gp) {
        // gp is short for grid_ptrs
        for (auto n : grid_stride_range(0, ext.size())) {
          auto idx = result[0].idx_at(n, ext);
          auto pos = idx.get_pos();
          if (dev_grid<Conf::dim>().is_in_bound(pos)) {
            auto idx_mx = idx.dec_x();
            result[0][idx] += dt * ((circ0(b, gp.lb, idx_mx, idx) -
                                     circ0(b0, gp.lb, idx_mx, idx)) /
                                        gp.Ae[0][idx] -
                                    j[0][idx]);

            auto idx_my = idx.dec_y();
            result[1][idx] += dt * ((circ1(b, gp.lb, idx_my, idx) -
                                     circ1(b0, gp.lb, idx_my, idx)) /
                                        gp.Ae[1][idx] -
                                    j[1][idx]);

            result[2][idx] +=
                dt * ((circ2(b, gp.lb, idx_mx, idx_my, idx, idx) -
                       circ2(b0, gp.lb, idx_mx, idx_my, idx, idx)) /
                          gp.Ae[2][idx] -
                      j[2][idx]);
          }
        }
      },
      result.get_ptrs(), b.get_ptrs(), b0.get_ptrs(), j.get_ptrs(),
      grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
}

template <typename Conf>
void
compute_b_update_explicit(vector_field<Conf>& result,
                          const vector_field<Conf>& e,
                          const vector_field<Conf>& e0,
                          const grid_logsph_t<Conf>& grid,
                          typename Conf::value_t dt) {
  auto ext = grid.extent();
  kernel_launch(
      [dt, ext] __device__(auto result, auto e, auto e0, auto gp) {
        // gp is short for grid_ptrs
        for (auto n : grid_stride_range(0, ext.size())) {
          auto idx = result[0].idx_at(n, ext);
          auto pos = idx.get_pos();
          if (dev_grid<Conf::dim>().is_in_bound(pos)) {
            auto idx_px = idx.inc_x();
            result[0][idx] += dt * (circ0(e, gp.le, idx, idx_px) -
                                    circ0(e0, gp.le, idx, idx_px)) /
                gp.Ab[0][idx];

            auto idx_py = idx.inc_y();
            result[1][idx] += dt * (circ1(e, gp.le, idx, idx_py) -
                                    circ1(e0, gp.le, idx, idx_py)) /
                gp.Ab[1][idx];

            result[2][idx] +=
                dt * (circ2(e, gp.le, idx, idx, idx_px, idx_py) -
                      circ2(e0, gp.le, idx, idx, idx_px, idx_py)) /
                gp.Ab[2][idx];
          }
        }
      },
      result.get_ptrs(), e.get_ptrs(), e0.get_ptrs(),
      grid.get_grid_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
}

template <typename Conf>
void
axis_boundary(vector_field<Conf>& e, vector_field<Conf>& b,
              const vector_field<Conf>& e0, const vector_field<Conf>& b0,
              const grid_logsph_t<Conf>& grid) {
  auto ext = grid.extent();
  typedef typename Conf::idx_t idx_t;
  kernel_launch([ext]__device__ (auto e, auto b, auto e0, auto b0) {
      auto& grid = dev_grid<Conf::dim>();
      for (auto n0 : grid_stride_range(0, dev_grid<Conf::dim>().dims[0])) {
        auto n1_0 = grid.guard[1];
        auto n1_pi = grid.dims[1] - grid.guard[1];
        if (abs(grid.template pos<1>(n1_0, true)) < TINY) {
          // At the theta = 0 axis

          // Set E_phi and B_theta to zero
          auto idx = idx_t(index_t<2>(n0, n1_0), ext);
          e[2][idx] = 0.0;
          b[1][idx] = 0.0;
          b[2][idx.dec_y()] = b[2][idx];
        }
        if (abs(grid.template pos<1>(n1_pi, true)) < TINY) {
          // At the theta = pi axis
          auto idx = idx_t(index_t<2>(n0, n1_pi), ext);
          e[2][idx] = 0.0;
          b[1][idx] = 0.0;
          b[2][idx] = b[2][idx.dec_y()];
        }
      }
    }, e.get_ptrs(), b.get_ptrs(), e0.get_ptrs(), b0.get_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
}

template <typename Conf>
void
field_solver_logsph<Conf>::init() {
  this->m_env.params().get_value("implicit_alpha", m_alpha);
  this->m_env.params().get_value("implicit_beta", m_beta);

  m_tmp_b1 =
      std::make_unique<vector_field<Conf>>(this->m_grid, MemType::device_only);
  m_tmp_b2 =
      std::make_unique<vector_field<Conf>>(this->m_grid, MemType::device_only);
}

template <typename Conf>
void
field_solver_logsph<Conf>::update(double dt, uint32_t step) {
  double time = this->m_env.get_time();
  update_semi_impl(dt, m_alpha, m_beta, time);
}

template <typename Conf>
void
field_solver_logsph<Conf>::update_explicit(double dt, double time) {
  auto& grid = dynamic_cast<const grid_logsph_t<Conf>&>(this->m_grid);
  compute_b_update_explicit(*(this->B), *(this->E), *(this->E0), grid, dt);

  // Communicate B guard cells
  if (this->m_comm != nullptr) this->m_comm->send_guard_cells(*(this->B));

  compute_e_update_explicit(*(this->E), *(this->B), *(this->B0), *(this->J), grid, dt);

  // Communicate E guard cells
  if (this->m_comm != nullptr) this->m_comm->send_guard_cells(*(this->E));

  // TODO: Compute divE and divB

  // apply coordinate boundary condition
  axis_boundary(*(this->E), *(this->B), *(this->E0), *(this->B0), grid);
}

template <typename Conf>
void
field_solver_logsph<Conf>::update_semi_impl(double dt, double alpha,
                                            double beta, double time) {
  // set m_tmp_b1 to B - B0
  m_tmp_b1->copy_from(*(this->B));
  m_tmp_b1->add_by(*(this->B0), -1.0);

  auto& grid = dynamic_cast<const grid_logsph_t<Conf>&>(this->m_grid);
  compute_double_circ(*m_tmp_b2, *m_tmp_b1, grid, -alpha * beta * dt * dt);
  m_tmp_b1->add_by(*m_tmp_b2);

  // Send guard cells for m_tmp_b1
  if (this->m_comm != nullptr) this->m_comm->send_guard_cells(*m_tmp_b1);
}

template <typename Conf>
void
field_solver_logsph<Conf>::update_b(double dt, double alpha, double beta) {}

template <typename Conf>
void
field_solver_logsph<Conf>::update_e(double dt, double alpha, double beta) {}

template class field_solver_logsph<Config<2>>;

}  // namespace Aperture
