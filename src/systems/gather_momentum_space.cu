#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "framework/config.h"
#include "framework/environment.h"
#include "framework/params_store.h"
#include "gather_momentum_space.h"
#include "utils/kernel_helper.hpp"

namespace Aperture {

template <typename Conf>
void
gather_momentum_space_cu<Conf>::update(double dt, uint32_t step) {
  // Convert these into things that can be passed onto the gpu
  vec_t<int, 3> num_bins(this->momentum->m_num_bins);
  vec_t<float, 3> lower(this->momentum->m_lower);
  vec_t<float, 3> upper(this->momentum->m_upper);

  // Loop over the particle array to gather momentum space information
  auto num = this->ptc->number();

  kernel_launch(
      [num, num_bins, lower, upper] __device__(auto ptc, auto e_p1, auto e_p2,
                                               auto e_p3, auto p_p1, auto p_p2,
                                               auto p_p3, int downsample) {
        auto &grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        auto ext_out = grid.extent_less() / downsample;
        using idx_t = default_idx_t<Conf::dim + 1>;
        for (auto n : grid_stride_range(0, num)) {
          uint32_t cell = ptc.cell[n];
          if (cell == empty_cell) continue;

          auto idx = Conf::idx(cell, ext);
          auto pos = idx.get_pos();
          if (grid.is_in_bound(pos)) {
            index_t<Conf::dim + 1> pos_out(0, (pos - grid.guards()) / downsample);

            auto weight = ptc.weight[n];
            auto flag = ptc.flag[n];
            auto sp = get_ptc_type(flag);

            auto p1 = ptc.p1[n];
            auto p2 = ptc.p2[n];
            auto p3 = ptc.p3[n];

            int bin1 = floor((p1 - lower[0]) / (upper[0] - lower[0]) * num_bins[0]);
            int bin2 = floor((p2 - lower[1]) / (upper[1] - lower[1]) * num_bins[1]);
            int bin3 = floor((p3 - lower[2]) / (upper[2] - lower[2]) * num_bins[2]);

            if (sp == (int)PtcType::electron) {
              pos_out[0] = bin1;
              atomicAdd(&e_p1[idx_t(pos_out, extent_t<Conf::dim + 1>(num_bins[0], ext_out))],
                        weight);
              pos_out[1] = bin2;
              atomicAdd(&e_p2[idx_t(pos_out, extent_t<Conf::dim + 1>(num_bins[1], ext_out))],
                        weight);
              pos_out[2] = bin3;
              atomicAdd(&e_p3[idx_t(pos_out, extent_t<Conf::dim + 1>(num_bins[2], ext_out))],
                        weight);
            } else if (sp == (int)PtcType::positron) {
              pos_out[0] = bin1;
              atomicAdd(&p_p1[idx_t(pos_out, extent_t<Conf::dim + 1>(num_bins[0], ext_out))],
                        weight);
              pos_out[1] = bin2;
              atomicAdd(&p_p2[idx_t(pos_out, extent_t<Conf::dim + 1>(num_bins[1], ext_out))],
                        weight);
              pos_out[2] = bin3;
              atomicAdd(&p_p3[idx_t(pos_out, extent_t<Conf::dim + 1>(num_bins[2], ext_out))],
                        weight);
            }
          }
        }
      },
      this->ptc->dev_ptrs(), this->momentum->e_p1.dev_ndptr(),
      this->momentum->e_p2.dev_ndptr(), this->momentum->e_p3.dev_ndptr(),
      this->momentum->p_p1.dev_ndptr(), this->momentum->p_p2.dev_ndptr(),
      this->momentum->p_p3.dev_ndptr(), this->momentum->m_downsample);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

INSTANTIATE_WITH_CONFIG(gather_momentum_space_cu);

}  // namespace Aperture
