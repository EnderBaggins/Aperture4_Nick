#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "framework/config.h"
#include "framework/environment.h"
#include "framework/params_store.h"
#include "gather_momentum_space.h"
#include "utils/kernel_helper.hpp"

namespace Aperture {

template <typename Conf>
void
gather_momentum_space_cu<Conf>::update(double dt, uint32_t step) {
  if (step % this->m_data_interval != 0) return;
  this->momentum->init();
  // Convert these into things that can be passed onto the gpu
  vec_t<int, 3> num_bins(this->momentum->m_num_bins);
  vec_t<float, 3> lower(this->momentum->m_lower);
  vec_t<float, 3> upper(this->momentum->m_upper);

  // Loop over the particle array to gather momentum space information
  auto num = this->ptc->number();

  Logger::print_info("gathering particle momentum space");
  kernel_launch(
      [num, num_bins, lower, upper] __device__(auto ptc, auto e_p1, auto e_p2,
                                               auto e_p3, auto p_p1, auto p_p2,
                                               auto p_p3, int downsample) {
        auto &grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();
        auto ext_out = grid.extent_less() / downsample;
        using idx_t = default_idx_t<Conf::dim + 1>;
        for (auto n : grid_stride_range(0, num)) {
          uint32_t cell = ptc.cell[n];
          if (cell == empty_cell) continue;

          auto idx = Conf::idx(cell, ext);
          auto pos = idx.get_pos();
          if (grid.is_in_bound(pos)) {
            index_t<Conf::dim + 1> pos_out(0, (pos - grid.guards()) / downsample);

            auto weight = ptc.weight[n];
            auto flag = ptc.flag[n];
            auto sp = get_ptc_type(flag);

            auto p1 = clamp(ptc.p1[n], lower[0], upper[0]);
            auto p2 = clamp(ptc.p2[n], lower[1], upper[1]);
            auto p3 = clamp(ptc.p3[n], lower[2], upper[2]);

            int bin1 = floor((p1 - lower[0]) / (upper[0] - lower[0]) * (num_bins[0] - 1));
            int bin2 = floor((p2 - lower[1]) / (upper[1] - lower[1]) * (num_bins[1] - 1));
            int bin3 = floor((p3 - lower[2]) / (upper[2] - lower[2]) * (num_bins[2] - 1));

            if (sp == (int)PtcType::electron) {
              pos_out[0] = bin1;
              atomicAdd(&e_p1[idx_t(pos_out, extent_t<Conf::dim + 1>(num_bins[0], ext_out))],
                        weight);
              pos_out[0] = bin2;
              atomicAdd(&e_p2[idx_t(pos_out, extent_t<Conf::dim + 1>(num_bins[1], ext_out))],
                        weight);
              pos_out[0] = bin3;
              atomicAdd(&e_p3[idx_t(pos_out, extent_t<Conf::dim + 1>(num_bins[2], ext_out))],
                        weight);
            } else if (sp == (int)PtcType::positron) {
              pos_out[0] = bin1;
              atomicAdd(&p_p1[idx_t(pos_out, extent_t<Conf::dim + 1>(num_bins[0], ext_out))],
                        weight);
              pos_out[0] = bin2;
              atomicAdd(&p_p2[idx_t(pos_out, extent_t<Conf::dim + 1>(num_bins[1], ext_out))],
                        weight);
              pos_out[0] = bin3;
              atomicAdd(&p_p3[idx_t(pos_out, extent_t<Conf::dim + 1>(num_bins[2], ext_out))],
                        weight);
            }
          }
        }
      },
      this->ptc->dev_ptrs(), this->momentum->e_p1.dev_ndptr(),
      this->momentum->e_p2.dev_ndptr(), this->momentum->e_p3.dev_ndptr(),
      this->momentum->p_p1.dev_ndptr(), this->momentum->p_p2.dev_ndptr(),
      this->momentum->p_p3.dev_ndptr(), this->momentum->m_downsample);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

INSTANTIATE_WITH_CONFIG(gather_momentum_space_cu);

}  // namespace Aperture
