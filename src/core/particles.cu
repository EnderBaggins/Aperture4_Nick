#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "core/constant_mem.h"
#include "core/math.hpp"
#include "core/typedefs_and_constants.h"
#include "framework/config.h"
#include "particles_impl.hpp"
#include "utils/for_each_dual.hpp"
#include "utils/kernel_helper.hpp"
#include "utils/range.hpp"
#include "utils/timer.h"
#include "visit_struct/visit_struct.hpp"

#include <thrust/binary_search.h>
#include <thrust/device_ptr.h>
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/replace.h>
#include <thrust/sort.h>

namespace Aperture {

template <typename Conf>
void
compute_target_buffers(const uint32_t* cells, size_t num,
                       buffer<int>& buffer_num, size_t* index) {
  kernel_launch(
      [num] __device__(auto cells, auto buffer_num, auto index) {
        auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();
        int zone_offset = 0;
        if (Conf::dim == 2)
          zone_offset = 9;
        else if (Conf::dim == 1)
          zone_offset = 12;
        for (auto n : grid_stride_range(0, num)) {
          uint32_t cell = cells[n];
          if (cell == empty_cell) continue;
          auto idx = Conf::idx(cell, ext);
          auto grid_pos = get_pos(idx, ext);
          size_t zone = grid.find_zone(grid_pos) + zone_offset;
          if (zone == 13) continue;
          size_t pos = atomicAdd(&buffer_num[zone], 1);
          // printf("pos is %lu, zone is %lu\n", pos, zone);
          // Zone is less than 32, so we can use 5 bits to represent this. The
          // rest of the bits go to encode the index of this particle in that
          // zone.
          index[n] = ((zone & 0b11111) << (sizeof(size_t) * 8 - 5)) + pos;
        }
      },
      cells, buffer_num.dev_ptr(), index);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf, typename PtcPtrs>
void
copy_component_to_buffer(PtcPtrs ptc_data, size_t num, size_t* index,
                         buffer<PtcPtrs>& ptc_buffers) {
  kernel_launch(
      [num] __device__(auto ptc_data, auto index, auto ptc_buffers) {
        auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();
        int bitshift_width = (sizeof(size_t) * 8 - 5);
        int zone_offset = 0;
        if (Conf::dim == 2)
          zone_offset = 9;
        else if (Conf::dim == 1)
          zone_offset = 12;
        // loop through the particle array
        for (auto n : grid_stride_range(0, num)) {
          auto cell = ptc_data.cell[n];
          if (cell == empty_cell) continue;
          size_t i = index[n];
          size_t zone = ((i >> bitshift_width) & 0b11111);
          if (zone == 13 || zone > 27) continue;
          size_t pos = i - (zone << bitshift_width);
          // Copy the particle data from ptc_data[n] to ptc_buffers[zone][pos]
          assign_ptc(ptc_buffers[zone - zone_offset], pos, ptc_data, n);
          // printf("pos is %lu, %u, %u\n", pos, ptc_buffers[zone -
          //                                                 zone_offset].cell[pos],
          //                                                 ptc_data.cell[n]);
          // printf("target zone is %lu\n", zone - zone_offset);
          // Compute particle cell delta
          int dz = (Conf::dim > 2 ? (zone / 9) - 1 : 0);
          int dy = (Conf::dim > 1 ? (zone / 3) % 3 - 1 : 0);
          int dx = zone % 3 - 1;
          auto idx = Conf::idx(cell, ext);
          // int dcell =
          //     -dz * grid.reduced_dim(2) * grid.dims[0] * grid.dims[1] -
          //     dy * grid.reduced_dim(1) * grid.dims[0] -
          //     dx * grid.reduced_dim(0);
          ptc_buffers[zone - zone_offset].cell[pos] =
              idx.dec_z(dz * grid.reduced_dim(2))
                  .dec_y(dy * grid.reduced_dim(1))
                  .dec_x(dx * grid.reduced_dim(0))
                  .linear;
          // printf("dc is %d, cell is %u, cell after is %u, zone is %lu\n",
          // dcell,
          //        ptc_data.cell[n],
          //        ptc_buffers[zone - zone_offset].cell[pos],
          //        zone - zone_offset);
          // Set the particle to empty
          ptc_data.cell[n] = empty_cell;
        }
      },
      ptc_data, index, ptc_buffers.dev_ptr());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename BufferType>
void
particles_base<BufferType>::resize_tmp_arrays() {
  if (m_index.size() != m_sort_segment_size ||
      m_tmp_data.size() != m_sort_segment_size) {
    m_index.resize(m_sort_segment_size);
    m_tmp_data.resize(m_sort_segment_size);
    m_segment_nums.set_memtype(MemType::host_device);
    m_segment_nums.resize(m_size / m_sort_segment_size + 1);
  }
}

template <typename BufferType>
void
particles_base<BufferType>::rearrange_arrays(const std::string& skip,
                                             size_t offset, size_t num) {
  auto ptc = typename BufferType::single_type{};
  for_each_double_with_name(
      m_dev_ptrs, ptc,
      [this, offset, num, &skip](const char* name, auto& x, auto& u) {
        typedef typename std::remove_reference<decltype(x)>::type x_type;
        auto ptr_index = thrust::device_pointer_cast(m_index.dev_ptr());
        if (std::strcmp(name, skip.c_str()) == 0) return;

        auto x_ptr = thrust::device_pointer_cast(x + offset);
        auto tmp_ptr = thrust::device_pointer_cast(
            reinterpret_cast<x_type>(m_tmp_data.dev_ptr()));
        thrust::gather(ptr_index, ptr_index + num, x_ptr, tmp_ptr);
        thrust::copy_n(tmp_ptr, num, x_ptr);
        CudaCheckError();
      });
}

template <typename BufferType>
void
particles_base<BufferType>::sort_by_cell_dev(size_t max_cell) {
  if (m_number > 0) {
    // Lazy resize the tmp arrays
    resize_tmp_arrays();
    m_segment_nums.assign_host(0);

    // 1st: Sort the particle array segment by segment
    for (int n = 0; n < m_number / m_sort_segment_size + 1; n++) {
      // Logger::print_info("Sorting segment {}", n);
      size_t offset = n * m_sort_segment_size;
      // Fringe case of m_number being an exact multiple of segment_size
      if (offset == m_number) {
        m_segment_nums[n] = 0;
        continue;
      }
      // Generate particle index array
      auto ptr_cell =
          thrust::device_pointer_cast(this->cell.dev_ptr() + offset);
      auto ptr_idx = thrust::device_pointer_cast(m_index.dev_ptr());

      // Sort the index array by key
      size_t sort_size = std::min(m_sort_segment_size, m_number - offset);
      thrust::counting_iterator<size_t> iter(0);
      thrust::copy_n(iter, sort_size, ptr_idx);

      // Logger::print_info("Sort_size is {}, offset is {}", sort_size, offset);
      thrust::sort_by_key(ptr_cell, ptr_cell + sort_size, ptr_idx);

      // Move the rest of particle array using the new index
      // Logger::print_info("Rearranging");
      rearrange_arrays("cell", offset, sort_size);

      // Update the new number of particles in each sorted segment
      m_segment_nums[n] =
          thrust::upper_bound(ptr_cell, ptr_cell + sort_size,
                              empty_cell - 1) -
          ptr_cell;
      // Logger::print_info("segment[{}] has size {}", n, m_segment_nums[n]);
    }

    // 2nd: Defragment the particle array
    int last_segment = m_number / m_sort_segment_size;
    for (int m = 0; m < last_segment; m++) {
      // Logger::print_info(
      //     "Filling segment {}, last_segment is {}, num_last is {}", m,
      //     last_segment, m_segment_nums[last_segment]);

      while (m_segment_nums[m] < m_sort_segment_size) {
        // deficit is how many "holes" do we have in this segment
        int deficit = m_sort_segment_size - m_segment_nums[m];
        // do not copy more particles than the number in the last segment
        int num_to_copy = std::min(deficit, m_segment_nums[last_segment]);
        // calculate offsets
        size_t offset_from = last_segment * m_sort_segment_size +
                             m_segment_nums[last_segment] - num_to_copy;
        size_t offset_to = m * m_sort_segment_size + m_segment_nums[m];
        // Logger::print_info(
        //     "deficit is {}, num_to_copy is {}, offset_from is {}", deficit,
        //     num_to_copy, offset_from);

        // Copy the particles from the end of the last segment to the end of
        // this segment
        copy_from(*this, num_to_copy, offset_from, offset_to);
        // Erase particles from the last segment
        erase(offset_from, num_to_copy);

        m_segment_nums[m] += num_to_copy;
        m_segment_nums[last_segment] -= num_to_copy;
        // Logger::print_info("Segment num is {}", m_segment_nums[m]);

        if (m_segment_nums[last_segment] == 0) {
          last_segment -= 1;
          if (last_segment == m) break;
        }
      }
    }

    // Logger::print_info("Last segment size is {}",
    // m_segment_nums[last_segment]);
    m_number =
        last_segment * m_sort_segment_size + m_segment_nums[last_segment];

    hipDeviceSynchronize();
    CudaCheckError();
  }
}

template <typename BufferType>
void
particles_base<BufferType>::append_dev(const vec_t<Scalar, 3>& x,
                                       const vec_t<Scalar, 3>& p, uint32_t cell,
                                       Scalar weight, uint32_t flag) {
  if (m_number == m_size) return;
  kernel_launch(
      {1, 1},
      [x, p, cell, weight, flag] __device__(auto ptrs, size_t pos) {
        ptrs.x1[pos] = x[0];
        ptrs.x2[pos] = x[1];
        ptrs.x3[pos] = x[2];
        ptrs.p1[pos] = p[0];
        ptrs.p2[pos] = p[1];
        ptrs.p3[pos] = p[2];
        ptrs.E[pos] =
            math::sqrt(1.0f + p[0] * p[0] + p[1] * p[1] + p[2] * p[2]);
        ptrs.weight[pos] = weight;
        ptrs.cell[pos] = cell;
        ptrs.flag[pos] = flag;
      },
      m_dev_ptrs, m_number);
  CudaSafeCall(hipDeviceSynchronize());
  m_number += 1;
}

template <typename BufferType>
template <typename Conf>
void
particles_base<BufferType>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Conf>& grid) {
  if (m_number > 0) {
    // timer::stamp("compute_buffer");
    if (m_index.size() != m_size) m_index.resize(m_size);
    m_index.assign_dev(0, m_number, size_t(-1));
    auto ptr_idx = thrust::device_pointer_cast(m_index.dev_ptr());

    m_zone_buffer_num.assign_dev(0);
    compute_target_buffers<Conf>(this->cell.dev_ptr(), m_number,
                                 m_zone_buffer_num, m_index.dev_ptr());
    m_zone_buffer_num.copy_to_host();
    CudaSafeCall(hipDeviceSynchronize());
    // timer::show_duration_since_stamp("Computing target buffers", "ms",
    // "compute_buffer");

    int zone_offset = 0;
    if (buffers.size() == 9)
      zone_offset = 9;
    else if (buffers.size() == 3)
      zone_offset = 12;
    for (unsigned int i = 0; i < buffers.size(); i++) {
      // Logger::print_debug("zone {} buffer has {} ptc", i + zone_offset,
      //                     m_zone_buffer_num[i + zone_offset]);
      if (i + zone_offset == 13) continue;
      buffers[i].set_num(m_zone_buffer_num[i + zone_offset]);
    }
    // timer::stamp("copy_to_buffer");
    copy_component_to_buffer<Conf>(m_dev_ptrs, m_number, m_index.dev_ptr(),
                                   buf_ptrs);
    // for (unsigned int i = 0; i < buffers.size(); i++) {
    //   if (buffers[i].number() > 0) {
    //     buffers[i].copy_to_host();
    //   }
    // }
    // if (buffers[7].number() > 0) {
    //   buffers[7].copy_to_host();
    //   Logger::print_debug("buffer[7] cell[0] is {}", buffers[7].cell[0]);
    // }
    CudaSafeCall(hipDeviceSynchronize());
    // timer::show_duration_since_stamp("Copy to buffer", "ms",
    // "copy_to_buffer");
  }
}

// Explicit instantiation
template class particles_base<ptc_buffer>;
template void particles_base<ptc_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<1, float>>& grid);
template void particles_base<ptc_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<1, double>>& grid);
template void particles_base<ptc_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<2, float>>& grid);
template void particles_base<ptc_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<2, double>>& grid);
template void particles_base<ptc_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<3, float>>& grid);
template void particles_base<ptc_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<3, double>>& grid);

template class particles_base<ph_buffer>;
template void particles_base<ph_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<1, float>>& grid);
template void particles_base<ph_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<1, double>>& grid);
template void particles_base<ph_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<2, float>>& grid);
template void particles_base<ph_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<2, double>>& grid);
template void particles_base<ph_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<3, float>>& grid);
template void particles_base<ph_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<3, double>>& grid);

}  // namespace Aperture
