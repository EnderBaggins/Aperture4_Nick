#include "hip/hip_runtime.h"
#include "core/constant_mem.h"
#include "core/typedefs_and_constants.h"
#include "framework/config.h"
#include "particles_impl.hpp"
#include "utils/for_each_dual.hpp"
#include "utils/kernel_helper.hpp"
#include "utils/range.hpp"
#include "visit_struct/visit_struct.hpp"

#include <thrust/binary_search.h>
#include <thrust/device_ptr.h>
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/replace.h>
#include <thrust/sort.h>

namespace Aperture {

template <typename Conf>
void
compute_target_buffers(const uint32_t* cells, size_t num,
                       buffer<int>& buffer_num, size_t* idx) {
  kernel_launch(
      [num] __device__(auto cells, auto buffer_num, auto index) {
        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        int zone_offset = 0;
        if (Conf::dim == 2)
          zone_offset = 9;
        else if (Conf::dim == 1)
          zone_offset = 12;
        for (auto n : grid_stride_range(0, num)) {
          uint32_t cell = cells[n];
          if (cell == empty_cell) continue;
          auto idx = Conf::idx(cell, ext);
          auto grid_pos = idx.get_pos();
          size_t zone = grid.find_zone(grid_pos) + zone_offset;
          if (zone == 13) continue;
          size_t pos = atomicAdd(&buffer_num[zone], 1);
          // printf("pos is %lu, zone is %lu\n", pos, zone);
          // Zone is less than 32, so we can use 5 bits to represent this. The
          // rest of the bits go to encode the index of this particle in that
          // zone.
          index[n] = ((zone & 0b11111) << (sizeof(size_t) * 8 - 5)) + pos;
        }
      },
      cells, buffer_num.dev_ptr(), idx);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf, typename PtcPtrs>
void
copy_component_to_buffer(PtcPtrs ptc_data, size_t num, size_t* idx,
                         buffer<PtcPtrs>& ptc_buffers) {
  kernel_launch(
      [num] __device__(auto ptc_data, auto index, auto ptc_buffers) {
        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto n : grid_stride_range(0, num)) {
          int bitshift_width = (sizeof(size_t) * 8 - 5);
          int zone_offset = 0;
          if (Conf::dim == 2)
            zone_offset = 9;
          else if (Conf::dim == 1)
            zone_offset = 12;
          // loop through the particle array
          for (auto n : grid_stride_range(0, num)) {
            auto cell = ptc_data.cell[n];
            if (cell == empty_cell) continue;
            size_t i = index[n];
            size_t zone = ((i >> bitshift_width) & 0b11111);
            if (zone == 13 || zone > 27) continue;
            size_t pos = i - (zone << bitshift_width);
            // printf("in copy, pos is %lu, zone is %lu\n", pos, zone);
            // Copy the particle data from ptc_data[n] to ptc_buffers[zone][pos]
            assign_ptc(ptc_buffers[zone - zone_offset], pos, ptc_data, n);
            // printf("pos is %lu, %u, %u\n", pos, ptc_buffers[zone -
            //                                                 zone_offset].cell[pos], ptc_data.cell[n]);
            // Compute particle cell delta
            int dz = (Conf::dim > 2 ? (zone / 9) - 1 : 0);
            int dy = (Conf::dim > 1 ? (zone / 3) % 3 - 1 : 0);
            int dx = zone % 3 - 1;
            auto idx = Conf::idx(cell, ext);
            // int dcell =
            //     -dz * grid.reduced_dim(2) * grid.dims[0] * grid.dims[1] -
            //     dy * grid.reduced_dim(1) * grid.dims[0] -
            //     dx * grid.reduced_dim(0);
            ptc_buffers[zone - zone_offset].cell[pos] =
                idx.dec_z(dz * grid.reduced_dim(2))
                .dec_y(dy * grid.reduced_dim(1))
                .dec_x(dx * grid.reduced_dim(0)).linear;
            // printf("dc is %d, cell is %u, cell after is %u\n", dcell,
            //        ptc_data.cell[n],
            //        ptc_buffers[zone - zone_offset].cell[pos]);
            // Set the particle to empty
            ptc_data.cell[n] = empty_cell;
          }
        }
      },
      ptc_data, idx, ptc_buffers.dev_ptr());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename BufferType>
void
particles_base<BufferType>::rearrange_arrays(const std::string& skip) {
  const uint32_t padding = 100;
  auto ptc = typename BufferType::single_type{};
  for_each_double_with_name(
      m_dev_ptrs, ptc,
      [this, padding, &skip](const char* name, auto& x, auto& u) {
        typedef typename std::remove_reference<decltype(x)>::type x_type;
        auto ptr_index = thrust::device_pointer_cast(m_index.dev_ptr());
        if (std::strcmp(name, skip.c_str()) == 0) return;

        auto x_ptr = thrust::device_pointer_cast(x);
        auto tmp_ptr = thrust::device_pointer_cast(
            reinterpret_cast<x_type>(m_tmp_data.dev_ptr()));
        thrust::gather(ptr_index, ptr_index + m_number, x_ptr, tmp_ptr);
        thrust::copy_n(tmp_ptr, m_number, x_ptr);
        CudaCheckError();
      });
}

template <typename BufferType>
void
particles_base<BufferType>::sort_by_cell_dev(size_t max_cell) {
  if (m_number > 0) {
    // Lazy resize the tmp arrays
    if (m_index.size() != m_size || m_tmp_data.size() != m_size) {
      m_index.resize(m_size);
      m_tmp_data.resize(m_size);
    }

    // Generate particle index array
    auto ptr_cell = thrust::device_pointer_cast(this->cell.dev_ptr());
    auto ptr_idx = thrust::device_pointer_cast(m_index.dev_ptr());
    thrust::counting_iterator<size_t> iter(0);
    thrust::copy_n(iter, m_number, ptr_idx);

    // Sort the index array by key
    thrust::sort_by_key(ptr_cell, ptr_cell + m_number, ptr_idx);
    // hipDeviceSynchronize();
    // Logger::print_debug("Finished sorting");

    // Move the rest of particle array using the new index
    rearrange_arrays("cell");

    // Update the new number of particles
    const int padding = 0;
    m_number = thrust::upper_bound(ptr_cell, ptr_cell + m_number + padding,
                                   empty_cell - 1) -
               ptr_cell;

    // Logger::print_info("Sorting complete, there are {} particles in the pool",
    //                    m_number);
    hipDeviceSynchronize();
    CudaCheckError();
  }
}

template <typename BufferType>
void
particles_base<BufferType>::append_dev(const vec_t<Pos_t, 3>& x,
                                       const vec_t<Scalar, 3>& p, uint32_t cell,
                                       Scalar weight, uint32_t flag) {
  if (m_number == m_size) return;
  kernel_launch(
      {1, 1},
      [x, p, cell, weight, flag] __device__(auto ptrs, size_t pos) {
        ptrs.x1[pos] = x[0];
        ptrs.x2[pos] = x[1];
        ptrs.x3[pos] = x[2];
        ptrs.p1[pos] = p[0];
        ptrs.p2[pos] = p[1];
        ptrs.p3[pos] = p[2];
        ptrs.weight[pos] = weight;
        ptrs.cell[pos] = cell;
        ptrs.flag[pos] = flag;
      },
      m_dev_ptrs, m_number);
  CudaSafeCall(hipDeviceSynchronize());
  m_number += 1;
}

template <typename BufferType>
template <typename Conf>
void
particles_base<BufferType>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Conf>& grid) {
  if (m_number > 0) {
    if (m_index.size() != m_size) m_index.resize(m_size);
    auto ptr_idx = thrust::device_pointer_cast(m_index.dev_ptr());
    thrust::fill_n(ptr_idx, m_number, -1);
    for (int i = 0; i < 27; i++) {
      m_zone_buffer_num[i] = 0;
    }
    m_zone_buffer_num.copy_to_device();
    compute_target_buffers<Conf>(this->cell.dev_ptr(), m_number, m_zone_buffer_num,
                                 m_index.dev_ptr());

    CudaSafeCall(hipDeviceSynchronize());

    int zone_offset = 0;
    if (buffers.size() == 9)
      zone_offset = 9;
    else if (buffers.size() == 3)
      zone_offset = 12;
    for (unsigned int i = 0; i < buffers.size(); i++) {
      // Logger::print_debug("zone {} buffer has {} ptc", i + zone_offset,
      //                     m_zone_buffer_num[i + zone_offset]);
      if (i + zone_offset == 13) continue;
      buffers[i].set_num(m_zone_buffer_num[i + zone_offset]);
    }
    copy_component_to_buffer<Conf>(m_dev_ptrs, m_number, m_index.dev_ptr(),
                                   buf_ptrs);

    CudaSafeCall(hipDeviceSynchronize());
  }
}

// Explicit instantiation
template class particles_base<ptc_buffer>;
template void particles_base<ptc_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<1>>& grid);
template void particles_base<ptc_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<2>>& grid);
template void particles_base<ptc_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<3>>& grid);

template class particles_base<ph_buffer>;
template void particles_base<ph_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<1>>& grid);
template void particles_base<ph_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<2>>& grid);
template void particles_base<ph_buffer>::copy_to_comm_buffers(
    std::vector<self_type>& buffers, buffer<ptrs_type>& buf_ptrs,
    const grid_t<Config<3>>& grid);

}  // namespace Aperture
