#include "hip/hip_runtime.h"
#include "boundary_condition.h"
#include "framework/config.h"
#include "systems/grid_sph.h"
#include "utils/kernel_helper.hpp"

namespace Aperture {

template <typename Conf>
void
boundary_condition<Conf>::init() {
  m_env.get_data("Edelta", &E);
  m_env.get_data("E0", &E0);
  m_env.get_data("Bdelta", &B);
  m_env.get_data("B0", &B0);

  m_env.params().get_value("omega", m_omega_0);
  m_env.params().get_value("omega_t", m_omega_t);
}

template <typename Conf>
void
boundary_condition<Conf>::update(double dt, uint32_t step) {
  auto ext = m_grid.extent();
  typedef typename Conf::idx_t idx_t;
  typedef typename Conf::value_t value_t;

  value_t time = m_env.get_time();
  value_t omega;
  // if (m_omega_t * time < 5000.0)
  if (time < 3.0)
    omega = m_omega_0 * sin(2.0 * M_PI * m_omega_t * time);
  else
    omega = 0.0;
  Logger::print_debug("time is {}, Omega is {}", time, omega);

  kernel_launch([ext, time, omega] __device__ (auto e, auto b, auto e0, auto b0) {
      auto& grid = dev_grid<Conf::dim>();
      for (auto n1 : grid_stride_range(0, grid.dims[1])) {
        value_t theta = grid_sph_t<Conf>::theta(grid.template pos<1>(n1, false));
        value_t theta_s = grid_sph_t<Conf>::theta(grid.template pos<1>(n1, true));

        // For quantities that are not continuous across the surface
        for (int n0 = 0; n0 < grid.skirt[0]; n0++) {
          auto idx = idx_t(index_t<2>(n0, n1), ext);
          e[0][idx] = 0.0;
          b[1][idx] = 0.0;
          b[2][idx] = 0.0;
        }
        // For quantities that are continuous across the surface
        for (int n0 = 0; n0 < grid.skirt[0] + 1; n0++) {
          auto idx = idx_t(index_t<2>(n0, n1), ext);
          value_t r = grid_sph_t<Conf>::radius(grid.template pos<0>(n0, false));
          value_t r_s = grid_sph_t<Conf>::radius(grid.template pos<0>(n0, true));
          b[0][idx] = 0.0;
          e[1][idx] = -omega * sin(theta) * r_s * b0[0][idx];
          e[2][idx] = 0.0;
        }
      }
    }, E->get_ptrs(), B->get_ptrs(), E0->get_ptrs(), B0->get_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
}


template class boundary_condition<Config<2>>;

}
