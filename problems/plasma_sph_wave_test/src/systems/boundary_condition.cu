#include "hip/hip_runtime.h"
#include "boundary_condition.hpp"
#include "framework/config.h"
#include "utils/kernel_helper.hpp"

namespace Aperture {

template <typename Conf>
void
boundary_condition<Conf>::init() {
  m_env.params().get_value("omega", m_omega_0);
  m_env.params().get_value("omega_t", m_omega_t);
}

template <typename Conf>
void
boundary_condition<Conf>::update(double dt, uint32_t step) {
  auto ext = m_grid.extent();
  typedef typename Conf::idx_t idx_t;
  typedef typename Conf::value_t value_t;

  value_t time = m_env.get_time();
  value_t omega;
  // if (m_omega_t * time < 5000.0)
  if (time < 3.0)
    omega = m_omega_0 * sin(2.0 * M_PI * m_omega_t * time);
  else
    omega = 0.0;
  Logger::print_debug("time is {}, Omega is {}", time, omega);

  kernel_launch([ext, time, omega] __device__ (auto e, auto b, auto e0, auto b0) {
      auto& grid = dev_grid<Conf::dim>();
      for (auto n1 : grid_stride_range(0, grid.dims[1])) {
        value_t theta = grid.template pos<1>(n1, false);
        value_t theta_s = grid.template pos<1>(n1, true);

        // For quantities that are not continuous across the surface
        for (int n0 = 0; n0 < grid.skirt[0] + 1; n0++) {
          auto idx = idx_t(index_t<2>(n0, n1), ext);
          e[0][idx] = e0[0][idx];
          b[1][idx] = b0[1][idx];
          b[2][idx] = b0[2][idx];
        }
        // For quantities that are continuous across the surface
        for (int n0 = 0; n0 < grid.skirt[0] + 2; n0++) {
          auto idx = idx_t(index_t<2>(n0, n1), ext);
          value_t r = exp(grid.template pos<0>(n0, false));
          b[0][idx] = b0[0][idx];
          e[1][idx] = e0[1][idx] - omega * sin(theta_s) * r * b0[0][idx];
          e[2][idx] = e0[2][idx];
        }
      }
    }, E->get_ptrs(), B->get_ptrs(), E0->get_ptrs(), B0->get_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
}


template class boundary_condition<Config<2>>;

}
