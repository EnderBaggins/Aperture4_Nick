#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "boundary_condition.h"
#include "core/math.hpp"
#include "framework/config.h"
#include "systems/grid.h"
#include "utils/kernel_helper.hpp"
#include "utils/util_functions.h"

namespace Aperture {

struct wpert_cart_t {
  float tp_start, tp_end, nT, dw0, y_start, y_end, q_e;

  HD_INLINE wpert_cart_t(float tp_s, float tp_e, float nT_, float dw0_, float qe)
      : tp_start(tp_s), tp_end(tp_e), nT(nT_), dw0(dw0_), q_e(qe) {
    y_start = 0.5f;
    y_end = 4.5f;
  }

  HD_INLINE Scalar operator()(Scalar t, Scalar x, Scalar y) {
    if (t >= tp_start && t <= tp_end && y > y_start && y < y_end) {
      Scalar omega =
          dw0 *
          math::sin((t - tp_start) * 2.0f * M_PI * nT / (tp_end - tp_start)) *
          math::sin(M_PI * (y - y_start) / (y_end - y_start));
          // math::sin((t - tp_start) * 2.0f * M_PI * nT / (tp_end - tp_start));
      return omega;
    } else {
      return 0.0;
    }
  }

  HD_INLINE Scalar j_x(Scalar t, Scalar x, Scalar y, Scalar theta) {
    return 0.0;
  }

  HD_INLINE Scalar j_y(Scalar t, Scalar x, Scalar y, Scalar theta) {
    return 0.0;
  }
};

HOST_DEVICE Scalar
pml_sigma(Scalar x, Scalar xh, Scalar pmlscale, Scalar sig0) {
  if (x > xh)
    return sig0 * square((x - xh) / pmlscale);
  else
    return 0.0;
}

template <typename Conf>
void
inject_particles(particle_data_t& ptc, curand_states_t& rand_states,
                 buffer<float>& surface_ne, buffer<float>& surface_np,
                 int num_per_cell, typename Conf::value_t weight,
                 const grid_t<Conf>& grid, const wpert_cart_t& wpert,
                 int multiplicity) {
  surface_ne.assign_dev(0.0f);
  surface_np.assign_dev(0.0f);

  auto ptc_num = ptc.number();
  // First measure surface density
  kernel_launch(
      [ptc_num] __device__(auto ptc, auto surface_ne, auto surface_np) {
        auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();
        for (auto n : grid_stride_range(0, ptc_num)) {
          auto c = ptc.cell[n];
          if (c == empty_cell) continue;

          auto idx = typename Conf::idx_t(c, ext);
          auto pos = idx.get_pos();
          if (pos[0] == grid.guard[0]) {
            auto flag = ptc.flag[n];
            auto sp = get_ptc_type(flag);

            if (sp == 0)
              atomicAdd(&surface_ne[pos[1]],
                        ptc.weight[n] * math::abs(dev_charges[sp]));
            else if (sp == 1)
              atomicAdd(&surface_np[pos[1]],
                        ptc.weight[n] * math::abs(dev_charges[sp]));
          }
        }
      },
      ptc.get_dev_ptrs(), surface_ne.dev_ptr(), surface_np.dev_ptr());
  CudaSafeCall(hipDeviceSynchronize());

  // Then inject particles
  kernel_launch(
      [ptc_num, weight] __device__(auto ptc, auto surface_ne, auto surface_np,
                                   auto num_inj, auto states) {
        auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();
        int inj_n0 = grid.guard[0];
        int id = threadIdx.x + blockIdx.x * blockDim.x;
        cuda_rng_t rng(&states[id]);
        for (auto n1 :
             grid_stride_range(grid.guard[1], grid.dims[1] - grid.guard[1])) {
          size_t offset = ptc_num + n1 * num_inj * 2;
          auto pos = index_t<Conf::dim>(inj_n0, n1);
          auto cell_x2 = grid.template coord<1>(n1, false);
          if (cell_x2 < 0.2 || cell_x2 > 4.8) continue;
          auto idx = typename Conf::idx_t(pos, ext);
          if (std::min(surface_ne[pos[1]], surface_np[pos[1]]) >
              square(1.0f / grid.delta[0]))
            continue;
          for (int i = 0; i < num_inj; i++) {
            float x2 = rng();
            ptc.x1[offset + i * 2] = ptc.x1[offset + i * 2 + 1] = 1.0f;
            ptc.x2[offset + i * 2] = ptc.x2[offset + i * 2 + 1] = x2;
            ptc.x3[offset + i * 2] = ptc.x3[offset + i * 2 + 1] = 0.0f;
            ptc.p1[offset + i * 2] = ptc.p1[offset + i * 2 + 1] = 0.0f;
            ptc.p2[offset + i * 2] = ptc.p2[offset + i * 2 + 1] = 0.0f;
            ptc.p3[offset + i * 2] = ptc.p3[offset + i * 2 + 1] = 0.0f;
            ptc.E[offset + i * 2] = ptc.E[offset + i * 2 + 1] = 1.0f;
            ptc.cell[offset + i * 2] = ptc.cell[offset + i * 2 + 1] =
                idx.linear;
            ptc.weight[offset + i * 2] = ptc.weight[offset + i * 2 + 1] =
                weight;
            ptc.flag[offset + i * 2] = set_ptc_type_flag(0, PtcType::electron);
            ptc.flag[offset + i * 2 + 1] =
                set_ptc_type_flag(0, PtcType::positron);
          }
        }
      },
      ptc.get_dev_ptrs(), surface_ne.dev_ptr(), surface_np.dev_ptr(),
      num_per_cell, rand_states.states());
  CudaSafeCall(hipDeviceSynchronize());

  ptc.add_num(num_per_cell * 2 * grid.dims[1]);
}

template <typename Conf>
boundary_condition<Conf>::boundary_condition(sim_environment& env,
                                             const grid_t<Conf>& grid)
    : system_t(env), m_grid(grid) {
  using multi_array_t = typename Conf::multi_array_t;
  m_env.params().get_value("damping_length", m_damping_length);
  m_env.params().get_value("pmllen", m_pmllen);
  m_env.params().get_value("sigpml", m_sigpml);

  m_prev_E1 = std::make_unique<multi_array_t>(
      extent(m_damping_length, m_grid.dims[1]), MemType::device_only);
  m_prev_E2 = std::make_unique<multi_array_t>(
      extent(m_damping_length, m_grid.dims[1]), MemType::device_only);
  m_prev_E3 = std::make_unique<multi_array_t>(
      extent(m_damping_length, m_grid.dims[1]), MemType::device_only);
  m_prev_B1 = std::make_unique<multi_array_t>(
      extent(m_damping_length, m_grid.dims[1]), MemType::device_only);
  m_prev_B2 = std::make_unique<multi_array_t>(
      extent(m_damping_length, m_grid.dims[1]), MemType::device_only);
  m_prev_B3 = std::make_unique<multi_array_t>(
      extent(m_damping_length, m_grid.dims[1]), MemType::device_only);

  m_prev_E1->assign_dev(0.0f);
  m_prev_E2->assign_dev(0.0f);
  m_prev_E3->assign_dev(0.0f);
  m_prev_B1->assign_dev(0.0f);
  m_prev_B2->assign_dev(0.0f);
  m_prev_B3->assign_dev(0.0f);

  m_prev_E.set_memtype(MemType::host_device);
  m_prev_B.set_memtype(MemType::host_device);
  m_prev_E.resize(3);
  m_prev_B.resize(3);
  m_prev_E[0] = m_prev_E1->dev_ptr();
  m_prev_E[1] = m_prev_E2->dev_ptr();
  m_prev_E[2] = m_prev_E3->dev_ptr();
  m_prev_B[0] = m_prev_B1->dev_ptr();
  m_prev_B[1] = m_prev_B2->dev_ptr();
  m_prev_B[2] = m_prev_B3->dev_ptr();
  m_prev_E.copy_to_device();
  m_prev_B.copy_to_device();
}

template <typename Conf>
void
boundary_condition<Conf>::init() {
  m_env.get_data("Edelta", &E);
  m_env.get_data("E0", &E0);
  m_env.get_data("Bdelta", &B);
  m_env.get_data("B0", &B0);
  m_env.get_data("rand_states", &rand_states);
  m_env.get_data("particles", &ptc);

  m_env.params().get_value("tp_start", m_tp_start);
  m_env.params().get_value("tp_end", m_tp_end);
  m_env.params().get_value("nT", m_nT);
  m_env.params().get_value("dw0", m_dw0);
  m_env.params().get_value("q_e", m_qe);
  m_env.params().get_value("damping_coef", m_damping_coef);
  m_env.params().get_value("muB", m_muB);

  m_surface_ne.set_memtype(MemType::host_device);
  m_surface_ne.resize(m_grid.dims[1]);
  m_surface_np.set_memtype(MemType::host_device);
  m_surface_np.resize(m_grid.dims[1]);
}

template <typename Conf>
void
boundary_condition<Conf>::update(double dt, uint32_t step) {
  typedef typename Conf::idx_t idx_t;
  typedef typename Conf::value_t value_t;

  value_t time = m_env.get_time();
  // wpert_cart_t wpert(m_tp_start, m_tp_end, m_nT, m_dw0, m_qe);

  // Apply twist on the stellar surface
  // kernel_launch(
  //     [time] __device__(auto e, auto b, auto e0, auto b0, auto wpert) {
  //       auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
  //       auto ext = grid.extent();
  //       for (auto n1 : grid_stride_range(0, grid.dims[1])) {
  //         value_t y = grid.template coord<1>(n1, false);
  //         value_t y_s = grid.template coord<1>(n1, true);

  //         // For quantities that are not continuous across the surface
  //         for (int n0 = 0; n0 < grid.guard[0]; n0++) {
  //           auto idx = idx_t(index_t<2>(n0, n1), ext);
  //           value_t x = grid.template coord<0>(n0, false);
  //           value_t omega = wpert(time, x, y_s);
  //           // printf("omega is %f\n", omega);
  //           e[0][idx] = omega * b0[1][idx];
  //           b[1][idx] = 0.0;
  //           b[2][idx] = 0.0;
  //         }
  //         // For quantities that are continuous across the surface
  //         for (int n0 = 0; n0 < grid.guard[0] + 1; n0++) {
  //           auto idx = idx_t(index_t<2>(n0, n1), ext);
  //           value_t x_s = grid.template coord<0>(n0, true);
  //           value_t omega = wpert(time, x_s, y);
  //           b[0][idx] = 0.0;
  //           e[1][idx] = -omega * b0[0][idx];
  //           e[2][idx] = 0.0;
  //         }
  //       }
  //     },
  //     E->get_ptrs(), B->get_ptrs(), E0->get_ptrs(), B0->get_ptrs(), wpert);
  // CudaSafeCall(hipDeviceSynchronize());
  // CudaCheckError();

  // Apply damping boundary condition on the other side
  kernel_launch(
      [] __device__(auto e, auto b, auto prev_e, auto prev_b, auto damping_length,
                    auto damping_coef) {
        auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();
        auto ext_damping = extent(damping_length, grid.dims[1]);
        for (auto n0 : grid_stride_range(0, grid.dims[0])) {
          // auto n0_start = grid.dims[0] - damping_length;
          // auto xh = grid.template coord<0>(n0_start, true);
          // for (int n0 = n0_start; n0 < grid.dims[0]; n0++) {
          //   auto idx = idx_t(index(n0, n1), ext);
          //   auto idx_damping = idx_t(index(n0 - n0_start, n1), ext_damping);
          //   auto x = grid.template coord<0>(n0, false);
          //   auto x_s = grid.template coord<0>(n0, true);
          //   auto sig = pml_sigma(x, xh, pmllen, sigpml);
          //   if (sig > TINY) {
          //     auto exp_sig = math::exp(-sig);
          //     // e[0][idx] = exp_sig * prev_e[0][idx_damping] +
          //     //     (1.0f - exp_sig) / sig * (e[0][idx] - prev_e[0][idx_damping]);
          //     // b[1][idx] = exp_sig * prev_b[1][idx_damping] +
          //     //     (1.0f - exp_sig) / sig * (b[1][idx] - prev_b[1][idx_damping]);
          //     // b[2][idx] = exp_sig * prev_b[2][idx_damping] +
          //     //     (1.0f - exp_sig) / sig * (b[2][idx] - prev_b[2][idx_damping]);
          //     e[0][idx] *= exp_sig;
          //     b[1][idx] *= exp_sig;
          //     b[2][idx] *= exp_sig;
          //   }
          //   // if (n1 == 50 && n0 - n0_start == 64) {
          //   //   printf("sig is %f, b2 is %f, prev_b2 is %f\n", sig,
          //   //          b[2][idx], prev_b[2][idx_damping]);
          //   // }
          //   auto sig_s = pml_sigma(x_s, xh, pmllen, sigpml);
          //   if (sig_s > TINY) {
          //     auto exp_sig = math::exp(-sig_s);
          //     // e[1][idx] = exp_sig * prev_e[1][idx_damping] +
          //     //     (1.0f - exp_sig) / sig_s * (e[1][idx] - prev_e[1][idx_damping]);
          //     // e[2][idx] = exp_sig * prev_e[2][idx_damping] +
          //     //     (1.0f - exp_sig) / sig_s * (e[2][idx] - prev_e[2][idx_damping]);
          //     // b[0][idx] = exp_sig * prev_b[0][idx_damping] +
          //     //     (1.0f - exp_sig) / sig_s * (b[0][idx] - prev_b[0][idx_damping]);
          //     b[0][idx] *= exp_sig;
          //     e[1][idx] *= exp_sig;
          //     e[2][idx] *= exp_sig;
          //   }
          // }
          for (int i = 0; i < damping_length; i++) {
            int n1 = i;
            auto idx = idx_t(index_t<2>(n0, n1), ext);
            // value_t lambda =
            //     1.0f - damping_coef * cube((value_t)i / (damping_length - 1));
            value_t lambda = 0.0f;
            e[0][idx] *= lambda;
            // e[1][idx] *= lambda;
            // e[2][idx] *= lambda;
            // b[1][idx] *= lambda;
            // b[2][idx] *= lambda;
          }
        }
      },
      E->get_ptrs(), B->get_ptrs(), m_prev_E.dev_ptr(), m_prev_B.dev_ptr(),
      m_damping_length, m_damping_coef);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template class boundary_condition<Config<2>>;

}  // namespace Aperture
