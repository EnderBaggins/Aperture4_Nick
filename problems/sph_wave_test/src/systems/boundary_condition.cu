#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "boundary_condition.h"
#include "framework/config.h"
#include "systems/grid_sph.hpp"
#include "utils/kernel_helper.hpp"

namespace Aperture {

template <typename Conf>
void
boundary_condition<Conf>::init() {
  sim_env().get_data("Edelta", &E);
  sim_env().get_data("E0", &E0);
  sim_env().get_data("Bdelta", &B);
  sim_env().get_data("B0", &B0);

  sim_env().params().get_value("E0", m_E0);
  sim_env().params().get_value("omega_t", m_omega_t);
  sim_env().params().get_value("Bp", m_Bp);
}

template <typename Conf>
void
boundary_condition<Conf>::update(double dt, uint32_t step) {
  auto ext = m_grid.extent();
  typedef typename Conf::idx_t idx_t;
  typedef typename Conf::value_t value_t;

  value_t time = sim_env().get_time();
  value_t Bp = m_Bp;
  value_t omega;
  // if (m_omega_t * time < 5000.0)
  value_t phase = time * m_omega_t;
  if (phase < 8.0)
    omega = m_E0 * sin(2.0 * M_PI * phase);
  else
    omega = 0.0;
  Logger::print_debug("time is {}, Omega is {}", time, omega);

  kernel_launch([ext, time, omega, Bp] __device__ (auto e, auto b, auto e0, auto b0) {
      auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
      for (auto n1 : grid_stride_range(0, grid.dims[1])) {
        value_t theta = grid_sph_t<Conf>::theta(grid.template pos<1>(n1, false));
        value_t theta_s = grid_sph_t<Conf>::theta(grid.template pos<1>(n1, true));

        // For quantities that are not continuous across the surface
        for (int n0 = 0; n0 < grid.guard[0] + 1; n0++) {
          auto idx = idx_t(index_t<2>(n0, n1), ext);
          e[0][idx] = 0.0;
          b[1][idx] = 0.0;
          b[2][idx] = 0.0;
        }
        // For quantities that are continuous across the surface
        for (int n0 = 0; n0 < grid.guard[0] + 1; n0++) {
          auto idx = idx_t(index_t<2>(n0, n1), ext);
          value_t r = grid_sph_t<Conf>::radius(grid.template pos<0>(n0, false));
          value_t r_s = grid_sph_t<Conf>::radius(grid.template pos<0>(n0, true));
          b[0][idx] = 0.0;
          e[1][idx] = 0.0;
          // if (theta_s > 0.7 && theta_s < 1.2)
          // e[2][idx] = -omega * sin(theta_s) * r_s * b0[0][idx];
          e[2][idx] = -omega * sin(theta_s) * r_s * Bp;
          // else
          //   e[2][idx] = 0.0;
          // e[2][idx] = 0.0;
        }
      }
    }, E->get_ptrs(), B->get_ptrs(), E0->get_ptrs(), B0->get_ptrs());
  CudaSafeCall(hipDeviceSynchronize());
}


template class boundary_condition<Config<2>>;

}
