#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "core/math.hpp"
#include "data/curand_states.h"
#include "data/fields.h"
#include "data/particle_data.h"
#include "framework/config.h"
#include "framework/environment.h"
#include "utils/kernel_helper.hpp"
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

namespace {

using namespace Aperture;

struct alfven_wave_solution {
  Scalar sinth = 0.1;
  Scalar lambda_x = 1.0;
  Scalar y0 = 1.0;
  Scalar x0 = 0.0;
  Scalar delta_y = 1.0;
  Scalar B0 = 5000;

  Scalar costh;
  Scalar lambda;
  Scalar delta_eta;
  Scalar eta0;

  HD_INLINE Scalar xi(Scalar x, Scalar y) const { return x * sinth + y * costh; }

  HD_INLINE Scalar eta(Scalar x, Scalar y) const { return x * costh - y * sinth; }

  HOST_DEVICE alfven_wave_solution(Scalar sinth_, Scalar lambda_x_, Scalar x0_, Scalar y0_,
                                   Scalar delta_y_, Scalar B0_)
      : sinth(sinth_),
        lambda_x(lambda_x_),
        x0(x0_),
        y0(y0_),
        delta_y(delta_y_),
        B0(B0_) {
    costh = math::sqrt(1.0f - sinth * sinth);
    lambda = lambda_x / sinth;
    delta_eta = delta_y * sinth;
    eta0 = eta(0.0, y0);
  }

  HD_INLINE Scalar wave_arg(Scalar t, Scalar x, Scalar y) const {
    return 2.0 * M_PI *
           (xi(x - x0, y) - t + eta(x - x0, y) * costh / sinth) / lambda;
  }

  HD_INLINE Scalar wave_arg_clamped(Scalar t, Scalar x, Scalar y) const {
    return 2.0 * M_PI *
           clamp<Scalar>((xi(x - x0, y) - t + eta(x - x0, y) * costh / sinth) / lambda,
                         0.0, 2.0);
  }

  HD_INLINE Scalar width_arg(Scalar x, Scalar y) const {
    return (eta(x - x0, y) - eta0) / delta_eta;
  }

  HD_INLINE Scalar width_arg_clamped(Scalar x, Scalar y) const {
    return clamp<Scalar>((eta(x - x0, y) - eta0) / delta_eta, 0.0, 1.0);
  }

  HD_INLINE Scalar width_prof(Scalar w) const { return square(math::sin(M_PI * w)); }

  HD_INLINE Scalar d_width(Scalar w) const {
    return 2.0 * M_PI * math::sin(M_PI * w) * math::cos(M_PI * w) / delta_eta;
  }

  HD_INLINE Scalar wave_profile(Scalar x) const {
    return math::sin(x) * square(math::sin(0.25 * x));
  }

  HD_INLINE Scalar d_wave_profile(Scalar x) const {
    return 2.0 * M_PI * (math::cos(x) * square(math::sin(0.25 * x))
                         + 0.5 * math::sin(x) * math::sin(0.25 * x) * math::cos(0.25 * x));
  }

  HD_INLINE Scalar Bz(Scalar t, Scalar x, Scalar y) const {
    return B0 * wave_profile(wave_arg_clamped(t, x, y));
  }

  HD_INLINE Scalar Ex(Scalar t, Scalar x, Scalar y) const {
    return costh * (-Bz(t, x, y));
  }

  HD_INLINE Scalar Ey(Scalar t, Scalar x, Scalar y) const {
    return sinth * (Bz(t, x, y));
  }

  HD_INLINE Scalar Jx(Scalar t, Scalar x, Scalar y) const {
    return -B0 * sinth *
           (d_wave_profile(wave_arg_clamped(t, x, y)) * costh / lambda /
                sinth);
  }

  HD_INLINE Scalar Jy(Scalar t, Scalar x, Scalar y) const {
    return -B0 * costh *
           (d_wave_profile(wave_arg_clamped(t, x, y)) * costh / lambda /
                sinth);
  }

  HD_INLINE Scalar Rho(Scalar t, Scalar x, Scalar y) const {
    return -B0 * (d_wave_profile(wave_arg_clamped(t, x, y)) * costh / lambda /
                sinth);
  }
};

template <typename Conf>
void
compute_ptc_per_cell(alfven_wave_solution& wave, multi_array<int, Conf::dim> &num_per_cell,
                     Scalar q_e, int mult, int mult_wave) {
  // num_per_cell.assign_dev(2 * mult);
  kernel_launch([q_e, mult, mult_wave, wave] __device__(auto num_per_cell) {
      // q_e *= 10.0;
      auto &grid = dev_grid<Conf::dim, typename Conf::value_t>();
      auto ext = grid.extent();

      for (auto n : grid_stride_range(0, ext.size())) {
        auto idx = Conf::idx(n, ext);
        auto pos = idx.get_pos();
        if (grid.is_in_bound(pos)) {
          num_per_cell[idx] = 2 * mult;
          Scalar x = grid.template pos<0>(pos, 0.0f);
          Scalar y = grid.template pos<1>(pos, 0.0f);
          auto wave_arg = wave.wave_arg(0.0f, x, y);

          if (wave_arg > 0.0f && wave_arg < 4.0f * M_PI) {
            auto rho = wave.Rho(0.0f, x, y);
            int num = floor(math::abs(rho) / q_e);

            // atomicAdd(&num_per_cell[idx], num * (mult_wave * 2 + 1));
            num_per_cell[idx] += num * (mult_wave * 2 + 1);
          }
        }
      }
    }, num_per_cell.dev_ndptr());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

}

namespace Aperture {

template <typename Conf>
void
initial_condition_wave(sim_environment &env, vector_field<Conf> &B,
                       vector_field<Conf> &E, vector_field<Conf> &B0,
                       particle_data_t &ptc, curand_states_t &states, int mult,
                       Scalar weight) {
  Scalar weight_enhance_factor = 1.0f;
  Scalar sinth = env.params().get_as<double>("muB", 0.1);
  Scalar Bp = env.params().get_as<double>("Bp", 5000.0);
  Scalar q_e = env.params().get_as<double>("q_e", 1.0);
  q_e *= weight_enhance_factor;
  Scalar Bwave_factor = env.params().get_as<double>("waveB", 0.1);
  Scalar Bwave = Bwave_factor * Bp;
  int mult_wave = 1;

  alfven_wave_solution wave(sinth, 1.0, 0.05, 4.0, 2.0, Bwave);

  B0.set_values(
      0, [Bp, sinth](Scalar x, Scalar y, Scalar z) { return Bp * sinth; });
  B0.set_values(1, [Bp, sinth](Scalar x, Scalar y, Scalar z) {
    return Bp * math::sqrt(1.0 - sinth * sinth);
  });
  B.set_values(
      2, [wave](Scalar x, Scalar y, Scalar z) { return wave.Bz(0.0, x, y); });
  E.set_values(
      0, [wave](Scalar x, Scalar y, Scalar z) { return wave.Ex(0.0, x, y); });
  E.set_values(
      1, [wave](Scalar x, Scalar y, Scalar z) { return wave.Ey(0.0, x, y); });

  auto num = ptc.number();

  // Compute injection number per cell
  auto ext = B.grid().extent();
  multi_array<int, Conf::dim> num_per_cell(ext, MemType::host_device);
  multi_array<int, Conf::dim> cum_num_per_cell(ext, MemType::host_device);

  num_per_cell.assign_dev(0);
  cum_num_per_cell.assign_dev(0);

  compute_ptc_per_cell<Conf>(wave, num_per_cell, q_e, mult, mult_wave);

  thrust::device_ptr<int> p_num_per_cell(num_per_cell.dev_ptr());
  thrust::device_ptr<int> p_cum_num_per_cell(cum_num_per_cell.dev_ptr());

  thrust::exclusive_scan(p_num_per_cell, p_num_per_cell + ext.size(),
                         p_cum_num_per_cell);
  CudaCheckError();
  num_per_cell.copy_to_host();
  cum_num_per_cell.copy_to_host();
  int new_particles = (cum_num_per_cell[ext.size() - 1] + num_per_cell[ext.size() - 1]);
  Logger::print_info("Initializing {} particles", new_particles);

  // Actually inject particles
  // kernel_launch(
  //     [num, mult, q_e, weight, wave] __device__(auto ptc, auto states, auto num_per_cell,
  //                                               auto cum_num_per_cell) {
  //       auto &grid = dev_grid<Conf::dim, typename Conf::value_t>();
  //       auto ext = grid.extent();
  //       int id = threadIdx.x + blockIdx.x * blockDim.x;
  //       cuda_rng_t rng(&states[id]);
  //       for (auto cell : grid_stride_range(0, ext.size())) {
  //         auto idx = Conf::idx(cell, ext);
  //         auto pos = idx.get_pos();
  //         for (int i = 0; i < num_per_cell[cell]; i++) {
  //           int offset = num + cum_num_per_cell[cell] * 2 + i * 2;
  //           auto x1 = rng();
  //           auto x2 = rng();
  //           ptc.x1[offset] = ptc.x1[offset + 1] = x1;
  //           ptc.x2[offset] = ptc.x2[offset + 1] = x2;
  //           ptc.x3[offset] = ptc.x3[offset + 1] = 0.0f;
  //           ptc.cell[offset] = ptc.cell[offset + 1] = cell;
  //           ptc.weight[offset] = ptc.weight[offset + 1] = weight;
  //           ptc.flag[offset] = set_ptc_type_flag(0, PtcType::electron);
  //           ptc.flag[offset + 1] = set_ptc_type_flag(0, PtcType::positron);

  //           auto x = grid.template pos<0>(pos[0], x1);
  //           auto y = grid.template pos<1>(pos[1], x2);
  //           auto jx = wave.Jx(0.0f, x, y);
  //           auto jy = wave.Jy(0.0f, x, y);
  //           auto j = math::sqrt(jx * jx + jy * jy);
  //           auto v = j / (q_e * mult);
  //           auto gamma = 1.0f / math::sqrt(1.0f - v * v);
  //           ptc.p1[offset] = -jx * gamma / (q_e * mult);
  //           ptc.p2[offset] = -jy * gamma / (q_e * mult);
  //           ptc.p3[offset] = 0.0f;
  //           ptc.E[offset] = gamma;
  //           ptc.p1[offset + 1] = 0.0f;
  //           ptc.p2[offset + 1] = 0.0f;
  //           ptc.p3[offset + 1] = 0.0f;
  //           ptc.E[offset + 1] = 1.0f;
  //         }
  //       }
  //     }, ptc.dev_ptrs(), states.states(), num_per_cell.dev_ndptr_const(),
  //     cum_num_per_cell.dev_ndptr_const());
  // CudaSafeCall(hipDeviceSynchronize());
  // ptc.set_num(num + new_pairs);

  // Initialize the particles
  // num = ptc.number();
  // kernel_launch(
  //     [mult, mult_wave, num, q_e, wave, Bp, weight_enhance_factor]
  //     __device__(auto ptc, auto states, auto w, auto num_per_cell, auto cum_num_per_cell) {
  //       auto &grid = dev_grid<Conf::dim, typename Conf::value_t>();
  //       auto ext = grid.extent();
  //       int id = threadIdx.x + blockIdx.x * blockDim.x;
  //       cuda_rng_t rng(&states[id]);
  //       for (auto cell : grid_stride_range(0, ext.size())) {
  //         auto idx = Conf::idx(cell, ext);
  //         auto pos = idx.get_pos();
  //         // auto idx_row = idx_row_major_t<Conf::dim>(pos, ext);
  //         if (grid.is_in_bound(pos)) {
  //           for (int i = 0; i < num_per_cell[idx]; i++) {
  //             uint32_t offset = num + cum_num_per_cell[idx] + i;
  //             // uint32_t offset = num + idx_row.linear * mult * 2 + i * 2;
  //             ptc.x1[offset] = 0.1f * rng();
  //             ptc.x2[offset] = 0.1f * rng();
  //             ptc.x3[offset] = 0.0f;

  //             ptc.cell[offset] = cell;
  //             Scalar x = grid.template pos<0>(pos[0], ptc.x1[offset]);

  //             if (i < mult * 2) {
  //               if (x < 0.4 * grid.sizes[0]) {
  //                 // Scalar weight = w * (1.0f + 29.0f * (1.0f - x / grid.sizes[0]));
  //                 Scalar weight = w;
  //                 // if (x > 0.4 * grid.sizes[0]) weight *= 0.02;
  //                 ptc.p1[offset] = 0.0f;
  //                 ptc.p2[offset] = 0.0f;
  //                 ptc.p3[offset] = 0.0f;
  //                 ptc.E[offset] = 1.0f;
  //                 ptc.weight[offset] = weight;
  //                 ptc.flag[offset] = set_ptc_type_flag(flag_or(PtcFlag::primary),
  //                                                      ((i % 2 == 0) ? PtcType::electron : PtcType::positron));
  //               }
  //             } else {
  //               Scalar x = grid.template pos<0>(pos, 0.0f);
  //               Scalar y = grid.template pos<1>(pos, 0.0f);
  //               // auto width_arg = wave.width_arg(x, y);
  //               auto wave_arg = wave.wave_arg(0.0f, x, y);
  //               auto rho = wave.Rho(0.0f, x, y);
  //               int num = floor(math::abs(rho) / q_e);

  //               Scalar v;
  //               if (i < mult * 2 + mult_wave * num) {
  //                 v = 0.0f;
  //               } else {
  //                 v = 1.0f / (mult_wave + 1);
  //               }
  //               auto v_d = wave.Bz(0.0f, x, y) / math::sqrt(square(wave.Bz(0.0f, x, y)) + Bp * Bp);
  //               auto gamma = 1.0f / math::sqrt(1.0f - v * v - v_d * v_d);
  //               ptc.p1[offset] = v * wave.sinth * gamma;
  //               ptc.p2[offset] = v * wave.costh * gamma;
  //               ptc.p3[offset] = v_d * gamma;
  //               ptc.E[offset] = gamma;
  //               ptc.weight[offset] = weight_enhance_factor * math::abs(rho) / num / q_e;
  //               if (i < mult * 2 + mult_wave * num) {
  //                 ptc.flag[offset] = set_ptc_type_flag(flag_or(PtcFlag::primary, PtcFlag::initial),
  //                                                      ((rho < 0.0f) ? PtcType::positron : PtcType::electron));
  //               } else {
  //                 ptc.flag[offset] = set_ptc_type_flag(flag_or(PtcFlag::primary, PtcFlag::initial),
  //                                                      ((rho < 0.0f) ? PtcType::electron : PtcType::positron));
  //               }
  //             }


  //             // Scalar weight = w * cube(
  //             //     math::abs(0.5f * grid.sizes[0] - x) * 2.0f / grid.sizes[0]);
  //             // Scalar weight = std::max(w * cube(
  //             //     (0.5f * grid.sizes[0] - x) * 2.0f / grid.sizes[0]), 0.02f * w);
  //             // Scalar weight = (pos[0] > grid.dims[0] * 0.4f ? 0.020 * w : w);
  //             // Scalar weight = w;
  //             // if (wave_arg > 0.0f && wave_arg < 2.0f * M_PI) {
  //             //   auto rho = wave.Rho(0.0f, x, y);
  //             //   // auto jx = rho * wave.sinth;
  //             //   // auto jy = rho * wave.costh;
  //             //   // auto jy = wave.Jy(0.0f, x, y);
  //             //   // auto j = math::sqrt(jx * jx + jy * jy);
  //             //   // auto v = j / (2.0f * q_e * mult);
  //             //   // auto v = math::abs(rho) / (2.0f * q_e * mult * weight);
  //             //   auto v = math::abs(rho) / (math::abs(rho) * 3.0f / (q_e * mult)) / (q_e * mult);
  //             //   // auto v3 = math::abs(rho) * 3.0f / (2.0f * q_e * mult * weight);
  //             //   auto v_d = wave.Bz(0.0f, x, y) / math::sqrt(square(wave.Bz(0.0f, x, y)) + Bp * Bp);
  //             //   // auto v_d = 0.0f;
  //             //   auto gamma = 1.0f / math::sqrt(1.0f - v * v - v_d * v_d);
  //             //   // auto gamma3 = 1.0f / math::sqrt(1.0f - v3 * v3 - v_d * v_d);
  //             //   // auto sgn_jy = sgn(jy);

  //             //   if (rho > 0.0f) {
  //             //     ptc.p1[offset] = 0.0f;
  //             //     ptc.p1[offset + 1] = v * wave.sinth * gamma;
  //             //     ptc.p2[offset] = 0.0f;
  //             //     ptc.p2[offset + 1] = v * wave.costh * gamma;
  //             //     ptc.p3[offset] = v_d / math::sqrt(1.0f - v_d * v_d);
  //             //     ptc.p3[offset + 1] = v_d * gamma;
  //             //     ptc.E[offset] = 1.0f / math::sqrt(1.0f - v_d * v_d);
  //             //     ptc.E[offset + 1] = gamma;
  //             //     ptc.weight[offset] = weight + math::abs(rho) * 2.0f / (q_e * mult);
  //             //     ptc.weight[offset + 1] = weight + math::abs(rho) * 3.0f / (q_e * mult);
  //             //     // ptc.weight[offset] = weight;
  //             //     // ptc.weight[offset + 1] = weight;
  //             //   } else {
  //             //     ptc.p1[offset] = v * wave.sinth * gamma;
  //             //     ptc.p1[offset + 1] = 0.0f;
  //             //     ptc.p2[offset] = v * wave.costh * gamma;
  //             //     ptc.p2[offset + 1] = 0.0f;
  //             //     ptc.p3[offset] = v_d * gamma;
  //             //     ptc.p3[offset + 1] = v_d / math::sqrt(1.0f - v_d * v_d);
  //             //     ptc.E[offset] = gamma;
  //             //     ptc.E[offset + 1] = 1.0f / math::sqrt(1.0f - v_d * v_d);
  //             //     ptc.weight[offset] = weight + math::abs(rho) * 3.0f / (q_e * mult);
  //             //     ptc.weight[offset + 1] = weight + math::abs(rho) * 2.0f / (q_e * mult);
  //             //     // ptc.weight[offset] = weight;
  //             //     // ptc.weight[offset + 1] = weight;
  //             //   }
  //             //   // ptc.p1[offset] = -jx * gamma / (2.0f * q_e * mult * weight);
  //             //   // ptc.p1[offset + 1] = jx * gamma / (2.0f * q_e * mult * weight);
  //             //   // ptc.p2[offset] = -jy * gamma / (2.0f * q_e * mult * weight);
  //             //   // ptc.p2[offset + 1] = jy * gamma / (2.0f * q_e * mult * weight);
  //             //   // ptc.p3[offset] = v_d * gamma;
  //             //   // ptc.p3[offset + 1] = v_d * gamma;
  //             //   // ptc.E[offset] = gamma;
  //             //   // ptc.E[offset + 1] = gamma;
  //             //   // ptc.weight[offset] = ptc.weight[offset + 1] = weight;
  //             // } else {
  //             //   ptc.p1[offset] = ptc.p1[offset + 1] = 0.0f;
  //             //   ptc.p2[offset] = ptc.p2[offset + 1] = 0.0f;
  //             //   ptc.p3[offset] = ptc.p3[offset + 1] = 0.0f;
  //             //   ptc.E[offset] = ptc.E[offset + 1] = 1.0f;
  //             //   ptc.weight[offset] = ptc.weight[offset + 1] = weight;
  //             // }

  //             // ptc.cell[offset] = ptc.cell[offset + 1] = idx.linear;

  //             // // Scalar x = grid.template pos<0>(pos[0], ptc.x1[offset]);
  //             // ptc.flag[offset] = set_ptc_type_flag(flag_or(PtcFlag::primary),
  //             //                                      PtcType::electron);
  //             // ptc.flag[offset + 1] = set_ptc_type_flag(
  //             //     flag_or(PtcFlag::primary), PtcType::positron);
  //           }
  //         }
  //       }
  //     },
  //     ptc.dev_ptrs(), states.states(), weight, num_per_cell.dev_ndptr(), cum_num_per_cell.dev_ndptr());
  // CudaSafeCall(hipDeviceSynchronize());
  // ptc.set_num(num + new_particles);
}

template void initial_condition_wave<Config<2>>(
    sim_environment &env, vector_field<Config<2>> &B,
    vector_field<Config<2>> &E, vector_field<Config<2>> &B0,
    particle_data_t &ptc, curand_states_t &states, int mult, Scalar weight);

} // namespace Aperture
