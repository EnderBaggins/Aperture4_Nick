#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "core/math.hpp"
#include "data/curand_states.h"
#include "data/fields.h"
#include "data/particle_data.h"
#include "framework/config.h"
#include "framework/environment.h"
#include "utils/kernel_helper.hpp"
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

namespace {

using namespace Aperture;

template <typename value_t>
struct alfven_wave_solution {
  value_t sinth = 0.1;
  value_t lambda_x = 1.0;
  value_t y0 = 1.0;
  value_t x0 = 0.0;
  value_t delta_y = 1.0;
  value_t B0 = 5000;

  value_t costh;
  value_t lambda;
  value_t delta_eta;
  value_t eta0;
  value_t length = 4.0f;
  value_t smooth_width = 0.1f;

  HD_INLINE value_t xi(value_t x, value_t y) const {
    return x * sinth + y * costh;
  }

  HD_INLINE value_t eta(value_t x, value_t y) const {
    return x * costh - y * sinth;
  }

  HOST_DEVICE alfven_wave_solution(value_t sinth_, value_t lambda_x_,
                                   value_t x0_, value_t y0_, value_t delta_y_,
                                   value_t B0_)
      : sinth(sinth_),
        lambda_x(lambda_x_),
        x0(x0_),
        y0(y0_),
        delta_y(delta_y_),
        B0(B0_) {
    costh = math::sqrt(1.0f - sinth * sinth);
    lambda = lambda_x / sinth;
    delta_eta = delta_y * sinth;
    eta0 = eta(0.0, y0);
  }

  HD_INLINE value_t wave_arg(value_t t, value_t x, value_t y) const {
    return 2.0 * M_PI * (xi(x - x0, y) - t + eta(x - x0, y) * costh / sinth) /
           lambda;
  }

  HD_INLINE value_t wave_arg_clamped(value_t t, value_t x, value_t y) const {
    return 2.0 * M_PI *
           clamp<value_t>(
               (xi(x - x0, y) - t + eta(x - x0, y) * costh / sinth) / lambda,
               0.0f, length);
  }

  // HD_INLINE value_t width_arg(value_t x, value_t y) const {
  //   return (eta(x - x0, y) - eta0) / delta_eta;
  // }

  // HD_INLINE value_t width_arg_clamped(value_t x, value_t y) const {
  //   return clamp<value_t>((eta(x - x0, y) - eta0) / delta_eta, 0.0, 1.0);
  // }

  // HD_INLINE value_t width_prof(value_t w) const { return
  // square(math::sin(M_PI * w)); }

  // HD_INLINE value_t d_width(value_t w) const {
  //   return 2.0 * M_PI * math::sin(M_PI * w) * math::cos(M_PI * w) /
  //   delta_eta;
  // }

  HD_INLINE value_t wave_profile(value_t x) const {
    // Convert x into a number between 0 and 1
    value_t arg = clamp<value_t>(x / (2.0 * M_PI) / length, 0.0f, 1.0f);
    value_t prof = 0.0f;
    if (arg < smooth_width) {
      prof = square(math::sin(arg * M_PI / (smooth_width * 2.0f)));
    } else if (arg > (1.0f - smooth_width)) {
      prof = square(
          math::sin((arg - 1.0f + smooth_width) * M_PI / (smooth_width * 2.0f) +
                    0.5f * M_PI));
    } else {
      prof = 1.0f;
    }
    return math::sin(x) * prof;
    // return math::sin(x) * square(math::sin(0.5 * x / length));
  }

  HD_INLINE value_t d_wave_profile(value_t x) const {
    // Convert x into a number between 0 and 1
    value_t norm_x = clamp<value_t>(x / (2.0 * M_PI) / length, 0.0f, 1.0f);
    if (norm_x < smooth_width) {
      value_t arg = norm_x * M_PI / (smooth_width * 2.0f);
      return 2.0 * M_PI *
             (math::cos(x) * square(math::sin(arg)) +
              math::sin(x) * math::sin(arg) * math::cos(arg) /
                  (length * smooth_width * 2.0));
    } else if (norm_x > (1.0f - smooth_width)) {
      value_t arg = (norm_x - 1.0f + smooth_width) * M_PI / (smooth_width * 2.0f) +
                    0.5f * M_PI;
      return 2.0 * M_PI *
             (math::cos(x) * square(math::sin(arg)) +
              math::sin(x) * math::sin(arg) * math::cos(arg) /
                  (length * smooth_width * 2.0));
    } else {
      return 2.0 * M_PI * math::cos(x);
    }
    // return math::sin(x) * prof;
    // return 2.0 * M_PI *
    //        (math::cos(x) * square(math::sin(0.5 * x / length)) +
    //         math::sin(x) * math::sin(0.5 * x / length) *
    //             math::cos(0.5 * x / length) / length);
  }

  HD_INLINE value_t Bz(value_t t, value_t x, value_t y) const {
    return B0 * wave_profile(wave_arg_clamped(t, x, y));
  }

  HD_INLINE value_t Ex(value_t t, value_t x, value_t y) const {
    return costh * (-Bz(t, x, y));
  }

  HD_INLINE value_t Ey(value_t t, value_t x, value_t y) const {
    return sinth * (Bz(t, x, y));
  }

  HD_INLINE value_t Jx(value_t t, value_t x, value_t y) const {
    return -B0 * sinth *
           (d_wave_profile(wave_arg_clamped(t, x, y)) * costh / lambda / sinth);
  }

  HD_INLINE value_t Jy(value_t t, value_t x, value_t y) const {
    return -B0 * costh *
           (d_wave_profile(wave_arg_clamped(t, x, y)) * costh / lambda / sinth);
  }

  HD_INLINE value_t Rho(value_t t, value_t x, value_t y) const {
    return -B0 *
           (d_wave_profile(wave_arg_clamped(t, x, y)) * costh / lambda / sinth);
  }
};

template <typename Conf>
void
compute_ptc_per_cell(alfven_wave_solution<typename Conf::value_t> &wave,
                     multi_array<int, Conf::dim> &num_per_cell, Scalar q_e,
                     int mult, int mult_wave) {
  // num_per_cell.assign_dev(2 * mult);
  kernel_launch(
      [q_e, mult, mult_wave, wave] __device__(auto num_per_cell) {
        // q_e *= 10.0;
        auto &grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();

        for (auto n : grid_stride_range(0, ext.size())) {
          auto idx = Conf::idx(n, ext);
          auto pos = idx.get_pos();
          if (grid.is_in_bound(pos)) {
            num_per_cell[idx] = 2 * mult;
            Scalar x = grid.template pos<0>(pos, 0.0f);
            Scalar y = grid.template pos<1>(pos, 0.0f);
            auto wave_arg = wave.wave_arg(0.0f, x, y);

            if (wave_arg > 0.0f && wave_arg < 2.0f * M_PI * wave.length) {
              auto rho = wave.Rho(0.0f, x, y);
              int num = floor(math::abs(rho) / q_e);

              // atomicAdd(&num_per_cell[idx], num * (mult_wave * 2 + 1));
              num_per_cell[idx] += num * (mult_wave * 2 + 1);
            }
          }
        }
      },
      num_per_cell.dev_ndptr());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

}  // namespace

namespace Aperture {

template <typename Conf>
void
initial_condition_wave(vector_field<Conf> &B,
                       vector_field<Conf> &E, vector_field<Conf> &B0,
                       particle_data_t &ptc, curand_states_t &states, int mult,
                       Scalar weight) {
  Scalar weight_enhance_factor = 1.0f;
  Scalar sinth = sim_env().params().get_as<double>("muB", 0.1);
  Scalar Bp = sim_env().params().get_as<double>("Bp", 5000.0);
  Scalar q_e = sim_env().params().get_as<double>("q_e", 1.0);
  q_e *= weight_enhance_factor;
  Scalar Bwave_factor = sim_env().params().get_as<double>("waveB", 0.1);
  Scalar Bwave = Bwave_factor * Bp;
  int mult_wave = 1;

  alfven_wave_solution<Scalar> wave(sinth, 1.0, 0.05, 4.0, 2.0, Bwave);

  B0.set_values(
      0, [Bp, sinth](Scalar x, Scalar y, Scalar z) { return Bp * sinth; });
  B0.set_values(1, [Bp, sinth](Scalar x, Scalar y, Scalar z) {
    return Bp * math::sqrt(1.0 - sinth * sinth);
  });
  B.set_values(
      2, [wave](Scalar x, Scalar y, Scalar z) { return wave.Bz(0.0, x, y); });
  E.set_values(
      0, [wave](Scalar x, Scalar y, Scalar z) { return wave.Ex(0.0, x, y); });
  E.set_values(
      1, [wave](Scalar x, Scalar y, Scalar z) { return wave.Ey(0.0, x, y); });

  auto num = ptc.number();

  // Compute injection number per cell
  auto ext = B.grid().extent();
  multi_array<int, Conf::dim> num_per_cell(ext, MemType::host_device);
  multi_array<int, Conf::dim> cum_num_per_cell(ext, MemType::host_device);

  num_per_cell.assign_dev(0);
  cum_num_per_cell.assign_dev(0);

  compute_ptc_per_cell<Conf>(wave, num_per_cell, q_e, mult, mult_wave);

  thrust::device_ptr<int> p_num_per_cell(num_per_cell.dev_ptr());
  thrust::device_ptr<int> p_cum_num_per_cell(cum_num_per_cell.dev_ptr());

  thrust::exclusive_scan(p_num_per_cell, p_num_per_cell + ext.size(),
                         p_cum_num_per_cell);
  CudaCheckError();
  num_per_cell.copy_to_host();
  cum_num_per_cell.copy_to_host();
  int new_particles =
      (cum_num_per_cell[ext.size() - 1] + num_per_cell[ext.size() - 1]);
  Logger::print_info("Initializing {} particles", new_particles);

  // Initialize the particles
  num = ptc.number();
  kernel_launch(
      [mult, mult_wave, num, q_e, wave, Bp, weight_enhance_factor] __device__(
          auto ptc, auto states, auto w, auto num_per_cell,
          auto cum_num_per_cell) {
        auto &grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();
        int id = threadIdx.x + blockIdx.x * blockDim.x;
        cuda_rng_t rng(&states[id]);
        for (auto cell : grid_stride_range(0, ext.size())) {
          auto idx = Conf::idx(cell, ext);
          auto pos = idx.get_pos();
          // auto idx_row = idx_row_major_t<Conf::dim>(pos, ext);
          if (grid.is_in_bound(pos)) {
            for (int i = 0; i < num_per_cell[idx]; i++) {
              uint32_t offset = num + cum_num_per_cell[idx] + i;
              // uint32_t offset = num + idx_row.linear * mult * 2 + i * 2;
              ptc.x1[offset] = 0.1f * rng();
              ptc.x2[offset] = 0.1f * rng();
              ptc.x3[offset] = 0.0f;

              ptc.cell[offset] = cell;
              Scalar x = grid.template pos<0>(pos[0], ptc.x1[offset]);

              if (i < mult * 2) {
                // if (x < 0.4 * grid.sizes[0]) {
                if (x < 1.0 * grid.sizes[0]) {
                  // Scalar weight = w * (1.0f + 29.0f * (1.0f - x /
                  // grid.sizes[0]));
                  Scalar weight = w;

                  // if (x > 0.4 * grid.sizes[0]) weight *= 0.02;
                  ptc.p1[offset] = 0.0f;
                  ptc.p2[offset] = 0.0f;
                  ptc.p3[offset] = 0.0f;
                  ptc.E[offset] = 1.0f;
                  ptc.weight[offset] = weight;
                  ptc.flag[offset] = set_ptc_type_flag(
                      flag_or(PtcFlag::primary),
                      ((i % 2 == 0) ? PtcType::electron : PtcType::positron));
                }
              } else {
                Scalar x = grid.template pos<0>(pos, 0.0f);
                Scalar y = grid.template pos<1>(pos, 0.0f);
                // auto width_arg = wave.width_arg(x, y);
                auto wave_arg = wave.wave_arg(0.0f, x, y);
                auto rho = wave.Rho(0.0f, x, y);
                int num = floor(math::abs(rho) / q_e);

                Scalar v;
                if (i < mult * 2 + mult_wave * num) {
                  v = 0.0f;
                } else {
                  v = 1.0f / (mult_wave + 1);
                }
                auto v_d = wave.Bz(0.0f, x, y) /
                           math::sqrt(square(wave.Bz(0.0f, x, y)) + Bp * Bp);
                auto gamma = 1.0f / math::sqrt(1.0f - v * v - v_d * v_d);
                ptc.p1[offset] = v * wave.sinth * gamma;
                ptc.p2[offset] = v * wave.costh * gamma;
                ptc.p3[offset] = v_d * gamma;
                ptc.E[offset] = gamma;
                ptc.weight[offset] =
                    weight_enhance_factor * math::abs(rho) / num / q_e;
                if (i < mult * 2 + mult_wave * num) {
                  ptc.flag[offset] = set_ptc_type_flag(
                      flag_or(PtcFlag::primary, PtcFlag::initial),
                      ((rho < 0.0f) ? PtcType::positron : PtcType::electron));
                } else {
                  ptc.flag[offset] = set_ptc_type_flag(
                      flag_or(PtcFlag::primary, PtcFlag::initial),
                      ((rho < 0.0f) ? PtcType::electron : PtcType::positron));
                }
              }

              // Scalar weight = w * cube(
              //     math::abs(0.5f * grid.sizes[0] - x) * 2.0f /
              //     grid.sizes[0]);
              // Scalar weight = std::max(w * cube(
              //     (0.5f * grid.sizes[0] - x) * 2.0f / grid.sizes[0]), 0.02f *
              //     w);
              // Scalar weight = (pos[0] > grid.dims[0] * 0.4f ? 0.020 * w : w);
              // Scalar weight = w;
              // if (wave_arg > 0.0f && wave_arg < 2.0f * M_PI) {
              //   auto rho = wave.Rho(0.0f, x, y);
              //   // auto jx = rho * wave.sinth;
              //   // auto jy = rho * wave.costh;
              //   // auto jy = wave.Jy(0.0f, x, y);
              //   // auto j = math::sqrt(jx * jx + jy * jy);
              //   // auto v = j / (2.0f * q_e * mult);
              //   // auto v = math::abs(rho) / (2.0f * q_e * mult * weight);
              //   auto v = math::abs(rho) / (math::abs(rho) * 3.0f / (q_e *
              //   mult)) / (q_e * mult);
              //   // auto v3 = math::abs(rho) * 3.0f / (2.0f * q_e * mult *
              //   weight); auto v_d = wave.Bz(0.0f, x, y) /
              //   math::sqrt(square(wave.Bz(0.0f, x, y)) + Bp * Bp);
              //   // auto v_d = 0.0f;
              //   auto gamma = 1.0f / math::sqrt(1.0f - v * v - v_d * v_d);
              //   // auto gamma3 = 1.0f / math::sqrt(1.0f - v3 * v3 - v_d *
              //   v_d);
              //   // auto sgn_jy = sgn(jy);

              //   if (rho > 0.0f) {
              //     ptc.p1[offset] = 0.0f;
              //     ptc.p1[offset + 1] = v * wave.sinth * gamma;
              //     ptc.p2[offset] = 0.0f;
              //     ptc.p2[offset + 1] = v * wave.costh * gamma;
              //     ptc.p3[offset] = v_d / math::sqrt(1.0f - v_d * v_d);
              //     ptc.p3[offset + 1] = v_d * gamma;
              //     ptc.E[offset] = 1.0f / math::sqrt(1.0f - v_d * v_d);
              //     ptc.E[offset + 1] = gamma;
              //     ptc.weight[offset] = weight + math::abs(rho) * 2.0f / (q_e
              //     * mult); ptc.weight[offset + 1] = weight + math::abs(rho)
              //     * 3.0f / (q_e * mult);
              //     // ptc.weight[offset] = weight;
              //     // ptc.weight[offset + 1] = weight;
              //   } else {
              //     ptc.p1[offset] = v * wave.sinth * gamma;
              //     ptc.p1[offset + 1] = 0.0f;
              //     ptc.p2[offset] = v * wave.costh * gamma;
              //     ptc.p2[offset + 1] = 0.0f;
              //     ptc.p3[offset] = v_d * gamma;
              //     ptc.p3[offset + 1] = v_d / math::sqrt(1.0f - v_d * v_d);
              //     ptc.E[offset] = gamma;
              //     ptc.E[offset + 1] = 1.0f / math::sqrt(1.0f - v_d * v_d);
              //     ptc.weight[offset] = weight + math::abs(rho) * 3.0f / (q_e
              //     * mult); ptc.weight[offset + 1] = weight + math::abs(rho)
              //     * 2.0f / (q_e * mult);
              //     // ptc.weight[offset] = weight;
              //     // ptc.weight[offset + 1] = weight;
              //   }
              //   // ptc.p1[offset] = -jx * gamma / (2.0f * q_e * mult *
              //   weight);
              //   // ptc.p1[offset + 1] = jx * gamma / (2.0f * q_e * mult *
              //   weight);
              //   // ptc.p2[offset] = -jy * gamma / (2.0f * q_e * mult *
              //   weight);
              //   // ptc.p2[offset + 1] = jy * gamma / (2.0f * q_e * mult *
              //   weight);
              //   // ptc.p3[offset] = v_d * gamma;
              //   // ptc.p3[offset + 1] = v_d * gamma;
              //   // ptc.E[offset] = gamma;
              //   // ptc.E[offset + 1] = gamma;
              //   // ptc.weight[offset] = ptc.weight[offset + 1] = weight;
              // } else {
              //   ptc.p1[offset] = ptc.p1[offset + 1] = 0.0f;
              //   ptc.p2[offset] = ptc.p2[offset + 1] = 0.0f;
              //   ptc.p3[offset] = ptc.p3[offset + 1] = 0.0f;
              //   ptc.E[offset] = ptc.E[offset + 1] = 1.0f;
              //   ptc.weight[offset] = ptc.weight[offset + 1] = weight;
              // }

              // ptc.cell[offset] = ptc.cell[offset + 1] = idx.linear;

              // // Scalar x = grid.template pos<0>(pos[0], ptc.x1[offset]);
              // ptc.flag[offset] = set_ptc_type_flag(flag_or(PtcFlag::primary),
              //                                      PtcType::electron);
              // ptc.flag[offset + 1] = set_ptc_type_flag(
              //     flag_or(PtcFlag::primary), PtcType::positron);
            }
          }
        }
      },
      ptc.dev_ptrs(), states.states(), weight, num_per_cell.dev_ndptr(),
      cum_num_per_cell.dev_ndptr());
  CudaSafeCall(hipDeviceSynchronize());
  ptc.set_num(num + new_particles);
}

template void initial_condition_wave<Config<2>>(
    vector_field<Config<2>> &B,
    vector_field<Config<2>> &E, vector_field<Config<2>> &B0,
    particle_data_t &ptc, curand_states_t &states, int mult, Scalar weight);

}  // namespace Aperture
