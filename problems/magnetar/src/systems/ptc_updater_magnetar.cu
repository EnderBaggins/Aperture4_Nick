#include "hip/hip_runtime.h"
#include "ptc_updater_magnetar.h"
#include "data/curand_states.h"
#include "framework/config.h"
#include "systems/grid_sph.h"
#include "systems/forces/sync_cooling.hpp"
#include "systems/forces/gravity.hpp"
#include "systems/helpers/ptc_update_helper.hpp"
#include "utils/kernel_helper.hpp"

namespace Aperture {

template <typename Pusher>
struct pusher_impl_magnetar {
  Pusher pusher;
  double cooling_coef = 0.0, B0 = 1.0, g0 = 0.1;
  double res_drag_coef = 0.0, BQ = 1.0, star_kT = 0.01;
  bool gravity_on = false;
  hiprandState* rand_states;

  pusher_impl_magnetar(sim_environment& env) {
    env.params().get_value("sync_cooling_coef", cooling_coef);
    env.params().get_value("Bp", B0);
    env.params().get_value("gravity", g0);
    env.params().get_value("gravity_on", gravity_on);
    env.params().get_value("res_drag_coef", res_drag_coef);
    env.params().get_value("BQ", BQ);
    env.params().get_value("star_kT", star_kT);

    curand_states_t* rand;
    env.get_data("rand_states", &rand);
    rand_states = rand->states();
  }

  HOST_DEVICE pusher_impl_magnetar(const pusher_impl_magnetar<Pusher>& other) = default;

  template <typename Scalar>
  __device__ void operator()(ptc_ptrs& ptc, uint32_t n, EB_t<Scalar>& EB,
                            Scalar qdt_over_2m, Scalar dt) {
    using Conf = Config<2>;
    auto& grid = dev_grid<2>();
    auto ext = grid.extent();
    auto idx = Conf::idx(ptc.cell[n], ext);
    auto pos = idx.get_pos();

    Scalar p1 = ptc.p1[n], p2 = ptc.p2[n], p3 = ptc.p3[n];
    Scalar gamma = ptc.E[n];
    Scalar r = grid_sph_t<Conf>::radius(grid.template pos<0>(pos[0], ptc.x1[n]));

    pusher(p1, p2, p3, gamma, EB.E1, EB.E2,
           EB.E3, EB.B1, EB.B2, EB.B3, qdt_over_2m, dt);

    if (gravity_on) {
      gravity(p1, p2, p3, gamma, r, dt, (Scalar)g0, qdt_over_2m * 2.0f / dt);
    }

    sync_kill_perp(p1, p2, p3, gamma, EB.E1, EB.E2, EB.E3,
                   EB.B1, EB.B2, EB.B3, qdt_over_2m * 2.0f / dt,
                   (Scalar)cooling_coef, (Scalar)B0);

    auto flag = ptc.flag[n];
    int sp = get_ptc_type(flag);
    if (sp != (int)PtcType::ion) {
      int tid = threadIdx.x + blockIdx.x * blockDim.x;
      cuda_rng_t rng(&rand_states[tid]);

      // Compute resonant drag
      Scalar p = sqrt(p1 * p1 + p2 * p2 + p3 * p3);
      Scalar B = sqrt(EB.B1 * EB.B1 + EB.B2 * EB.B2 + EB.B3 * EB.B3);
      Scalar pdotB = (p1 * EB.B1 + p2 * EB.B2 + p3 * EB.B3) / B;

      Scalar pB1 = p1 / p;
      Scalar pB2 = p2 / p;
      Scalar pB3 = p3 / p;

      Scalar mu = std::abs(EB.B1 / B);
      Scalar p_mag_signed = sgn(pdotB) * sgn(EB.B1) * std::abs(pdotB);
      Scalar g = sqrt(1.0f + p_mag_signed * p_mag_signed);

      Scalar beta = sqrt(1.0f - 1.0f / (g * g));
      Scalar y = std::abs((B / BQ) /
                          (star_kT * (g - p_mag_signed * mu)));
      if (y < 30.0f && y > 0.0f) {
        Scalar coef = res_drag_coef * square(star_kT) * y *
                      y / (r * r * (std::exp(y) - 1.0f));
        Scalar Nph = std::abs(coef / gamma) * dt;
        Scalar Eph =
            min(g - 1.0f,
                g * (1.0f - 1.0f / std::sqrt(1.0f + 2.0f * B / BQ)));

        if (Eph > 2.0f) {
          // Produce individual tracked photons
          if (Nph < 1.0f) {
            float u = rng();
            if (u < Nph)
              set_flag(ptc.flag[n], PtcFlag::emit_photon);
          } else {
            set_flag(ptc.flag[n], PtcFlag::emit_photon);
          }
        } else {
          // Compute analytically the drag force on the particle
          Scalar drag_coef =
              coef * star_kT * y * (g * mu - p_mag_signed);
          if (EB.B1 < 0.0f)
            drag_coef = -drag_coef;
          p1 += EB.B1 * dt * drag_coef / B;
          p2 += EB.B2 * dt * drag_coef / B;
          p3 += EB.B3 * dt * drag_coef / B;
        }
      }
    }

    ptc.p1[n] = p1;
    ptc.p2[n] = p2;
    ptc.p3[n] = p3;
    ptc.E[n] = gamma;
  }
};

template <typename Conf>
ptc_updater_magnetar<Conf>::ptc_updater_magnetar(sim_environment& env,
                                                 const grid_sph_t<Conf>& grid,
                                                 const domain_comm<Conf>* comm)
    : ptc_updater_sph_cu<Conf>(env, grid, comm) {}

template <typename Conf>
void
ptc_updater_magnetar<Conf>::init() {
  ptc_updater_sph_cu<Conf>::init();

  m_impl_boris = std::make_unique<pusher_impl_magnetar<boris_pusher>>(this->m_env);
  m_impl_vay = std::make_unique<pusher_impl_magnetar<vay_pusher>>(this->m_env);
  m_impl_higuera = std::make_unique<pusher_impl_magnetar<higuera_pusher>>(this->m_env);
}

template <typename Conf>
void
ptc_updater_magnetar<Conf>::register_data_components() {
  ptc_updater_sph_cu<Conf>::register_data_components();

  int ph_flux_n_th = 256, ph_flux_n_E = 100;
  this->m_env.params().get_value("ph_flux_n_th", ph_flux_n_th);
  this->m_env.params().get_value("ph_flux_n_E", ph_flux_n_E);
  m_ph_flux = this->m_env.template register_data<multi_array_data<float, 2>>(
      "ph_flux", extent(ph_flux_n_E, ph_flux_n_th), MemType::host_device);
}

template <typename Conf>
void
ptc_updater_magnetar<Conf>::push_default(double dt) {
  // dispatch according to enum. This will also instantiate all the versions of
  // push
  if (this->m_pusher == Pusher::boris) {
    this->push(dt, *m_impl_boris);
  } else if (this->m_pusher == Pusher::vay) {
    this->push(dt, *m_impl_vay);
  } else if (this->m_pusher == Pusher::higuera) {
    this->push(dt, *m_impl_higuera);
  }
}

#include "systems/ptc_updater_cu_impl.hpp"

template class ptc_updater_magnetar<Config<2>>;

}  // namespace Aperture
