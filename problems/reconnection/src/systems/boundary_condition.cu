#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "boundary_condition.h"
#include "core/math.hpp"
#include "framework/config.h"
#include "systems/grid.h"
#include "systems/policies/exec_policy_cuda.hpp"
#include "utils/kernel_helper.hpp"
#include "utils/range.hpp"
#include "utils/util_functions.h"

namespace Aperture {

HOST_DEVICE Scalar
pml_sigma(Scalar x, Scalar xh, Scalar pmlscale, Scalar sig0) {
  if (x > xh)
    return sig0 * square((x - xh) / pmlscale);
  else
    return 0.0;
}

template <typename Conf>
boundary_condition<Conf>::boundary_condition(const grid_t<Conf> &grid)
    : m_grid(grid) {
  using multi_array_t = typename Conf::multi_array_t;

  sim_env().params().get_value("damping_coef", m_damping_coef);
  value_t sigma = 10.0;
  sim_env().params().get_value("sigma", sigma);
  m_Bp = math::sqrt(sigma);
  sim_env().params().get_value("damping_length", m_damping_length);
  sim_env().params().get_value("upstream_kT", m_upstream_kT);
  sim_env().params().get_value("upstream_n", m_upstream_n);
  m_inj_length = m_damping_length / 2;

  Logger::print_info("Boundary condition Bp is {}", m_Bp);
  // m_prev_E1 = std::make_unique<multi_array_t>(
  //     extent(m_damping_length, m_grid.dims[1]), MemType::device_only);
  // m_prev_E2 = std::make_unique<multi_array_t>(
  //     extent(m_damping_length, m_grid.dims[1]), MemType::device_only);
  // m_prev_E3 = std::make_unique<multi_array_t>(
  //     extent(m_damping_length, m_grid.dims[1]), MemType::device_only);
  // m_prev_B1 = std::make_unique<multi_array_t>(
  //     extent(m_damping_length, m_grid.dims[1]), MemType::device_only);
  // m_prev_B2 = std::make_unique<multi_array_t>(
  //     extent(m_damping_length, m_grid.dims[1]), MemType::device_only);
  // m_prev_B3 = std::make_unique<multi_array_t>(
  //     extent(m_damping_length, m_grid.dims[1]), MemType::device_only);

  // m_prev_E1->assign_dev(0.0f);
  // m_prev_E2->assign_dev(0.0f);
  // m_prev_E3->assign_dev(0.0f);
  // m_prev_B1->assign_dev(0.0f);
  // m_prev_B2->assign_dev(0.0f);
  // m_prev_B3->assign_dev(0.0f);

  // m_prev_E.set_memtype(MemType::host_device);
  // m_prev_B.set_memtype(MemType::host_device);
  // m_prev_E.resize(3);
  // m_prev_B.resize(3);
  // m_prev_E[0] = m_prev_E1->dev_ptr();
  // m_prev_E[1] = m_prev_E2->dev_ptr();
  // m_prev_E[2] = m_prev_E3->dev_ptr();
  // m_prev_B[0] = m_prev_B1->dev_ptr();
  // m_prev_B[1] = m_prev_B2->dev_ptr();
  // m_prev_B[2] = m_prev_B3->dev_ptr();
  // m_prev_E.copy_to_device();
  // m_prev_B.copy_to_device();

  extent_t<Conf::dim> ext_inj(grid.reduced_dim(0), m_inj_length);
  m_dens_e1 = std::make_unique<multi_array_t>(ext_inj, MemType::device_only);
  m_dens_p1 = std::make_unique<multi_array_t>(ext_inj, MemType::device_only);
  m_dens_e2 = std::make_unique<multi_array_t>(ext_inj, MemType::device_only);
  m_dens_p2 = std::make_unique<multi_array_t>(ext_inj, MemType::device_only);
}

template <typename Conf>
void
boundary_condition<Conf>::init() {
  sim_env().get_data("Edelta", E);
  sim_env().get_data("E0", E0);
  sim_env().get_data("Bdelta", B);
  sim_env().get_data("B0", B0);
  // sim_env().get_data("rand_states", &rand_states);
  sim_env().get_data("particles", ptc);
  sim_env().get_data("rng_states", rng_states);
}

template <typename Conf>
void
boundary_condition<Conf>::update(double dt, uint32_t step) {
  damp_fields();
  inject_plasma();
}

template <typename Conf>
void
boundary_condition<Conf>::damp_fields() {
  typedef typename Conf::idx_t idx_t;
  value_t Bp = m_Bp;

  // Apply damping boundary condition on both Y boundaries
  kernel_launch(
      [Bp] __device__(auto e, auto b, auto prev_e, auto prev_b,
                      auto damping_length, auto damping_coef) {
        auto &grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();
        // auto ext_damping = extent(damping_length, grid.dims[0]);
        for (auto n0 : grid_stride_range(0, grid.dims[0])) {
          // y = -y_max boundary
          for (int i = 0; i < damping_length; i++) {
            int n1 = i;
            auto idx = idx_t(index_t<2>(n0, n1), ext);
            value_t lambda =
                1.0f - damping_coef * cube((value_t)(damping_length - i) /
                                           (damping_length - 1));
            e[0][idx] *= lambda;
            e[1][idx] *= lambda;
            e[2][idx] *= lambda;
            b[0][idx] = lambda * (b[0][idx] + Bp) - Bp;
            b[1][idx] *= lambda;
            b[2][idx] *= lambda;
          }
          // y = y_max boundary
          for (int i = 0; i < damping_length; i++) {
            int n1 = grid.dims[1] - damping_length + i;
            auto idx = idx_t(index_t<2>(n0, n1), ext);
            value_t lambda =
                1.0f - damping_coef * cube((value_t)i / (damping_length - 1));
            e[0][idx] *= lambda;
            e[1][idx] *= lambda;
            e[2][idx] *= lambda;
            b[0][idx] = lambda * (b[0][idx] - Bp) + Bp;
            b[1][idx] *= lambda;
            b[2][idx] *= lambda;
          }
        }
      },
      E->get_ptrs(), B->get_ptrs(), m_prev_E.dev_ptr(), m_prev_B.dev_ptr(),
      m_damping_length, m_damping_coef);
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();
}

template <typename Conf>
void
boundary_condition<Conf>::inject_plasma() {
  m_dens_e1->assign_dev(0.0f);
  m_dens_p1->assign_dev(0.0f);
  m_dens_e2->assign_dev(0.0f);
  m_dens_p2->assign_dev(0.0f);

  auto inj_length = m_inj_length;
  auto upstream_kT = m_upstream_kT;
  auto upstream_n = m_upstream_n;
  auto num = ptc->number();

  // Measure the density in the injection region and determine how many
  // particles need to be injected
  using policy = exec_policy_cuda<Conf>;
  policy::launch(
      [inj_length, num] __device__(auto ptc, auto dens_e1, auto dens_p1,
                                   auto dens_e2, auto dens_p2) {
        auto &grid = policy::grid();
        auto ext = grid.extent();
        auto ext_inj = extent_t<2>(grid.reduced_dim(0), inj_length);

        for (auto n : grid_stride_range(0, num)) {
          uint32_t cell = ptc.cell[n];
          if (cell == empty_cell) continue;

          auto idx = Conf::idx(cell, ext);
          auto pos = get_pos(idx, ext);

          if (pos[1] - grid.guard[1] < inj_length) {
            index_t<2> pos_inj(pos[0] - grid.guard[0], pos[1] - grid.guard[1]);
            auto sp = get_ptc_type(ptc.flag[n]);
            if (sp == 0) {
              atomic_add(&dens_e1[Conf::idx(pos_inj, ext_inj)], ptc.weight[n]);
            } else if (sp == 1) {
              atomic_add(&dens_p1[Conf::idx(pos_inj, ext_inj)], ptc.weight[n]);
            }
          } else if (pos[1] >= grid.dims[1] - grid.guard[1] - inj_length) {
            index_t<2> pos_inj(
                pos[0] - grid.guard[0],
                pos[1] - grid.dims[1] + grid.guard[1] + inj_length);
            auto sp = get_ptc_type(ptc.flag[n]);
            if (sp == 0) {
              atomic_add(&dens_e2[Conf::idx(pos_inj, ext_inj)], ptc.weight[n]);
            } else if (sp == 1) {
              atomic_add(&dens_p2[Conf::idx(pos_inj, ext_inj)], ptc.weight[n]);
            }
          }
        }
      },
      ptc, *m_dens_e1, *m_dens_p1, *m_dens_e2, *m_dens_p2);
  policy::sync();

  // Actually inject the particles
  buffer<int> offset(1, MemType::host_device);
  offset[0] = num;
  offset.copy_to_device();
  auto ext_inj = m_dens_e1->extent();

  policy::launch(
      [inj_length, upstream_kT, upstream_n, ext_inj] __device__(
          auto ptc, auto dens_e1, auto dens_p1, auto dens_e2, auto dens_p2,
          auto states, auto offset) {
        auto &grid = policy::grid();
        auto ext = grid.extent();
        rng_t rng(states);

        for (auto idx :
             grid_stride_range(Conf::begin(ext_inj), Conf::end(ext_inj))) {
          auto pos_inj = get_pos(idx, ext_inj);
          // First check lower boundary
          value_t dens = dens_e1[idx] + dens_p1[idx];
          if (dens < 2.0f - 2.0f / upstream_n) {
            auto n = atomic_add(offset, 2);
            index_t<2> pos = index_t<2>(pos_inj[0] + grid.guard[0],
                                        pos_inj[1] + grid.guard[1]);

            ptc.x1[n] = ptc.x1[n + 1] = rng.uniform<value_t>();
            ptc.x2[n] = ptc.x2[n + 1] = rng.uniform<value_t>();
            ptc.x3[n] = ptc.x3[n + 1] = rng.uniform<value_t>();
            auto p1 = rng.gaussian<value_t>(2.0f * upstream_kT);
            auto p2 = rng.gaussian<value_t>(2.0f * upstream_kT);
            auto p3 = rng.gaussian<value_t>(2.0f * upstream_kT);
            ptc.p1[n] = p1;
            ptc.p2[n] = p2;
            ptc.p3[n] = p3;
            ptc.E[n] = math::sqrt(1.0f + p1 * p1 + p2 * p2 + p3 * p3);

            p1 = rng.gaussian<value_t>(2.0f * upstream_kT);
            p2 = rng.gaussian<value_t>(2.0f * upstream_kT);
            p3 = rng.gaussian<value_t>(2.0f * upstream_kT);
            ptc.p1[n + 1] = p1;
            ptc.p2[n + 1] = p2;
            ptc.p3[n + 1] = p3;
            ptc.E[n + 1] = math::sqrt(1.0f + p1 * p1 + p2 * p2 + p3 * p3);

            ptc.weight[n] = ptc.weight[n + 1] = 1.0f / upstream_n;
            auto idx_p = Conf::idx(pos, ext);
            ptc.cell[n] = ptc.cell[n + 1] = idx_p.linear;
            ptc.flag[n] = set_ptc_type_flag(0, PtcType::electron);
            ptc.flag[n + 1] = set_ptc_type_flag(0, PtcType::positron);
          }

          // Then check upper boundary
          dens = dens_e2[idx] + dens_p2[idx];
          if (dens < 2.0f - 2.0f / upstream_n) {
            auto n = atomic_add(offset, 2);
            index_t<2> pos = index_t<2>(
                pos_inj[0] + grid.guard[0],
                pos_inj[1] + (grid.dims[1] - grid.guard[1] - inj_length));

            ptc.x1[n] = ptc.x1[n + 1] = rng.uniform<value_t>();
            ptc.x2[n] = ptc.x2[n + 1] = rng.uniform<value_t>();
            ptc.x3[n] = ptc.x3[n + 1] = rng.uniform<value_t>();
            auto p1 = rng.gaussian<value_t>(2.0f * upstream_kT);
            auto p2 = rng.gaussian<value_t>(2.0f * upstream_kT);
            auto p3 = rng.gaussian<value_t>(2.0f * upstream_kT);
            ptc.p1[n] = p1;
            ptc.p2[n] = p2;
            ptc.p3[n] = p3;
            ptc.E[n] = math::sqrt(1.0f + p1 * p1 + p2 * p2 + p3 * p3);

            p1 = rng.gaussian<value_t>(2.0f * upstream_kT);
            p2 = rng.gaussian<value_t>(2.0f * upstream_kT);
            p3 = rng.gaussian<value_t>(2.0f * upstream_kT);
            ptc.p1[n + 1] = p1;
            ptc.p2[n + 1] = p2;
            ptc.p3[n + 1] = p3;
            ptc.E[n + 1] = math::sqrt(1.0f + p1 * p1 + p2 * p2 + p3 * p3);

            ptc.weight[n] = ptc.weight[n + 1] = 1.0f / upstream_n;
            auto idx_p = Conf::idx(pos, ext);
            ptc.cell[n] = ptc.cell[n + 1] = idx_p.linear;
            ptc.flag[n] = set_ptc_type_flag(0, PtcType::electron);
            ptc.flag[n + 1] = set_ptc_type_flag(0, PtcType::positron);
          }
        }
      },
      ptc, *m_dens_e1, *m_dens_p1, *m_dens_e2, *m_dens_p2, rng_states, offset);
  policy::sync();

  offset.copy_to_host();
  ptc->set_num(offset[0]);
  Logger::print_info("Injected {} particles", offset[0] - num);
}

template class boundary_condition<Config<2>>;

}  // namespace Aperture
