#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "data/rng_states.h"
#include "framework/config.h"
#include "framework/environment.h"
#include "systems/compute_lorentz_factor.h"
#include "systems/data_exporter.h"
#include "systems/domain_comm.h"
#include "systems/field_solver.h"
#include "systems/gather_momentum_space.h"
#include "systems/ptc_injector_new.h"
// #include "systems/legacy/ptc_updater_old.h"
#include "systems/policies/coord_policy_cartesian_impl_cooling.hpp"
#include "systems/ptc_updater.h"
#include <iostream>

using namespace std;
using namespace Aperture;

template <typename Conf>
void init_upstream(vector_field<Conf> &E,
                   vector_field<Conf> &B, particle_data_t &ptc,
                   rng_states_t &states) {
  using value_t = typename Conf::value_t;
  // auto delta = sim_env().params().get_as<double>("current_sheet_delta", 5.0);
  value_t B_g = sim_env().params().get_as<double>("guide_field", 0.0);
  value_t sigma = sim_env().params().get_as<double>("sigma", 1.0e3);
  value_t kT_cs = sim_env().params().get_as<double>("current_sheet_kT", 1.0);
  value_t kT_upstream = sim_env().params().get_as<double>("upstream_kT", 0.01);
  value_t beta_d =
      sim_env().params().get_as<double>("current_sheet_drift", 0.5);
  value_t gamma_d = 1.0f / math::sqrt(1.0f - beta_d * beta_d);

  value_t delta = 2.0f * kT_cs / (math::sqrt(sigma) * gamma_d * beta_d);
  value_t n_d = gamma_d * sigma / (4.0f * kT_cs);

  int n_cs = sim_env().params().get_as<int64_t>("current_sheet_n", 15);
  int n_upstream = sim_env().params().get_as<int64_t>("upstream_n", 5);
  value_t q_e = sim_env().params().get_as<double>("q_e", 1.0);

  // Our unit for length will be upstream c/\omega_p, therefore sigma determines
  // the upstream field strength
  value_t B0 = math::sqrt(sigma);
  auto &grid = B.grid();
  auto ext = grid.extent();
  value_t ysize = grid.sizes[1];

  // Initialize the magnetic field values
  B.set_values(0, [B0, delta, ysize](auto x, auto y, auto z) {
    return B0;
  });
  E.set_values(2, [B0, delta, ysize](auto x, auto y, auto z) {
    return 0.1 * B0;
  });

  auto injector = sim_env().register_system<ptc_injector<Conf, exec_policy_gpu>>(grid);

  injector->inject_pairs(
      [] __device__(auto &pos, auto &grid, auto &ext) { return true; },
      [n_upstream] __device__(auto &pos, auto &grid, auto &ext) {
        return 2 * n_upstream;
      },
      [kT_upstream] __device__(auto &pos, auto &grid, auto &ext, rng_t<exec_tags::device> &rng,
                               PtcType type) {
        auto p1 = rng.gaussian<value_t>(2.0f * kT_upstream);
        auto p2 = rng.gaussian<value_t>(2.0f * kT_upstream);
        auto p3 = rng.gaussian<value_t>(2.0f * kT_upstream);
        return vec_t<value_t, 3>(p1, p2, p3);
      },
      // [n_upstream] __device__(auto &pos, auto &grid, auto &ext) {
      [n_upstream] __device__(auto& x_global) {
        return 1.0 / n_upstream;
      });

  Logger::print_info("After initial condition, there are {} particles", ptc.number());
}

int main(int argc, char *argv[]) {
  typedef Config<2> Conf;
  // sim_environment env(&argc, &argv);
  auto &env = sim_environment::instance(&argc, &argv);

  env.params().add("log_level", (int64_t)LogLevel::info);

  // auto comm = env.register_system<domain_comm<Conf>>(env);
  domain_comm<Conf> comm;
  // auto grid = env.register_system<grid_t<Conf>>(env, comm);
  grid_t<Conf> grid(comm);
  // auto pusher = env.register_system<ptc_updater_old_cu<Conf>>(grid, &comm);
  auto pusher = env.register_system<ptc_updater<
      Conf, exec_policy_gpu, coord_policy_cartesian_impl_cooling>>(grid, &comm);
  auto lorentz = env.register_system<compute_lorentz_factor_cu<Conf>>(grid);
  auto momentum =
      env.register_system<gather_momentum_space<Conf, exec_policy_gpu>>(grid);
  auto solver = env.register_system<field_solver_cu<Conf>>(grid, &comm);
  // auto rad = env.register_system<ph_freepath_dev<Conf>>(*grid, comm);
  auto exporter = env.register_system<data_exporter<Conf>>(grid, &comm);

  env.init();

  vector_field<Conf> *E0, *B0, *Bdelta, *Edelta;
  particle_data_t *ptc;
  // curand_states_t *states;
  rng_states_t *states;
  env.get_data("B0", &B0);
  env.get_data("E0", &E0);
  env.get_data("Bdelta", &Bdelta);
  env.get_data("Edelta", &Edelta);
  env.get_data("particles", &ptc);
  env.get_data("rng_states", &states);
  // env.get_data("rand_states", &states);

  init_upstream(*E0, *B0, *ptc, *states);

  env.run();
  return 0;
}
