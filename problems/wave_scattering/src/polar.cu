#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "data/fields.h"
#include "framework/config.h"
#include "framework/environment.h"
#include "framework/system.h"
#include "systems/compute_lorentz_factor.h"
#include "systems/data_exporter.h"
#include "systems/field_solver_polar.h"
// #include "systems/legacy/ptc_updater_sph.h"
#include "systems/gather_momentum_space.h"
#include "systems/grid_polar.hpp"
#include "systems/policies/coord_policy_polar.hpp"
#include "systems/ptc_updater_base.h"
#include "utils/kernel_helper.hpp"
// #include "systems/ptc_injector.h"
#include <iostream>

namespace Aperture {

template <typename Conf>
class boundary_condition : public system_t {
 protected:
  const grid_curv_t<Conf>& m_grid;
  double m_E0 = 1.0;
  double m_omega_0 = 0.0;
  double m_omega_t = 0.0;
  double m_Bp = 1.0;

  vector_field<Conf>*E, *B, *E0, *B0;

 public:
  static std::string name() { return "boundary_condition"; }

  boundary_condition(const grid_curv_t<Conf>& grid) : m_grid(grid) {}

  void init() override {
    sim_env().get_data("Edelta", &E);
    sim_env().get_data("E0", &E0);
    sim_env().get_data("Bdelta", &B);
    sim_env().get_data("B0", &B0);

    sim_env().params().get_value("E0", m_E0);
    sim_env().params().get_value("omega_t", m_omega_t);
    sim_env().params().get_value("Bp", m_Bp);
  }

  void update(double dt, uint32_t step) override {
    auto ext = m_grid.extent();
    typedef typename Conf::idx_t idx_t;
    typedef typename Conf::value_t value_t;

    value_t time = sim_env().get_time();
    value_t Bp = m_Bp;
    value_t omega;
    // if (m_omega_t * time < 5000.0)
    value_t phase = time * m_omega_t;
    if (phase < 8.0)
      omega = m_E0 * sin(2.0 * M_PI * phase);
    else
      omega = 0.0;
    Logger::print_debug("time is {}, Omega is {}", time, omega);

    kernel_launch(
        [ext, time, omega, Bp] __device__(auto e, auto b, auto e0, auto b0) {
          auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
          for (auto n1 : grid_stride_range(0, grid.dims[1])) {
            value_t theta =
                grid_polar_t<Conf>::theta(grid.template pos<1>(n1, false));
            value_t theta_s =
                grid_polar_t<Conf>::theta(grid.template pos<1>(n1, true));

            // For quantities that are not continuous across the surface
            for (int n0 = 0; n0 < grid.guard[0] + 1; n0++) {
              auto idx = idx_t(index_t<2>(n0, n1), ext);
              e[0][idx] = 0.0;
              b[1][idx] = 0.0;
              b[2][idx] = 0.0;
            }
            // For quantities that are continuous across the surface
            for (int n0 = 0; n0 < grid.guard[0] + 1; n0++) {
              auto idx = idx_t(index_t<2>(n0, n1), ext);
              value_t r =
                  grid_polar_t<Conf>::radius(grid.template pos<0>(n0, false));
              value_t r_s =
                  grid_polar_t<Conf>::radius(grid.template pos<0>(n0, true));
              b[0][idx] = 0.0;
              e[1][idx] = 0.0;
              // if (theta_s > 0.7 && theta_s < 1.2)
              // e[2][idx] = -omega * sin(theta_s) * r_s * b0[0][idx];
              e[2][idx] = -omega * Bp;
              // else
              //   e[2][idx] = 0.0;
              // e[2][idx] = 0.0;
            }
          }
        },
        E->get_ptrs(), B->get_ptrs(), E0->get_ptrs(), B0->get_ptrs());
    CudaSafeCall(hipDeviceSynchronize());
  }
};

}  // namespace Aperture

using namespace std;
using namespace Aperture;

int
main(int argc, char* argv[]) {
  typedef Config<2> Conf;
  auto& env = sim_environment::instance(&argc, &argv, false);

  domain_comm<Conf> comm;
  grid_polar_t<Conf> grid(comm);
  grid.init();

  auto pusher = env.register_system<ptc_updater_new<
      Conf, exec_policy_cuda, coord_policy_polar>>(grid, comm);
  auto solver = env.register_system<field_solver_polar_cu<Conf>>(grid, &comm);
  auto bc = env.register_system<boundary_condition<Conf>>(grid);
  auto exporter = env.register_system<data_exporter<Conf>>(grid, &comm);

  env.init();

  // Initial conditions
  vector_field<Conf> *B0, *Bdelta, *Edelta;
  env.get_data("B0", &B0);
  double Bp = sim_env().params().get_as<double>("Bp", 100.0);
  B0->set_values(1, [Bp](auto x, auto y, auto z) {
      return Bp / x;
    });

  particle_data_t *ptc;
  env.get_data("particles", &ptc);

  env.run();
  return 0;
}
