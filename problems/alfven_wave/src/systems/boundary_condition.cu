#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "boundary_condition.h"
#include "core/math.hpp"
#include "framework/config.h"
#include "systems/grid_sph.hpp"
#include "utils/kernel_helper.hpp"
#include "utils/util_functions.h"

namespace Aperture {

struct wpert_sph_t {
  float rpert1, rpert2;
  float tp_start, tp_end, nT, dw0;

  HD_INLINE wpert_sph_t(float rp1, float rp2, float tp_s, float tp_e, float nT_,
                        float dw0_)
      : rpert1(rp1),
        rpert2(rp2),
        tp_start(tp_s),
        tp_end(tp_e),
        nT(nT_),
        dw0(dw0_) {}

  HD_INLINE Scalar operator()(Scalar t, Scalar r, Scalar th) {
    Scalar th1 = math::acos(math::sqrt(1.0f - 1.0f / rpert1));
    Scalar th2 = math::acos(math::sqrt(1.0f - 1.0f / rpert2));
    if (th1 > th2)
      swap_values(th1, th2);

    Scalar mu = (th1 + th2) / 2.0;
    Scalar s = (mu - th1) / 3.0;
    if (t >= tp_start && t <= tp_end && th >= th1 && th <= th2) {
      Scalar omega =
          dw0 * math::exp(-0.5 * square((th - mu) / s)) *
          math::sin((t - tp_start) * 2.0 * M_PI * nT / (tp_end - tp_start));
      return omega;
    } else {
      return 0.0;
    }
  }
};

template <typename Conf>
void
inject_particles(particle_data_t& ptc, rng_states_t& rng_states,
                 buffer<float>& surface_n, int num_per_cell,
                 typename Conf::value_t weight,
                 const grid_curv_t<Conf>& grid,
                 typename Conf::value_t rpert1,
                 typename Conf::value_t rpert2) {
  surface_n.assign_dev(0.0f);

  auto ptc_num = ptc.number();
  // First measure surface density
  kernel_launch(
      [ptc_num, rpert1, rpert2] __device__(auto ptc, auto surface_n) {
        auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();
        // Scalar th1 = math::acos(math::sqrt(1.0f - 1.0f / rpert1));
        // Scalar th2 = math::acos(math::sqrt(1.0f - 1.0f / rpert2));
        // if (th1 > th2)
        //   swap_values(th1, th2);
        for (auto n : grid_stride_range(0, ptc_num)) {
          auto c = ptc.cell[n];
          if (c == empty_cell) continue;

          auto idx = typename Conf::idx_t(c, ext);
          auto pos = get_pos(idx, ext);
          // Scalar th = grid.pos<1>(pos[1], false);
          // if (pos[0] == grid.guard[0] && th > th1 && th < th2) {
          if (pos[0] == grid.guard[0]) {
            auto flag = ptc.flag[n];
            auto sp = get_ptc_type(flag);
            // surface_n[pos[1]] += ptc.weight[n];
            atomicAdd(&surface_n[pos[1]],
                      ptc.weight[n] * math::abs(dev_charges[sp]));
          }
        }
      },
      ptc.get_dev_ptrs(), surface_n.dev_ptr());
  CudaSafeCall(hipDeviceSynchronize());

  // Then inject particles
  kernel_launch(
      [ptc_num, weight] __device__(auto ptc, auto surface_n, auto num_inj,
                                   auto states) {
        auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();
        int inj_n0 = grid.guard[0];
        int id = threadIdx.x + blockIdx.x * blockDim.x;
        rng_t rng(states);
        for (auto n1 :
             grid_stride_range(grid.guard[1], grid.dims[1] - grid.guard[1])) {
          size_t offset = ptc_num + n1 * num_inj * 2;
          auto pos = index_t<Conf::dim>(inj_n0, n1);
          auto idx = typename Conf::idx_t(pos, ext);
          Scalar theta = grid.template pos<1>(n1, false);
          if (surface_n[pos[1]] >
              square(0.5f / grid.delta[1]) * math::sin(theta))
            continue;
          for (int i = 0; i < num_inj; i++) {
            auto x2 = rng.uniform<float>();
            theta = grid.template pos<1>(n1, x2);
            ptc.x1[offset + i * 2] = ptc.x1[offset + i * 2 + 1] = 0.5f;
            ptc.x2[offset + i * 2] = ptc.x2[offset + i * 2 + 1] = x2;
            ptc.x3[offset + i * 2] = ptc.x3[offset + i * 2 + 1] = 0.0f;
            ptc.p1[offset + i * 2] = ptc.p1[offset + i * 2 + 1] = 0.0f;
            ptc.p2[offset + i * 2] = ptc.p2[offset + i * 2 + 1] = 0.0f;
            ptc.p3[offset + i * 2] = ptc.p3[offset + i * 2 + 1] = 0.0f;
            ptc.E[offset + i * 2] = ptc.E[offset + i * 2 + 1] = 1.0f;
            ptc.cell[offset + i * 2] = ptc.cell[offset + i * 2 + 1] =
                idx.linear;
            ptc.weight[offset + i * 2] = ptc.weight[offset + i * 2 + 1] =
                weight * math::sin(theta);
            ptc.flag[offset + i * 2] = set_ptc_type_flag(0, PtcType::electron);
            ptc.flag[offset + i * 2 + 1] =
                set_ptc_type_flag(0, PtcType::positron);
          }
        }
      },
      ptc.get_dev_ptrs(), surface_n.dev_ptr(), num_per_cell,
      rng_states.states().dev_ptr());
  CudaSafeCall(hipDeviceSynchronize());

  ptc.add_num(num_per_cell * 2 * grid.dims[1]);
}

template <typename Conf>
void
boundary_condition<Conf>::init() {
  sim_env().get_data("Edelta", &E);
  sim_env().get_data("E0", &E0);
  sim_env().get_data("Bdelta", &B);
  sim_env().get_data("B0", &B0);
  sim_env().get_data("rng_states", &rng_states);
  sim_env().get_data("particles", &ptc);

  sim_env().params().get_value("rpert1", m_rpert1);
  sim_env().params().get_value("rpert2", m_rpert2);
  sim_env().params().get_value("tp_start", m_tp_start);
  sim_env().params().get_value("tp_end", m_tp_end);
  sim_env().params().get_value("nT", m_nT);
  sim_env().params().get_value("dw0", m_dw0);
  Logger::print_info("{}, {}, {}, {}, {}, {}", m_rpert1, m_rpert2, m_tp_start,
                     m_tp_end, m_nT, m_dw0);

  m_surface_n.resize(m_grid.dims[1]);
}

template <typename Conf>
void
boundary_condition<Conf>::update(double dt, uint32_t step) {
  // auto ext = m_grid.extent();
  typedef typename Conf::idx_t idx_t;
  typedef typename Conf::value_t value_t;

  value_t time = sim_env().get_time();
  wpert_sph_t wpert(m_rpert1, m_rpert2, m_tp_start, m_tp_end, m_nT, m_dw0);

  // Apply twist on the stellar surface
  kernel_launch(
      [time] __device__(auto e, auto b, auto e0, auto b0, auto wpert) {
        auto& grid = dev_grid<Conf::dim, typename Conf::value_t>();
        auto ext = grid.extent();
        for (auto n1 : grid_stride_range(0, grid.dims[1])) {
          value_t theta =
              grid_sph_t<Conf>::theta(grid.template pos<1>(n1, false));
          value_t theta_s =
              grid_sph_t<Conf>::theta(grid.template pos<1>(n1, true));

          // For quantities that are not continuous across the surface
          for (int n0 = 0; n0 < grid.guard[0]; n0++) {
            auto idx = idx_t(index_t<2>(n0, n1), ext);
            value_t r =
                grid_sph_t<Conf>::radius(grid.template pos<0>(n0, false));
            value_t omega = wpert(time, r, theta_s);
            // printf("omega is %f\n", omega);
            e[0][idx] = omega * sin(theta_s) * r * b0[1][idx];
            b[1][idx] = 0.0;
            b[2][idx] = 0.0;
          }
          // For quantities that are continuous across the surface
          for (int n0 = 0; n0 < grid.guard[0] + 1; n0++) {
            auto idx = idx_t(index_t<2>(n0, n1), ext);
            value_t r_s =
                grid_sph_t<Conf>::radius(grid.template pos<0>(n0, true));
            value_t omega = wpert(time, r_s, theta);
            b[0][idx] = 0.0;
            e[1][idx] = -omega * sin(theta) * r_s * b0[0][idx];
            e[2][idx] = 0.0;
          }
        }
      },
      E->get_ptrs(), B->get_ptrs(), E0->get_ptrs(), B0->get_ptrs(), wpert);
  CudaSafeCall(hipDeviceSynchronize());

  // Inject particles
  if (step % 1 == 0) {
    // inject_particles<Conf>(*ptc, *rng_states, m_surface_n, 10, 1.0, m_grid, m_rpert1, m_rpert2);
  }
}

template class boundary_condition<Config<2>>;

}  // namespace Aperture
