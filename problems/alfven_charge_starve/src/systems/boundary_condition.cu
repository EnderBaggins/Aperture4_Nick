#include "hip/hip_runtime.h"
#include "boundary_condition.h"
#include "core/math.hpp"
#include "framework/config.h"
#include "systems/grid.h"
#include "utils/kernel_helper.hpp"
#include "utils/util_functions.h"

namespace Aperture {

struct wpert_cart_t {
  float tp_start, tp_end, nT, dw0;

  HD_INLINE wpert_cart_t(float tp_s, float tp_e, float nT_,
                         float dw0_)
      : tp_start(tp_s),
        tp_end(tp_e),
        nT(nT_),
        dw0(dw0_) {}

  HD_INLINE Scalar operator()(Scalar t, Scalar x, Scalar y) {
    if (t >= tp_start && t <= tp_end) {
      Scalar omega =
          dw0 * math::sin((t - tp_start) * 2.0 * M_PI * nT / (tp_end - tp_start));
      return omega;
    } else {
      return 0.0;
    }
  }
};

template <typename Conf>
void
inject_particles(particle_data_t& ptc, curand_states_t& rand_states,
                 buffer<float>& surface_ne, buffer<float>& surface_np, int num_per_cell,
                 typename Conf::value_t weight, const grid_t<Conf>& grid) {
  surface_ne.assign_dev(0.0f);
  surface_np.assign_dev(0.0f);

  auto ptc_num = ptc.number();
  // First measure surface density
  kernel_launch(
      [ptc_num] __device__(auto ptc, auto surface_ne, auto surface_np) {
        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        for (auto n : grid_stride_range(0, ptc_num)) {
          auto c = ptc.cell[n];
          if (c == empty_cell) continue;

          auto idx = typename Conf::idx_t(c, ext);
          auto pos = idx.get_pos();
          if (pos[0] == grid.skirt[0]) {
            auto flag = ptc.flag[n];
            auto sp = get_ptc_type(flag);

            if (sp == 0)
              atomicAdd(&surface_ne[pos[1]],
                        ptc.weight[n] * math::abs(dev_charges[sp]));
            else if (sp == 1)
              atomicAdd(&surface_np[pos[1]],
                        ptc.weight[n] * math::abs(dev_charges[sp]));
           }
        }
      },
      ptc.get_dev_ptrs(), surface_ne.dev_ptr(), surface_np.dev_ptr());
  CudaSafeCall(hipDeviceSynchronize());

  // Then inject particles
  kernel_launch(
      [ptc_num, weight] __device__(auto ptc, auto surface_ne, auto surface_np,
                                   auto num_inj, auto states) {
        auto& grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        int inj_n0 = grid.skirt[0];
        int id = threadIdx.x + blockIdx.x * blockDim.x;
        cuda_rng_t rng(&states[id]);
        for (auto n1 :
             grid_stride_range(grid.skirt[1], grid.dims[1] - grid.skirt[1])) {
          size_t offset = ptc_num + n1 * num_inj * 2;
          auto pos = index_t<Conf::dim>(inj_n0, n1);
          auto idx = typename Conf::idx_t(pos, ext);
          if (std::min(surface_ne[pos[1]], surface_np[pos[1]]) > square(2.0f / grid.delta[0]))
            continue;
          for (int i = 0; i < num_inj; i++) {
            float x2 = rng();
            ptc.x1[offset + i * 2] = ptc.x1[offset + i * 2 + 1] = 1.0f;
            ptc.x2[offset + i * 2] = ptc.x2[offset + i * 2 + 1] = x2;
            ptc.x3[offset + i * 2] = ptc.x3[offset + i * 2 + 1] = 0.0f;
            ptc.p1[offset + i * 2] = ptc.p1[offset + i * 2 + 1] = 0.0f;
            ptc.p2[offset + i * 2] = ptc.p2[offset + i * 2 + 1] = 0.0f;
            ptc.p3[offset + i * 2] = ptc.p3[offset + i * 2 + 1] = 0.0f;
            ptc.E[offset + i * 2] = ptc.E[offset + i * 2 + 1] = 1.0f;
            ptc.cell[offset + i * 2] = ptc.cell[offset + i * 2 + 1] =
                idx.linear;
            ptc.weight[offset + i * 2] = ptc.weight[offset + i * 2 + 1] =
                weight;
            ptc.flag[offset + i * 2] = set_ptc_type_flag(0, PtcType::electron);
            ptc.flag[offset + i * 2 + 1] =
                set_ptc_type_flag(0, PtcType::positron);
          }
        }
      },
      ptc.get_dev_ptrs(), surface_ne.dev_ptr(), surface_np.dev_ptr(), num_per_cell,
      rand_states.states());
  CudaSafeCall(hipDeviceSynchronize());

  ptc.add_num(num_per_cell * 2 * grid.dims[1]);
}

template <typename Conf>
void
boundary_condition<Conf>::init() {
  m_env.get_data("Edelta", &E);
  m_env.get_data("E0", &E0);
  m_env.get_data("Bdelta", &B);
  m_env.get_data("B0", &B0);
  m_env.get_data("rand_states", &rand_states);
  m_env.get_data("particles", &ptc);

  m_env.params().get_value("tp_start", m_tp_start);
  m_env.params().get_value("tp_end", m_tp_end);
  m_env.params().get_value("nT", m_nT);
  m_env.params().get_value("dw0", m_dw0);

  m_surface_ne.set_memtype(MemType::host_device);
  m_surface_ne.resize(m_grid.dims[1]);
  m_surface_np.set_memtype(MemType::host_device);
  m_surface_np.resize(m_grid.dims[1]);
}

template <typename Conf>
void
boundary_condition<Conf>::update(double dt, uint32_t step) {
  auto ext = m_grid.extent();
  typedef typename Conf::idx_t idx_t;
  typedef typename Conf::value_t value_t;

  value_t time = m_env.get_time();
  wpert_cart_t wpert(m_tp_start, m_tp_end, m_nT, m_dw0);

  // Apply twist on the stellar surface
  kernel_launch(
      [ext, time] __device__(auto e, auto b, auto e0, auto b0, auto wpert) {
        auto& grid = dev_grid<Conf::dim>();
        for (auto n1 : grid_stride_range(0, grid.dims[1])) {
          value_t y =
              grid.template pos<1>(n1, false);
          value_t y_s =
              grid.template pos<1>(n1, true);

          // For quantities that are not continuous across the surface
          for (int n0 = 0; n0 < grid.skirt[0]; n0++) {
            auto idx = idx_t(index_t<2>(n0, n1), ext);
            value_t x =
                grid.template pos<0>(n0, false);
            value_t omega = wpert(time, x, y_s);
            // printf("omega is %f\n", omega);
            e[0][idx] = omega * b0[1][idx];
            b[1][idx] = 0.0;
            b[2][idx] = 0.0;
          }
          // For quantities that are continuous across the surface
          for (int n0 = 0; n0 < grid.skirt[0] + 1; n0++) {
            auto idx = idx_t(index_t<2>(n0, n1), ext);
            value_t x_s =
                grid.template pos<0>(n0, true);
            value_t omega = wpert(time, x_s, y);
            b[0][idx] = 0.0;
            e[1][idx] = -omega * b0[0][idx];
            e[2][idx] = 0.0;
          }
        }
      },
      E->get_ptrs(), B->get_ptrs(), E0->get_ptrs(), B0->get_ptrs(), wpert);
  CudaSafeCall(hipDeviceSynchronize());

  // Inject particles
  if (step % 1 == 0 && time > m_tp_start && time < m_tp_end) {
    inject_particles<Conf>(*ptc, *rand_states, m_surface_ne, m_surface_np, 2, 1.0, m_grid);
  }
}

template class boundary_condition<Config<2>>;

}  // namespace Aperture
