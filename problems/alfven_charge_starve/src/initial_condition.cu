#include "hip/hip_runtime.h"
#include "core/math.hpp"
#include "data/curand_states.h"
#include "data/fields.h"
#include "data/particle_data.h"
#include "framework/config.h"
#include "framework/environment.h"
#include "utils/kernel_helper.hpp"

namespace Aperture {

template <typename Conf>
void set_initial_condition(sim_environment &env, vector_field<Conf> &B0,
                           particle_data_t &ptc, curand_states_t &states,
                           int mult, Scalar weight) {
  auto Bp = env.params().get_as<double>("Bp", 1000.0);
  auto muB = env.params().get_as<double>("muB", 1.0);
  B0.set_values(0,
                 [Bp, muB](Scalar x, Scalar y, Scalar z) { return Bp * muB; });
  B0.set_values(1, [Bp, muB](Scalar x, Scalar y, Scalar z) {
    return Bp * math::sqrt(1.0 - muB);
  });
  // pusher->fill_multiplicity(mult, weight);
  // ptc->append_dev({0.0f, 0.0f, 0.0f}, {0.0f, 100.0f, 0.0f}, 200 + 258 *
  // grid->dims[0],
  //                 100.0, set_ptc_type_flag(0, PtcType::positron));

  auto num = ptc.number();
  kernel_launch(
      [num, mult, weight] __device__(auto ptc, auto states) {
        auto &grid = dev_grid<Conf::dim>();
        auto ext = grid.extent();
        int id = threadIdx.x + blockIdx.x * blockDim.x;
        cuda_rng_t rng(&states[id]);
        for (auto n : grid_stride_range(0, ext.size())) {
          auto idx = Conf::idx(n, ext);
          auto pos = idx.get_pos();
          if (grid.is_in_bound(pos)) {
            for (int i = 0; i < mult; i++) {
              uint32_t offset = num + idx.linear * mult * 2 + i * 2;

              ptc.x1[offset] = ptc.x1[offset + 1] = rng();
              ptc.x2[offset] = ptc.x2[offset + 1] = rng();
              ptc.x3[offset] = ptc.x3[offset + 1] = rng();
              ptc.p1[offset] = ptc.p1[offset + 1] = 0.0;
              ptc.p2[offset] = ptc.p2[offset + 1] = 0.0;
              ptc.p3[offset] = ptc.p3[offset + 1] = 0.0;
              ptc.E[offset] = ptc.E[offset + 1] = 1.0;
              ptc.cell[offset] = ptc.cell[offset + 1] = idx.linear;
              Scalar x = grid.template pos<0>(pos[0], ptc.x1[offset]);
              ptc.weight[offset] = ptc.weight[offset + 1] =
                  cube(math::abs(grid.sizes[0] - x) / grid.sizes[0]);
              ptc.flag[offset] = set_ptc_type_flag(flag_or(PtcFlag::primary),
                                                   PtcType::electron);
              ptc.flag[offset + 1] = set_ptc_type_flag(
                  flag_or(PtcFlag::primary), PtcType::positron);
            }
          }
        }
      },
      ptc.dev_ptrs(), states.states());
  CudaSafeCall(hipDeviceSynchronize());
  ptc.set_num(num + mult * 2 * B0.grid().extent().size());
}

template void set_initial_condition<Config<2>>(sim_environment &env,
                                               vector_field<Config<2>> &B0,
                                               particle_data_t &ptc,
                                               curand_states_t &states,
                                               int mult, Scalar weight);

} // namespace Aperture
