#include "hip/hip_runtime.h"
#include "catch.hpp"
#include "core/multi_array.hpp"
#include "core/ndptr.hpp"
#include "utils/logger.h"
// #include "utils/ndptr.h"
#include "utils/range.hpp"
#include "utils/timer.h"
#include "utils/kernel_helper.hpp"
#include <random>
#include <thrust/tuple.h>

using namespace Aperture;

#ifdef CUDA_ENABLED

template <typename T, typename Index_t>
HOST_DEVICE T
interp(T* f, T x, T y, T z, const Index_t& idx) {
  T f11 = (1.0 - z) * f[idx.incX().incY().key] +
          z * f[idx.incX().incY().incZ().key];
  T f10 = (1.0 - z) * f[idx.incX().key] + z * f[idx.incX().incZ().key];
  T f01 = (1.0 - z) * f[idx.incY().key] + z * f[idx.incY().incZ().key];
  T f00 = (1.0 - z) * f[idx.key] + z * f[idx.incZ().key];
  T f1 = y * f11 + (1.0 - y) * f10;
  T f0 = y * f01 + (1.0 - y) * f00;
  return x * f1 + (1.0 - x) * f0;
}

TEST_CASE("Invoking kernels on multi_array", "[multi_array][kernel]") {
  uint32_t N1 = 100, N2 = 300;
  auto ext = extent(N1, N2);
  auto array = make_multi_array<float, MemoryModel::host_device>(ext);
  REQUIRE(array.host_allocated() == true);
  REQUIRE(array.dev_allocated() == true);

  kernel_launch(
      [] __device__ (auto p, float value, auto ext) {
        for (auto idx : grid_stride_range(0u, ext.size())) {
          p[idx] = value;
        }
      }, array.get_ptr(), 3.0f, ext);
  CudaSafeCall(hipDeviceSynchronize());

  array.copy_to_host();

  for (auto idx : array.indices()) {
    REQUIRE(array[idx] == 3.0f);
  }
}

TEST_CASE("Different indexing on multi_array",
          "[multi_array][kernel]") {
  Logger::init(0, LogLevel::debug);
  uint32_t N1 = 32, N2 = 32;
  // Extent ext(1, N2, N1);
  auto ext = extent(N2, N1);
  // multi_array<float, idx_row_major_t<>> array(
  // auto array = make_multi_array<float, MemoryModel::device_managed, idx_zorder_t>(ext);
  auto array = make_multi_array<float, MemoryModel::device_managed, idx_row_major_t>(ext);

  // assign_idx_array<<<128, 512>>>(array.get_ptr(), ext);
  kernel_launch(
      [] __device__(auto p, auto ext) {
        for (auto i : grid_stride_range(0u, ext.size())) {
          auto idx = p.idx_at(i, ext);
          auto pos = idx.get_pos();
          p[i] = pos[0] * pos[1];
        }
      }, array.get_ptr(), ext);
  CudaSafeCall(hipDeviceSynchronize());

  for (auto idx : array.indices()) {
    auto pos = idx.get_pos();
    REQUIRE(array[idx] == Approx((float)pos[0] * pos[1]));
  }
}

// TEST_CASE("Performance of different indexing schemes",
//           "[multi_array][kernel]") {
//   std::default_random_engine g;
//   std::uniform_real_distribution<double> dist(0.0, 1.0);

//   uint32_t N1 = 256, N2 = 256, N3 = 256;
//   Extent ext(N1, N2, N3);
//   // multi_array<float, idx_row_major_t<>> array(
//   multi_array<float, idx_col_major_t<>> v1(ext,
//                                            MemoryModel::device_managed);
//   multi_array<float, idx_zorder_t<>> v2(ext,
//                                         MemoryModel::device_managed);

//   uint32_t x, y, z;
//   for (auto idx : v1.indices()) {
//     idx.decode(x, y, z);
//     v1[idx] = x + y - z;
//   }
//   for (auto idx : v2.indices()) {
//     idx.decode(x, y, z);
//     v2[idx] = x + y - z;
//   }

//   // Generate M random numbers
//   int M = 100000;
//   buffer_t<float> xs(M, MemoryModel::device_managed);
//   buffer_t<float> ys(M, MemoryModel::device_managed);
//   buffer_t<float> zs(M, MemoryModel::device_managed);
//   buffer_t<float> result(M, MemoryModel::device_managed);
//   for (int n = 0; n < M; n++) {
//     xs[n] = dist(g);
//     ys[n] = dist(g);
//     zs[n] = dist(g);
//   }

//   auto interp_kernel = [N1, N2, N3, M] __device__(
//                            auto f, float* result, float* xs, float* ys,
//                            float* zs, Extent ext) {
//     float dx = 1.0f / N1;
//     float dy = 1.0f / N2;
//     float dz = 1.0f / N3;
//     for (int i : grid_stride_range(0, M)) {
//       uint32_t nx = floor(xs[i] / dx);
//       uint32_t ny = floor(ys[i] / dy);
//       uint32_t nz = floor(zs[i] / dz);
//       if (nx > 0 && nx < N1 - 1 && ny > 0 && ny < N2 - 1 && nz > 0 &&
//           nz < N3 - 1) {
//         float x = (xs[i] - nx * dx) / dx;
//         float y = (ys[i] - ny * dy) / dy;
//         float z = (zs[i] - nz * dz) / dz;
//         result[i] =
//             interp(f.p, x, y, z,
//                    typename decltype(f)::idx_type(nx, ny, nz, ext));
//       }
//     }
//   };

//   timer::stamp();
//   kernel_launch(interp_kernel, v1.get_ptr(), result.dev_ptr(),
//                 xs.dev_ptr(), ys.dev_ptr(), zs.dev_ptr(), ext);
//   timer::show_duration_since_stamp("normal indexing", "ms");

//   timer::stamp();
//   kernel_launch(interp_kernel, v2.get_ptr(), result.dev_ptr(),
//                 xs.dev_ptr(), ys.dev_ptr(), zs.dev_ptr(), ext);
//   timer::show_duration_since_stamp("morton indexing", "ms");
// }

#endif
