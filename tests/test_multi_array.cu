/*
 * Copyright (c) 2020 Alex Chen.
 * This file is part of Aperture (https://github.com/fizban007/Aperture4.git).
 *
 * Aperture is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * Aperture is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "catch.hpp"
#include "core/constant_mem_func.h"
#include "core/constant_mem.h"
#include "core/multi_array.hpp"
#include "core/multi_array_exp.hpp"
#include "core/ndptr.hpp"
#include "utils/logger.h"
#include "utils/interpolation.hpp"
#include "utils/kernel_helper.hpp"
#include "utils/range.hpp"
#include "utils/timer.h"
#include <algorithm>
#include <random>
#include <thrust/tuple.h>

using namespace Aperture;

#ifdef CUDA_ENABLED

TEST_CASE("Invoking kernels on multi_array", "[multi_array][kernel]") {
  uint32_t N1 = 100, N2 = 300;
  auto ext = extent(N1, N2);
  auto array = make_multi_array<float>(ext, MemType::host_device);
  REQUIRE(array.host_allocated() == true);
  REQUIRE(array.dev_allocated() == true);

  kernel_launch(
      [] __device__(auto p, float value, auto ext) {
        for (auto idx : grid_stride_range(0u, ext.size())) {
          p[idx] = value;
        }
      },
      array.dev_ndptr(), 3.0f, ext);
  CudaSafeCall(hipDeviceSynchronize());

  array.copy_to_host();

  for (auto idx : array.indices()) {
    REQUIRE(array[idx] == 3.0f);
  }
}

TEST_CASE("Different indexing on multi_array",
          "[multi_array][kernel]") {
  Logger::init(0, LogLevel::debug);
  uint32_t N1 = 32, N2 = 32;
  // Extent ext(1, N2, N1);
  auto ext = extent(N2, N1);
  // multi_array<float, idx_row_major_t<>> array(
  auto array = make_multi_array<float,
                                idx_zorder_t>(ext, MemType::device_managed);
  // auto array = make_multi_array<float, MemType::device_managed,
  // idx_row_major_t>(ext);

  // assign_idx_array<<<128, 512>>>(array.dev_ndptr(), ext);
  kernel_launch(
      [] __device__(auto p, auto ext) {
        for (auto i : grid_stride_range(0u, ext.size())) {
          auto idx = p.idx_at(i, ext);
          auto pos = idx.get_pos();
          p[i] = pos[0] * pos[1];
        }
      },
      array.dev_ndptr(), ext);
  CudaSafeCall(hipDeviceSynchronize());

  for (auto idx : array.indices()) {
    auto pos = idx.get_pos();
    REQUIRE(array[idx] == Approx((float)pos[0] * pos[1]));
  }
}

TEST_CASE("Performance of different indexing schemes",
          "[multi_array][performance][kernel][.]") {
  init_morton(morton2dLUT, morton3dLUT);
  uint32_t N = 128;
  uint32_t N1 = N, N2 = N, N3 = N;
  std::default_random_engine g;
  std::uniform_real_distribution<float> dist(0.0, 1.0);
  std::uniform_int_distribution<uint32_t> cell_dist(0, N1 * N2 * N3);

  auto ext = extent(N1, N2, N3);
  // multi_array<float, idx_row_major_t<>> array(
  auto v1 = make_multi_array<float,
                             idx_col_major_t>(ext, MemType::host_device);
  auto v2 =
      make_multi_array<float, idx_zorder_t>(
          ext, MemType::host_device);

  for (auto idx : v1.indices()) {
    auto pos = idx.get_pos();
    v1[idx] = float(0.3 * pos[0] + 0.4 * pos[1] - pos[2]);
  }
  for (auto idx : v2.indices()) {
    auto pos = idx.get_pos();
    v2[idx] = float(0.3 * pos[0] + 0.4 * pos[1] - pos[2]);
  }
  for (auto idx : v1.indices()) {
    auto pos = idx.get_pos();
    REQUIRE(v1(pos[0], pos[1], pos[2]) == v2(pos[0], pos[1], pos[2]));
  }
  v1.copy_to_device();
  v2.copy_to_device();

  // Generate M random numbers
  int M = 1000000;
  buffer<float> xs(M, MemType::host_device);
  buffer<float> ys(M, MemType::host_device);
  buffer<float> zs(M, MemType::host_device);
  buffer<float> result1(M, MemType::host_device);
  buffer<float> result2(M, MemType::host_device);
  buffer<uint32_t> cells1(M, MemType::host_device);
  buffer<uint32_t> cells2(M, MemType::host_device);
  for (int n = 0; n < M; n++) {
    xs[n] = dist(g);
    ys[n] = dist(g);
    zs[n] = dist(g);
    cells1[n] = cell_dist(g);
    auto pos = v1.idx_at(cells1[n]).get_pos();
    auto idx = v2.get_idx(pos[0], pos[1], pos[2]);
    cells2[n] = idx.linear;
    result1[n] = 0.0f;
    result2[n] = 0.0f;
  }
  // std::sort(cells1.host_ptr(), cells1.host_ptr() + cells1.size());
  // std::sort(cells2.host_ptr(), cells2.host_ptr() + cells2.size());
  xs.copy_to_device();
  ys.copy_to_device();
  zs.copy_to_device();
  cells1.copy_to_device();
  cells2.copy_to_device();
  result1.copy_to_device();
  result2.copy_to_device();

  auto interp_kernel = [N1, N2, N3, M] __device__(
                           auto f, float* result, float* xs, float* ys,
                           float* zs, uint32_t* cells, auto ext) {
    for (uint32_t i : grid_stride_range(0, M)) {
      uint32_t cell = cells[i];
      auto idx = f.idx_at(cell, ext);
      auto pos = idx.get_pos();
      if (pos[0] < N1 - 1 && pos[1] < N2 - 1 && pos[2] < N3 - 1) {
        // result[i] = x;
        result[i] = lerp3(f, xs[i], ys[i], zs[i], idx);
      }
    }
  };

  hipDeviceSynchronize();

  timer::stamp();
  kernel_launch(interp_kernel, v1.dev_ndptr_const(), result1.dev_ptr(),
                xs.dev_ptr(), ys.dev_ptr(), zs.dev_ptr(),
                cells1.dev_ptr(), ext);
  hipDeviceSynchronize();
  timer::show_duration_since_stamp("normal indexing", "us");

  timer::stamp();
  kernel_launch(interp_kernel, v2.dev_ndptr_const(), result2.dev_ptr(),
                xs.dev_ptr(), ys.dev_ptr(), zs.dev_ptr(),
                cells2.dev_ptr(), ext);
  hipDeviceSynchronize();
  timer::show_duration_since_stamp("morton indexing", "us");

  result1.copy_to_host();
  result2.copy_to_host();

  for (auto idx : range(0ul, result1.size())) {
    REQUIRE(result1[idx] == result2[idx]);
  }
}

TEST_CASE("Assign and copy on device", "[multi_array][kernel]") {
  auto v1 = make_multi_array<float>(extent(30, 30));
  auto v2 = make_multi_array<float>(extent(30, 30));

  v1.assign_dev(3.0f);
  v1.copy_to_host();
  for (auto idx : v1.indices()) {
    REQUIRE(v1[idx] == 3.0f);
  }
}

TEST_CASE("Add ndptr on device", "[multi_array][exp_template]") {
  auto ext = extent(30, 30);
  auto v1 = make_multi_array<float>(ext);
  auto v2 = make_multi_array<float>(ext);
  auto v3 = make_multi_array<float>(ext);

  v1.assign_dev(1.0f);
  v2.assign_dev(2.0f);

  kernel_launch({30, 30}, [ext]__device__(auto p1, auto p2, auto p3) {
      using idx_t = default_idx_t<2>;
      for (auto idx : grid_stride_range(idx_t(0, ext), idx_t(ext.size(), ext))) {
        p3[idx] = (p1 * p2)[idx];
      }
    }, v1.dev_ndptr_const(), v2.dev_ndptr_const(), v3.dev_ndptr());
  CudaSafeCall(hipDeviceSynchronize());
  CudaCheckError();

  v3.copy_to_host();
  for (auto idx : v3.indices()) {
    REQUIRE(v3[idx] == 2.0f);
  }
}


#endif
